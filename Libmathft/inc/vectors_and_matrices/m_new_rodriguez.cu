#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   m_new_rodriguez.cu                                 :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 13:18:18 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/26 13:19:17 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Calculates and returns the roation matrix from vector b to vector a.
*/

__host__ __device__
void	m_new_rodriguez(t_matrix *m, t_vec3 a, t_vec3 b)
{
	t_matrix	n;
	t_matrix	identity;
	t_vec3		v;
	double		angle;
	int			i;

	v = v_norm(v_cross(a, b));
	angle = acos(v_dot(a, b) / ((v_length(a) * v_length(b))));
	m_new_identity(&n);
	m_new_identity(&identity);
	n[1][0] = v.z;
	n[2][0] = -v.y;
	n[0][1] = -v.z;
	n[2][1] = v.x;
	n[0][2] = v.y;
	n[1][2] = -v.x;
	
	
	// scale n with sin angel
	// add the the result to a new identity
	// this is m
	// m = m_add(m_scale(n, sin(angle)), m_new_identity());
	
	m_scale(&n, sin(angle));
	m_add(&n, &identity);
	memcpy(m, &n, sizeof(t_matrix));
	
	
	// n au carre
	// scale le resultat (n) avec un truc de merde
	// add m
	// this is m

	m_mult(&n, &n);
	m_scale(&n, 1 - cos(angle));
	m_add(m, &n);

	// m = m_add(m, m_scale(m_mult(n, n), 1 - cos(angle)));
	
	
	
	i = -1;
	while (++i < 3)
	{
		(*m)[3][i] = 0;
		(*m)[i][3] = 0;
	}
	(*m)[3][3] = 1;
}
