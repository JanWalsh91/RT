#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ft_sqrtfi.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/10 13:36:57 by tgros             #+#    #+#             */
/*   Updated: 2017/05/10 13:57:16 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

__host__ __device__
t_im	ft_sqrtfi(t_im im)
{
	float	r;
	float	x;
	float	y;
	t_im	res;

	r = sqrtf(im.r * im.r + im.i * im.i);
	y = sqrtf((r - im.r) / 2.0);
	x = im.i / (2.0 * y);
	res.r = x;
	res.i = y;
	return (res);
}

