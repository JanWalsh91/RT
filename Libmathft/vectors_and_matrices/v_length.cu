#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   v_length.cu                                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 13:24:51 by jwalsh            #+#    #+#             */
/*   Updated: 2017/03/31 15:26:14 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Returns the length of the vector.
*/

__host__ __device__
double	v_length(t_vec3 v)
{
	return (sqrt(v_dot(v, v)));
}
