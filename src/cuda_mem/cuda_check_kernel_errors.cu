/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_check_kernel_errors.cu                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/03 14:13:47 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 11:06:43 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

/*
** An error checking function to be used agfter every kernel launch.
*/

void	cuda_check_kernel_errors(void)
{
	hipError_t	errSync;
	hipError_t	errAsync;

	errSync = hipGetLastError();
	errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("1 Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("1 Async kernel error: %s\n", hipGetErrorString(errAsync));
	if (errSync != hipSuccess || errAsync != hipSuccess)
		exit(-1);
}
