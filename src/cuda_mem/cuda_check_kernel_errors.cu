/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_check_kernel_errors.cu                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/03 14:13:47 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 10:34:29 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

void	cuda_check_kernel_errors(void)
{
	hipError_t	errSync;
	hipError_t	errAsync;

	errSync = hipGetLastError();
	errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("1 Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("1 Async kernel error: %s\n", hipGetErrorString(errAsync));
	if (errSync != hipSuccess || errAsync != hipSuccess)
		exit(-1);
}
