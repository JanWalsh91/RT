/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_free.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/04/28 16:51:52 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

int	cuda_free(t_raytracing_tools *r, int all)
{
	if (r->update.resolution == 2 || all)
		hipFree(r->d_pixel_map);
	if (r->update.objects == 2 || all)
		hipFree(r->h_d_scene->objects);
	if (r->update.lights == 2 || all)
		hipFree(r->h_d_scene->lights);
	if (r->update.cameras == 2 || all)
		hipFree(r->h_d_scene->cameras);
	if (r->update.scene == 2 || all)
		hipFree(r->d_scene);
	return (0);
}
