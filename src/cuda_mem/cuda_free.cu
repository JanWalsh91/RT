/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_free.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/05/26 21:42:07 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

/*
**
*/

int	cuda_free(t_raytracing_tools *r, int all)
{
	int i;

	printf("cudafree\n");
	if (r->update.resolution == 2 || all)
		hipFree(r->d_pixel_map);
	C(1)
	if (r->update.objects == 2 || all)
		hipFree(r->h_d_scene->objects);
	C(2)
	if (r->update.lights == 2 || all)
		hipFree(r->h_d_scene->lights);
	C(3)
	if (r->update.cameras == 2 || all)
		hipFree(r->h_d_scene->cameras);
	C(4)
	if (r->update.scene == 2 || all)
		hipFree(r->d_scene);
	C(5)
	printf("selected photon address: %p\n", r->scene->selected_photons);
	if ((r->update.photon_map == 2 || all) && r->scene->selected_photons)
	{
		C(6)
		i = -1;
		// while (++i < r->scene->res.x * r->scene->res.y)
			// hipFree(r->scene->selected_photons[i]);
		hipFree(r->scene->selected_photons);
	}
	C(7)
	return (0);
}
