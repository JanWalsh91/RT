/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_free.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/06/05 10:34:31 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

/*
** Frees lists malloc'ed with hipMalloc.
*/

int	cuda_free(t_raytracing_tools *r, int all)
{
	printf("cudafree\n");
	if (r->update.resolution == 2 || all)
		hipFree(r->d_pixel_map);
	if (r->update.objects == 2 || all)
		hipFree(r->h_d_scene->objects);
	if (r->update.lights == 2 || all)
		hipFree(r->h_d_scene->lights);
	if (r->update.cameras == 2 || all)
		hipFree(r->h_d_scene->cameras);
	if (r->update.scene == 2 || all)
		hipFree(r->d_scene);
	return (0);
}
