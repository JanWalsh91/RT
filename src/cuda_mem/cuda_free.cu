/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_free.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/05/17 12:38:58 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

int	cuda_free(t_raytracing_tools *r, int all)
{
	int i;

	if (r->update.resolution == 2 || all)
		hipFree(r->d_pixel_map);
	if (r->update.objects == 2 || all)
		hipFree(r->h_d_scene->objects);
	if (r->update.lights == 2 || all)
		hipFree(r->h_d_scene->lights);
	if (r->update.cameras == 2 || all)
		hipFree(r->h_d_scene->cameras);
	if (r->update.scene == 2 || all)
		hipFree(r->d_scene);
	if (r->update.photon_map == 2 || all)
	{
		i = -1;
		while (++i < r->scene->res.x * r->scene->res.y)
			hipFree(r->scene->selected_photons[i]);
		hipFree(r->scene->selected_photons);
	}
	return (0);
}
