/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_free.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/04/28 13:31:04 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

int	cuda_free(t_raytracing_tools *r)
{
	if (r->update.resolution == 2)
		hipFree(r->d_pixel_map);
	if (r->update.objects == 2)
		hipFree(r->h_d_scene->objects);
	if (r->update.lights == 2)
		hipFree(r->h_d_scene->lights);
	if (r->update.cameras == 2)
		hipFree(r->h_d_scene->cameras);
	if (r->update.scene == 2)
		hipFree(r->d_scene);
	return (0);
}
