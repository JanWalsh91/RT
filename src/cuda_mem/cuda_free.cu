/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_free.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/06/08 14:14:09 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.cuh"

/*
** Frees lists malloc'ed with hipMalloc.
*/

int	cuda_free(t_raytracing_tools *r, int all)
{
	if (r->update.resolution == 2 || all)
		hipFree(r->d_pixel_map);
	if (r->update.objects == 2 || all)
		hipFree(r->h_d_scene->objects);
	if (r->update.lights == 2 || all)
		hipFree(r->h_d_scene->lights);
	if (r->update.cameras == 2 || all)
		hipFree(r->h_d_scene->cameras);
	if (r->update.scene == 2 || all)
		hipFree(r->d_scene);
	return (0);
}
