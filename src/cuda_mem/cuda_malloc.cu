/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 12:51:28 by tgros             #+#    #+#             */
/*   Updated: 2017/05/27 14:04:05 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"
#include "photon_mapping.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static void		reset_update_struct(t_raytracing_tools *r);
static void		cuda_malloc_scene(t_raytracing_tools *r);

/*
** Allocates memory on the device and on pinned memory the various sturctures in scene.
*/

int				cuda_malloc(t_raytracing_tools *r)
{
	t_scene		h_scene_to_array;

	r->scene->photon_count = 24;
	if (!(memcpy(&h_scene_to_array, r->scene, sizeof(t_scene) - (sizeof(void *) * 3))))
		exit (0);
	memcpy(r->h_d_scene, r->scene, sizeof(t_scene) - (sizeof(void *) * 3));
	cuda_malloc_photon_map(r);
	cuda_malloc_objects(r, &h_scene_to_array);
	cuda_malloc_lights(r, &h_scene_to_array);
	cuda_malloc_camera(r);
	cuda_malloc_scene(r);
	gpuErrchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	reset_update_struct(r);
	return (1);
}

static void		reset_update_struct(t_raytracing_tools *r)
{
	r->update.resolution = 0;
	r->update.objects = 0;
	r->update.lights = 0;
	r->update.cameras = 0;
	r->update.scene = 0;
	r->update.ray_depth = 0;
	r->update.render = 0;
	r->update.photon_map = 0;
}

static void		cuda_malloc_scene(t_raytracing_tools *r)
{
	if (r->update.resolution == 2)
	{
		gpuErrchk((hipHostMalloc(&r->d_pixel_map, sizeof(t_color) * r->scene->res.y * r->scene->res.x)));
		if (r->scene->is_3d)
			gpuErrchk((hipHostMalloc(&r->d_pixel_map_3d, sizeof(t_color) * r->scene->res.y * r->scene->res.x)));
	}
	if (r->update.ray_depth == 2)
	{
		gpuErrchk(hipSetDevice(0));
		hipDeviceSetLimit(hipLimitStackSize, 1024 * r->scene->ray_depth);
	}
	if (r->update.anaglyph == 2)
			gpuErrchk((hipHostMalloc(&r->d_pixel_map_3d, sizeof(t_color) * r->scene->res.y * r->scene->res.x)));
	if (r->update.scene == 2)
		gpuErrchk(hipMalloc(&r->d_scene, sizeof(t_scene)));
}
