/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 12:51:28 by tgros             #+#    #+#             */
/*   Updated: 2017/04/25 16:56:42 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

t_light			*list_to_array_lights(t_light *light);
t_object		*list_to_array_objects(t_object *object);
size_t			get_objects_array_length(t_object *objects);
size_t			get_lights_array_length(t_light *lights);




int	cuda_malloc(t_raytracing_tools *r)
{
	t_scene		h_scene_to_array;

	if (!(memcpy(&h_scene_to_array, r->scene, sizeof ( t_scene ) - (sizeof ( void * ) * 5)  )))
		exit (0);
	memcpy(r->h_d_scene, r->scene, sizeof ( t_scene ) - (sizeof ( void * ) * 5)  );
	if (r->update.resolution == 2)
	{
		gpuErrchk((hipHostMalloc(&r->d_pixel_map, sizeof(t_color) * r->scene->res.y * r->scene->res.x)));
	}
	if (r->update.ray_depth == 2)
	{
		gpuErrchk(hipSetDevice(0));
		hipDeviceSetLimit(hipLimitStackSize, 1024 * r->scene->ray_depth);
	}
	if (r->update.objects >= 1)
	{
		h_scene_to_array.objects = list_to_array_objects(r->scene->objects);
		if (r->update.objects == 2)
			gpuErrchk(hipMalloc(&(r->h_d_scene->objects), get_objects_array_length(h_scene_to_array.objects)));
		gpuErrchk((hipMemcpy(r->h_d_scene->objects, h_scene_to_array.objects, get_objects_array_length(h_scene_to_array.objects), hipMemcpyHostToDevice)));
		free(h_scene_to_array.objects);
	}
	if (r->update.lights >= 1)
	{
		h_scene_to_array.lights = list_to_array_lights(r->scene->lights);
		if (r->update.lights == 2)
			gpuErrchk(hipMalloc(&(r->h_d_scene->lights), get_lights_array_length(h_scene_to_array.lights)));
		gpuErrchk((hipMemcpy(r->h_d_scene->lights, h_scene_to_array.lights, get_lights_array_length(h_scene_to_array.lights), hipMemcpyHostToDevice)));
		free(h_scene_to_array.lights);
	}
	if (r->update.cameras >= 1)
	{
		if (r->update.cameras == 2)
		{
			printf("Malloc cameras\n");
			gpuErrchk(hipMalloc(&(r->h_d_scene->cameras), sizeof(t_camera)));
		}
		gpuErrchk((hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice)));
	}
	if (r->update.scene == 2)
	{
		printf("malloc d_scene\n");
		gpuErrchk(hipMalloc(&r->d_scene, sizeof(t_scene)));
	}
	gpuErrchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	r->update.resolution = 0;
	r->update.objects = 0;
	r->update.lights = 0;
	r->update.cameras = 0;
	r->update.scene = 0;
	r->update.ray_depth = 0;
	r->update.render = 1;
	// printf("RENDER ADDR %p\n", &r->update.render);
	C(2)
	return (1);
}



t_object	*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	array = (t_object *)malloc(sizeof(t_object) * (size + 1)); // malloc error
	array[size].type = T_INVALID_TOKEN;
	object = head;
	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object)); // stack memcpy ?
		object = object->next;
	}
	return (array);
}

size_t			get_objects_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}

t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light	*head;
	t_light	*array;

	size = 0;
	head = light;
	while (light)
	{
		++size;
		light = light->next;
	}
	array = (t_light *)malloc(sizeof(t_light) * (size + 1)); // malloc error
	// bzero(array, sizeof(t_light) * (size + 1));
	array[size].col = v_new(NAN, NAN, NAN);
	light = head;
	size = -1;
	// printf("COLOR: %f\n", array[size].col.x);
	while (light)
	{
		array[++size].col = v_new(NAN, NAN, NAN);
		memcpy(&array[size], light, sizeof(t_light));
		light = light->next;
		// printf("%p\n", light);
	}
	// printf("COLOR: %f\n", array[size].col.x);
	return (array);
}

size_t			get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	// printf("%f\n", lights[size].col.x);
	while (!v_isnan(lights[size].col))
		++size;
	// C(3)
	return ((size + 1) * sizeof(t_light));
}
