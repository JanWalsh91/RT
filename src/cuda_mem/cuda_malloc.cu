/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 12:51:28 by tgros             #+#    #+#             */
/*   Updated: 2017/05/18 14:38:51 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"
#include "photon_mapping.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

t_light			*list_to_array_lights(t_light *light);
t_object		*list_to_array_objects(t_object *object);
size_t			get_objects_array_length(t_object *objects);
size_t			get_lights_array_length(t_light *lights);

int	cuda_malloc(t_raytracing_tools *r)
{
	t_scene		h_scene_to_array;
	int			i;

	if (!(memcpy(&h_scene_to_array, r->scene, sizeof ( t_scene ) - (sizeof ( void * ) * 6)  )))
		exit (0);
	memcpy(r->h_d_scene, r->scene, sizeof ( t_scene ) - (sizeof ( void * ) * 6)  );
	// sleep(2);
	if (r->scene->is_photon_mapping && r->update.photon_map == 2 ) 
	{
		r->scene->photon_count = 1000;
		gpuErrchk(hipHostMalloc(&(r->h_d_scene->photon_list), sizeof(t_photon *) * (r->scene->photon_count + 1)));
		i = -1;
		while (++i < r->scene->photon_count)
			gpuErrchk(hipHostMalloc(&(r->h_d_scene->photon_list[i]), sizeof(t_photon) * r->scene->ray_depth));
		//allocated pinned mem list in h_d_scene
		//SWICTH 10 WITH K (NUM OF PHOTONS TO GATHER)
		gpuErrchk(hipHostMalloc(&(r->h_d_scene->selected_photons), sizeof(t_selected_photon *) * (r->scene->res.x * r->scene->res.y)));
		i = -1;
		while (++i < r->scene->res.x * r->scene->res.y)
			gpuErrchk(hipHostMalloc(&(r->h_d_scene->selected_photons[i]), sizeof(t_selected_photon) * (10 + 1)));
		r->scene->photon_map = r->h_d_scene->photon_map;
		printf("selected_photons: [%p]\n", r->h_d_scene->selected_photons);
		printf("selected_photons: [%p]\n", r->h_d_scene->selected_photons[1]);
	}
	// printf("-----%p and %p\n", r->scene->photon_map, r->h_d_scene->photon_map);
	// exit(0);
	if (r->update.resolution == 2)
	{
		gpuErrchk((hipHostMalloc(&r->d_pixel_map, sizeof(t_color) * r->scene->res.y * r->scene->res.x)));
		if (r->scene->is_3d)
			gpuErrchk((hipHostMalloc(&r->d_pixel_map_3d, sizeof(t_color) * r->scene->res.y * r->scene->res.x)));
	}
	if (r->update.ray_depth == 2)
	{
		gpuErrchk(hipSetDevice(0));
		hipDeviceSetLimit(hipLimitStackSize, 1024 * r->scene->ray_depth);
	}
	// printf("Objects: %d\n", r->update.objects);
	if (r->update.objects >= 1)
	{
		h_scene_to_array.objects = list_to_array_objects(r->scene->objects);
		if (r->update.objects == 2)
			gpuErrchk(hipMalloc(&(r->h_d_scene->objects), get_objects_array_length(h_scene_to_array.objects)));
		// printf("Cuda memcpy avec %lu bytes\n", get_objects_array_length(h_scene_to_array.objects));
		gpuErrchk((hipMemcpy(r->h_d_scene->objects, h_scene_to_array.objects, get_objects_array_length(h_scene_to_array.objects), hipMemcpyHostToDevice)));
		free(h_scene_to_array.objects);
	}
	// printf("Lights: %d\n", r->update.lights);
	if (r->update.lights >= 1)
	{
		h_scene_to_array.lights = list_to_array_lights(r->scene->lights);
		if (r->update.lights == 2)
			gpuErrchk(hipMalloc(&(r->h_d_scene->lights), get_lights_array_length(h_scene_to_array.lights)));
		// printf("Cuda memcpy avec %lu bytes\n", get_lights_array_length(h_scene_to_array.lights));
		gpuErrchk((hipMemcpy(r->h_d_scene->lights, h_scene_to_array.lights, get_lights_array_length(h_scene_to_array.lights), hipMemcpyHostToDevice)));
		free(h_scene_to_array.lights);
	}
	if (r->update.cameras >= 1)
	{
		if (r->update.cameras == 2)
		{
			gpuErrchk(hipMalloc(&(r->h_d_scene->cameras), sizeof(t_camera)));
		}
		if (r->scene->is_3d) // l'enlever si on decoche l'opt 3d
			r->scene->cameras->filter = F_LEFT_RED;
		gpuErrchk((hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice)));
	}
	if (r->update.scene == 2)
	{
		printf("malloc d_scene\n");
		gpuErrchk(hipMalloc(&r->d_scene, sizeof(t_scene)));
	}
	gpuErrchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	r->update.resolution = 0;
	r->update.objects = 0;
	r->update.lights = 0;
	r->update.cameras = 0;
	r->update.scene = 0;
	r->update.ray_depth = 0;
	r->update.render = 0;
	r->update.photon_map = 0;
	// printf("Resolution: %d\n", r->update.resolution);
	// printf("RENDER ADDR %p\n", &r->update.render);
	return (1);
}



t_object	*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	array = (t_object *)malloc(sizeof(t_object) * (size + 1)); // malloc error
	array[size].type = T_INVALID_TOKEN;
	object = head;
	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object)); // stack memcpy ?
		object = object->next;
	}
	return (array);
}

size_t			get_objects_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	if (!objects)
		return (0);
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}

t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light	*head;
	t_light	*array;

	size = 0;
	head = light; 
	while (light)
	{
		++size;
		light = light->next;
	}
	printf("Size of light array : %d\n", size);
	array = (t_light *)malloc(sizeof(t_light) * (size + 1)); // malloc error
	// bzero(array, sizeof(t_light) * (size + 1));
	array[size].col = v_new(NAN, NAN, NAN);
	light = head;
	size = -1;
	// printf("COLOR: %f\n", array[size].col.x);
	while (light)
	{
		array[++size].col = v_new(NAN, NAN, NAN);
		memcpy(&array[size], light, sizeof(t_light));
		light = light->next;
		// printf("%p\n", light);
	}
	// printf("COLOR: %f\n", array[size].col.x);
	return (array);
}

size_t			get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	// printf("%f\n", lights[size].col.x);
	while (!v_isnan(lights[size].col))
		++size;
	// C(3)
	return ((size + 1) * sizeof(t_light));
}
