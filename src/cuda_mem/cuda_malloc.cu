/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 12:51:28 by tgros             #+#    #+#             */
/*   Updated: 2017/06/05 14:31:01 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

static void		reset_update_struct(t_raytracing_tools *r);
static void		cuda_malloc_scene(t_raytracing_tools *r);

/*
** Allocates memory on the device and on pinned memory the various sturctures
** in the scene.
*/

int				cuda_malloc(t_raytracing_tools *r)
{
	t_scene		h_scene_to_array;

	memcpy(&h_scene_to_array, r->scene, sizeof(t_scene) - (sizeof(void *) * 3));
	memcpy(r->h_d_scene, r->scene, sizeof(t_scene) - (sizeof(void *) * 3));
	cuda_malloc_objects(r, &h_scene_to_array);
	cuda_malloc_lights(r, &h_scene_to_array);
	cuda_malloc_camera(r);
	cuda_malloc_scene(r);
	gpu_errchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene),
		hipMemcpyHostToDevice));
	reset_update_struct(r);
	return (1);
}

static void		reset_update_struct(t_raytracing_tools *r)
{
	r->update.resolution = 0;
	r->update.objects = 0;
	r->update.lights = 0;
	r->update.cameras = 0;
	r->update.scene = 0;
	r->update.ray_depth = 0;
	r->update.render = 0;
	r->update.photon_map = 0;
}

static void		cuda_malloc_scene(t_raytracing_tools *r)
{
	if (r->update.resolution == 2)
	{
		gpu_errchk((hipHostMalloc(&r->d_pixel_map, sizeof(t_color) *
			r->scene->res.y * r->scene->res.x)));
		if (r->scene->is_3d)
			gpu_errchk((hipHostMalloc(&r->d_pixel_map_3d, sizeof(t_color) *
				r->scene->res.y * r->scene->res.x)));
	}
	if (r->update.ray_depth == 2)
	{
		gpu_errchk(hipSetDevice(0));
		hipDeviceSetLimit(hipLimitStackSize, 1024 * r->scene->ray_depth);
	}
	if (r->update.anaglyph == 2)
		gpu_errchk((hipHostMalloc(&r->d_pixel_map_3d, sizeof(t_color) *
			r->scene->res.y * r->scene->res.x)));
	if (r->update.scene == 2)
		gpu_errchk(hipMalloc(&r->d_scene, sizeof(t_scene)));
}
