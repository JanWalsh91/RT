/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_camera.cu                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:13:20 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/06 11:00:48 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

/*
** Allocated memory on the device for the first camera.
*/

bool		cuda_malloc_camera(t_raytracing_tools *r)
{
	if (r->update.cameras >= 1)
	{
		if (r->update.cameras == 2)
		{
			if(test_cuda_malloc((void **)(&r->h_d_scene->cameras), sizeof(t_camera)) == false)
				return(false);
		}
		if (r->scene->is_3d)
			r->scene->cameras->filter = F_LEFT_RED;
		gpu_errchk((hipMemcpy(r->h_d_scene->cameras, r->scene->cameras,
			sizeof(t_camera), hipMemcpyHostToDevice)));
	}
	return(true);
}
