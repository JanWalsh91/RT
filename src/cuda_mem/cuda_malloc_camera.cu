/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_camera.cu                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:13:20 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 10:58:57 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

/*
** Allocated memory on the device for the first camera.
*/

void		cuda_malloc_camera(t_raytracing_tools *r)
{
	if (r->update.cameras >= 1)
	{
		if (r->update.cameras == 2)
			gpu_errchk(hipMalloc(&(r->h_d_scene->cameras), sizeof(t_camera)));
		if (r->scene->is_3d)
			r->scene->cameras->filter = F_LEFT_RED;
		gpu_errchk((hipMemcpy(r->h_d_scene->cameras, r->scene->cameras,
			sizeof(t_camera), hipMemcpyHostToDevice)));
	}
}
