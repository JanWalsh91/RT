/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_camera.cu                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:13:20 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/18 16:14:28 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void		cuda_malloc_camera(t_raytracing_tools *r)
{
	if (r->update.cameras >= 1)
	{
		if (r->update.cameras == 2)
			gpuErrchk(hipMalloc(&(r->h_d_scene->cameras), sizeof(t_camera)));
		if (r->scene->is_3d) // l'enlever si on decoche l'opt 3d
			r->scene->cameras->filter = F_LEFT_RED;
		gpuErrchk((hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice)));
	}
}