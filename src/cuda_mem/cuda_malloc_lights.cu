/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_lights.cu                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:10:25 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/18 16:12:29 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static t_light		*list_to_array_lights(t_light *light);
static size_t		get_lights_array_length(t_light *lights);

void				cuda_malloc_lights(t_raytracing_tools *r, t_scene *h_scene_to_array)
{
	if (r->update.lights >= 1)
	{
		h_scene_to_array->lights = list_to_array_lights(r->scene->lights);
		if (r->update.lights == 2)
			gpuErrchk(hipMalloc(&(r->h_d_scene->lights), get_lights_array_length(h_scene_to_array->lights)));
		gpuErrchk((hipMemcpy(r->h_d_scene->lights, h_scene_to_array->lights, get_lights_array_length(h_scene_to_array->lights), hipMemcpyHostToDevice)));
		free(h_scene_to_array->lights);
	}
}

static t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light	*head;
	t_light	*array;

	size = 0;
	head = light; 
	while (light)
	{
		++size;
		light = light->next;
	}
	if (!(array = (t_light *)malloc(sizeof(t_light) * (size + 1))))
	{
		write(2, "Malloc error.\n", 14);
		exit(1);
	}
	array[size].col = v_new(NAN, NAN, NAN);
	light = head;
	size = -1;
	while (light)
	{
		array[++size].col = v_new(NAN, NAN, NAN);
		memcpy(&array[size], light, sizeof(t_light));
		light = light->next;
	}
	return (array);
}

static size_t		get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	while (!v_isnan(lights[size].col))
		++size;
	return ((size + 1) * sizeof(t_light));
}
