/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_objects.cu                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:06:29 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/22 11:06:58 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static t_object		*list_to_array_objects(t_object *object);
static size_t		get_objects_array_length(t_object *objects);

void				cuda_malloc_objects(t_raytracing_tools *r, t_scene *h_scene_to_array)
{
	if (r->update.objects >= 1)
	{
		h_scene_to_array->objects = list_to_array_objects(r->scene->objects);
		if (r->update.objects == 2)
			gpuErrchk(hipMalloc(&(r->h_d_scene->objects), get_objects_array_length(h_scene_to_array->objects)));
		gpuErrchk((hipMemcpy(r->h_d_scene->objects, h_scene_to_array->objects, get_objects_array_length(h_scene_to_array->objects), hipMemcpyHostToDevice)));
		free(h_scene_to_array->objects);
	}
}

static t_object		*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	if (!(array = (t_object *)malloc(sizeof(t_object) * (size + 1))))
	{
		write(2, "Malloc error.\n", 14);
		exit(1);
	}
	array[size].type = T_INVALID_TOKEN;
	object = head;
	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object));
		object = object->next;
	}
	return (array);
}

static size_t		get_objects_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	if (!objects)
		return (0);
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}
