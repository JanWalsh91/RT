/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_photon_map.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:15:30 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/26 21:33:33 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"
#include "photon_mapping.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void	cuda_malloc_photon_map(t_raytracing_tools *r)
{
	int	i;
	int	res;

	if (r->scene->is_photon_mapping && r->update.photon_map == 2) 
	{
		res = (r->scene->res.x * r->scene->res.y);
		
		printf("cuda_malloc_photon_map\n");
		gpuErrchk(hipHostMalloc(&(r->h_d_scene->photon_list), sizeof(t_photon) * PHOTON_BOUNCE_MAX * r->scene->photon_count));
		printf("Size mallocated : %lu\n", sizeof(t_photon[PHOTON_GATHER_MAX]) * (r->scene->photon_count + 1));
		//list of photons we gather per thread.
		gpuErrchk(hipHostMalloc(&(r->h_d_scene->selected_photons), sizeof(t_selected_photon) * PHOTON_GATHER_MAX * res));
		printf("END cuda_malloc_photon_map\n");
		// exit(0);
	}
}
