/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_photon_map.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:15:30 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/22 11:07:55 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.h"
#include "photon_mapping.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void	cuda_malloc_photon_map(t_raytracing_tools *r)
{
	int i;
	
	// printf("cuda_malloc_photon_map: is_photon_mapping: [%d], r->update.photon_map: [%d]\n", r->scene->is_photon_mapping, r->update.photon_map);
	// printf("selected photon address: %p\n", r->scene->selected_photons);
	if (r->scene->is_photon_mapping && r->update.photon_map == 2 ) 
	{
		gpuErrchk(hipHostMalloc(&(r->h_d_scene->photon_list), sizeof(t_photon *) * (r->scene->photon_count + 1)));
		// printf("size of photon: %lu\n", sizeof(t_photon));
		i = -1;
		while (++i < r->scene->photon_count)
			gpuErrchk(hipHostMalloc(&(r->h_d_scene->photon_list[i]), sizeof(t_photon) * r->scene->ray_depth));
		//SWICTH 10 WITH K (NUM OF PHOTONS TO GATHER)
		gpuErrchk(hipHostMalloc(&(r->h_d_scene->selected_photons), sizeof(t_selected_photon *) * (r->scene->res.x * r->scene->res.y)));
		i = -1;
		while (++i < r->scene->res.x * r->scene->res.y)
			gpuErrchk(hipHostMalloc(&(r->h_d_scene->selected_photons[i]), sizeof(t_selected_photon) * (10 + 1)));
		r->scene->photon_map = r->h_d_scene->photon_map;
		// printf("selected_photons: [%p]\n", r->h_d_scene->selected_photons);
		// printf("selected_photons: [%p]\n", r->h_d_scene->selected_photons[1]);
	}
	// printf("selected photon address: %p\n", r->scene->selected_photons);
	// printf("end cuda_malloc_photon_map\n");
}