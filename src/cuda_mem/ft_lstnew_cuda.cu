/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ft_lstnew_cuda.cu                                  :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/11/06 14:37:26 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 10:59:07 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

/*
** Allocated memory for and returns a new link with the content and
** content size passed as parameters. Cuda version
*/

#include "../../inc/rt.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

t_list		*ft_lstnew_cuda(void const *content, size_t content_size)
{
	t_list	*list;
	void	*new_content;

	if (hipHostMalloc((void **)&new_content, content_size) != 0)
		return (NULL);
	if (!content)
	{
		new_content = NULL;
		content_size = 0;
	}
	else
		memcpy(new_content, content, content_size);
	if (hipHostMalloc((void **)&list, sizeof(t_list)) != 0)
		return (NULL);
	list->content = new_content;
	list->content_size = content_size;
	list->next = NULL;
	return (list);
}
