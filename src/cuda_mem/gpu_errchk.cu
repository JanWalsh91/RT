/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   gpu_errchk.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/04 15:30:53 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 10:27:33 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "rt.cuh"

/*
** Checks a cuda function return value for errors and exits with an error 
** message.
*/

__host__
void gpu_errchk(int code)
{
	int l;
	const char *s;

	s = hipGetErrorString((hipError_t)code);
	l = strlen(s);
	if (code != hipSuccess)
	{
		write(2, "GPUassert: ", 11);
		write(2, s, l);
		write(2, "\n", 1);
		exit(code);
	}
}
