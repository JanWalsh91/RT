/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_free.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/05/26 21:42:07 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

bool	test_malloc(void *to_malloc, size_t size, t_gtk_tools *g)
{
	int i;

	i = 0;
	while (i < 5)
	{
		if (hipMalloc(&to_malloc, sizeof(size)))
			return(true);
		i++;
	}
	display_error_popup(NULL, g, "hipMalloc Fail.");
	return(false);
}