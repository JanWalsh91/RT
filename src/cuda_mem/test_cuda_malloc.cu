/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   test_cuda_malloc.cu                                :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/06/05 11:00:58 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include <stdlib.h>

/*
** Tries to call hipMalloc until success. Displays popup after 5 tries.
*/

bool	test_cuda_malloc(void **to_malloc, size_t size)
{
	int i;
	const char *s;
	char cuda[1];
	int code;

	i = 0;

	while (i < CUDA_TEST)
	{
		if ((code = hipMalloc(to_malloc, size)) == 0)
			return (true);
		i++;
	}
	s = hipGetErrorString((hipError_t)code);
	cuda[0] = CUDA_TEST + 48;
	if (code != hipSuccess)
	{
		write(2, "\e[1;93mcudaMalloc fail after ", 17);
		write(2, cuda, 1);
		write(2, " attempts\n", 9);
		write(2, "GPUassert : ", 12);
		write(2, s, strlen(s));
		write(2, "\e[0m\n", 5);
		exit(code);
	}
	return (false);
}
