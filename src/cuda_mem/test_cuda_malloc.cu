/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   test_cuda_malloc.cu                                :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by tgros             #+#    #+#             */
/*   Updated: 2017/06/05 11:00:58 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Tries to call hipMalloc until success. Displays popup after 5 tries.
*/

bool	test_malloc(void *to_malloc, size_t size, t_gtk_tools *g)
{
	int i;

	i = 0;
	while (i < 5)
	{
		if (hipMalloc(&to_malloc, sizeof(size)))
			return (true);
		i++;
	}
	display_error_popup(NULL, g, "hipMalloc Fail.");
	return (false);
}
