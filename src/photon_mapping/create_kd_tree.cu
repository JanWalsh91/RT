/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   create_kd_tree.cu                                  :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/08 12:37:57 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/02 09:51:14 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "photon_mapping.h"
#include "hip/hip_runtime.h"

static t_kd_tree	*create_kd_node(t_photon photon);

/*
** Creates a linked list with t_kd_tree nodes based off the photons gathered during the photon shooting
*/

void		create_kd_tree(t_photon *photon_list, t_kd_tree **root, int photon_count)
{
	int i;
	int y;
	t_kd_tree *node;

	// printf("create kd tree: photoncount: %d\n", photon_count);
	node = NULL;
	i = -1;
	// printf("create kd tree--\n");
	// printf("create kd tree: [%f, %f, %f]\n", photon_list[0]->pos.x, photon_list[0]->pos.y, photon_list[0]->pos.z);
	while (++i < photon_count)
	{
		y = -1;
		while (++y < MAX_RAY_DEPTH && !v_isnan((photon_list + i)[y].dir))
		{
			if (!node)
			{
				*root = create_kd_node((photon_list + i)[y]);
				if (*root)
					node = *root;
			}
			else
			{
				node->right = create_kd_node((photon_list + i)[y]);
				if (node->right)
					node = node->right;
			}
		}  
	}  
}

static t_kd_tree		*create_kd_node(t_photon photon)
{
	t_kd_tree	*node; 
	int ret;

	ret = hipHostMalloc(&(node), sizeof(t_kd_tree));
	if (ret)
	{
		printf("cudamallocerror for node\n");
		usleep(1000000);
		return (NULL);
	}
	node->pos = photon.pos;
	node->dir = photon.dir;
	node->col = photon.col;
	node->n = photon.n;
	node->right = NULL;
	node->left = NULL;
	return (node);
}

void		free_kd_tree(t_kd_tree *root)
{
	if (!root)
		return ;
	if (root->right)
		free_kd_tree(root->right);
	if (root->left)
		free_kd_tree(root->left);
	hipFree(root);
}