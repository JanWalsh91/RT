#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   create_photon_map.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/08 13:48:43 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 10:28:37 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "photon_mapping.h"
#include "rt.cuh"
#include "../../inc/cuda_call.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

void			print_photons(t_kd_tree *tree);
static int		shoot_photon_group(t_raytracing_tools *r, size_t photon_count);
static void		init_photon_group(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list);
static float	get_total_intensity(t_light *lights);
static void		init_photon(t_photon *photon, t_light *light);
static void		shoot_photon_wrapper(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list);
__global__
static void		shoot_photon(t_scene *scene, t_photon *init_photon_list, int photon_count, float rand_numbers[]);
__device__
static t_ray	init_kernel_photon(t_raytracing_tools *r, t_photon photon, float rand_numbers[]);

/*
** Frees previous photon map, shoots photons, creates and sorts the tree.
*/

void			update_photon_map(t_raytracing_tools *r)
{
	printf("update_photon_map\n");
	int			ret;
	t_kd_tree 	*sorted;
	
	// srand(time(NULL));
	ret = shoot_photon_group(r, r->scene->photon_count);
	
	if (r->scene->photon_kd_tree)
		free_kd_tree(r->scene->photon_kd_tree);
	r->scene->photon_kd_tree = NULL;
	create_kd_tree(r->h_d_scene->photon_list, &r->scene->photon_kd_tree, r->scene->photon_count);
	// exit(0);
	// // printf("-----%p and %p\n", r->scene->photon_kd_tree, r->d_scene->photon_kd_tree);
	// // exit(0);
	sorted = NULL;
	sort_kd_tree(&r->scene->photon_kd_tree, 0, &sorted); // CAUSES SEGFAULT
	// exit(0);
	// r->scene->photon_kd_tree = sorted;
	// printf("done creating photon map\n");
	t_kd_tree *p = sorted;
	// printf("first photon: pos: [%f, %f, %f], dir: [%f, %f, %f], col: [%d, %d, %d], n: [%f, %f, %f]\n",
	// p->pos.x, p->pos.y, p->pos.z, p->dir.x, p->dir.y, p->dir.z, p->col.r, p->col.g, p->col.b, p->n.x, p->n.y, p->n.z);

	// print_photons(r->scene->photon_kd_tree); 
}

static int	shoot_photon_group(t_raytracing_tools *r, size_t photon_count)
{
	// printf("shoot_photon_group\n");
	t_photon	*init_photon_list;
	hipError_t		code;

	if ((code = hipHostMalloc(&init_photon_list, sizeof(t_photon) * photon_count)))
	{
		printf("shoot_photon group malloc fail: %s, %d\n", hipGetErrorString(code), code);
		exit(1);
		 
		//if hipMalloc fails...
		// return (shoot_photon_group(r, photon_count / 2) +
		// shoot_photon_group(r, photon_count / 2));
	}
	init_photon_group(r, photon_count, init_photon_list);
	shoot_photon_wrapper(r, photon_count, init_photon_list);
	// hipFree(init_photon_list); // CAUSES INVALID DEVICE POINTER
	return (photon_count);
}

static void		init_photon_group(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list)
{
	// printf("init_photon_group\n");
	t_light		*l_ptr;
	float		total_intensity;
	float 		ratio;
	int			i;
	
	total_intensity = get_total_intensity(r->scene->lights);
	l_ptr = r->scene->lights;
	ratio = 0;
	i = -1;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
			continue ;
		//get ratio of photons to shoot for one light
		ratio += photon_count * l_ptr->intensity / total_intensity;
		//while photon count within ratio
		while (++i < ratio)
			//init the photon
			init_photon((init_photon_list + i), l_ptr);
		l_ptr = l_ptr->next;
	}
}		

static float	get_total_intensity(t_light *lights)
{
	// printf("get_total_intensity\n");
	t_light		*l_ptr;
	float		total_intensity;
	
	l_ptr = lights;
	total_intensity = 0;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
			continue ;
		total_intensity += l_ptr->intensity;
		l_ptr = l_ptr->next;
	}
	return (total_intensity);
}

static void		init_photon(t_photon *photon, t_light *light)
{
	// printf("init_photon\n");
	photon->pos = light->pos;
	photon->col = vec_to_col(light->col);
	photon->n = v_new(NAN, NAN, NAN);
}

/*
** Kernel wrapper and kernel for shooting photon
*/

static void		shoot_photon_wrapper(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list)
{
	printf("shoot_photon_wrapper\n");
	dim3 		blockSize;
	dim3 		gridSize;
	
	blockSize = dim3(BLOCK_DIM, 1, 1);
	gridSize = dim3(photon_count / BLOCK_DIM + ((photon_count % BLOCK_DIM) ? 1 : 0, 1));
	shoot_photon<<<gridSize, blockSize>>>(r->d_scene, init_photon_list, photon_count, rand());
	C(1)
		hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();
if (errSync != hipSuccess) 
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	// gpu_errchk((hipDeviceSynchronize()));
}

__global__
static void			shoot_photon(t_scene *scene, t_photon *init_photon_list, int photon_count, float rand_numbers[])
{
	t_raytracing_tools	r;
	t_ray				photon;
	hiprandState			state;
	
	// printf("shoot_photon: time: %d\n", rand_i);
	r.pix.x = 0;
	r.pix.y = 0;
	r.scene = scene;
    r.idx = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (r.idx >= photon_count)  
		return ; 
	r.devStates = &state;
	// hiprand_init (r.idx + (rand_i % 50), 0, 0, r.devStates);
	memset(&r.ior_list, 0, sizeof(float) * (PHOTON_BOUNCE_MAX + 1));
	photon = init_kernel_photon(&r, init_photon_list[r.idx], rand_numbers);
	// printf("photon [%i]: [%f, %f, %f]\n", r.idx, photon.dir.x, photon.dir.y, photon.dir.z);
	if (r.idx == 0)
		printf("photon cast primary ray\n");
	cast_primary_ray(&r, &photon);
	
	__syncthreads();
	
	//also need to update final photon position and normal at hitpoint
}
 
__device__ 
static t_ray		init_kernel_photon(t_raytracing_tools *r, t_photon photon, float rand_numbers[])
{
	t_ray		new_ray; 
	hiprandState localState;
	int 		i;
	
	//simplify rand nums here
	// printf("init_kernel_photon\n");
	localState = *r->devStates;
	new_ray.dir.x = hiprand(&localState);
	// r->devStates = &localState;
	new_ray.dir.y = hiprand(&localState);
	// r->devStates = &localState;
	new_ray.dir.z = hiprand(&localState);
	r->devStates = &localState;
	new_ray.dir = v_norm(new_ray.dir);
	// printf("init kernel photon: dir: [%f, %f, %f]\n", new_ray.dir.x, new_ray.dir.y, new_ray.dir.z);
	new_ray.type = R_DIRECT_PHOTON;
	new_ray.origin = photon.pos;
	new_ray.col = photon.col;
	new_ray.nhit = photon.n;
	new_ray.n_dir = 1;
	new_ray.depth = r->scene->ray_depth;
	new_ray.ior = r->scene->cameras->ior; ////////////
	r->ior_list[0] = r->scene->cameras->ior; ////////change for light ior!
	i = 0;
	while (i < r->scene->ray_depth && i < PHOTON_BOUNCE_MAX)
	{
		r->scene->photon_list[r->idx + i].pos = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx + i].dir = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx + i].n = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx + i].col = c_new(0, 0, 0);
		++i;
	}
	if (r->idx == 0)
		printf("done preping photon\n");
	return (new_ray);
}

void	print_photons(t_kd_tree *tree)
{
	// printf("PRINTF_PHOTONS\n");
	if (!tree)
	{
		printf("found nothing going up\n");
		return ;
	}
	printf("going left\n");
	print_photons(tree->left);
	printf("photon: [%p] [%f, %f, %f]\n", tree, tree->pos.x, tree->pos.y, tree->pos.z);
	printf("going right\n");
	print_photons(tree->right);
	printf("going up\n");
}