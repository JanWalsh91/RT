#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   malloc_region_map.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/05 15:14:03 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/08 12:25:10 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "hip/hip_runtime.h"
#include "../inc/cuda_call.h"

static void init_region_map(t_region *region_map, size_t size, float search_rad);

/*
** Allocated or reallocates memory for the CPU region map used in Progressive Photon Mapping. 
*/

//TODO ADD MALLOC PROTECTIONZ

void	malloc_region_map(t_raytracing_tools *r, t_tile tile)
{
	size_t	size1;
	size_t	size2;
	int		i;
	static int	old_tile_size = 0;

	if (r->scene->is_photon_mapping)
	{

		if (r->h_region_map)
		{
			i = -1;
			while (++i < old_tile_size)
				free(r->h_region_map[i]);
			free(r->h_region_map);
			old_tile_size = tile.max;
		}
		size1 = sizeof(t_region *) * tile.col * tile.row;
		size2 = sizeof(t_region) * tile.size * tile.size;
		if (!(r->h_region_map = (t_region **)malloc(size1)))
			exit(0);
		i = -1;
		while (++i < tile.max)
		{
			if (!(r->h_region_map[i] = (t_region *)malloc(size2)))
				exit(0);
			init_region_map(r->h_region_map[i], tile.size * tile.size, r->settings.photon_search_radius);
		}
	}
}

static void init_region_map(t_region *region_map, size_t size, float search_rad)
{
	int i;
	
	i = -1;
	while (++i < size)
	{
		region_map[i].hit_pt = v_new(NAN, NAN, NAN);
		region_map[i].ray_dir = v_new(NAN, NAN, NAN);
		region_map[i].normal = v_new(NAN, NAN, NAN);
		region_map[i].radius = search_rad;
		region_map[i].n = 0;
		region_map[i].power = v_new(0, 0, 0);
		region_map[i].kd = NAN;
	}
}

/*
** Allocated or reallocates memory for the GPU region map used in Progressive Photon Mapping. 
*/

void	cuda_malloc_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	size_t	size;

	if (r->scene->is_photon_mapping)
	{
		if (r->d_region_map)
			hipFree(r->d_region_map);
		size = sizeof(t_region) * tile.size * tile.size;
		gpu_errchk(hipMalloc(&(r->d_region_map), size));
	}
}
