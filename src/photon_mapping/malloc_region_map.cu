#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   malloc_region_map.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/05 15:14:03 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 15:14:20 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "hip/hip_runtime.h"
#include "../inc/cuda_call.h"

static void init_region_map(t_region *region_map, size_t size, float search_rad);

/*
** Allocated or reallocates memory for the CPU region map used in Progressive Photon Mapping. 
*/

//TODO ADD MALLOC PROTECTIONZ

void	malloc_region_map(t_raytracing_tools *r, t_tile tile)
{
	printf("malloc_region_map\n");
	size_t	size1;
	size_t	size2;
	int		i;

	r->update.photon_map = 2; ///////////
	if (r->scene->is_photon_mapping && r->update.photon_map == 2)
	{
		if (r->h_region_map)
			free(r->h_region_map);
		size1 = sizeof(t_region *) * tile.col * tile.row;
		size2 = sizeof(t_region) * tile.size * tile.size;
		r->h_region_map = (t_region **)malloc(size1);
		i = -1;
		while (++i < tile.max)
		{
			r->h_region_map[i] = (t_region *)malloc(size2);
			init_region_map(r->h_region_map[i], tile.size * tile.size, r->settings.photon_search_radius);
		}
		r->update.photon_map = 0;
	}
}

static void init_region_map(t_region *region_map, size_t size, float search_rad)
{
	int i;
	
	i = -1;
	while (++i < size)
	{
		region_map[i].hit_pt = v_new(NAN, NAN, NAN);
		region_map[i].ray_dir = v_new(NAN, NAN, NAN);
		region_map[i].normal = v_new(NAN, NAN, NAN);
		region_map[i].radius = search_rad;
		region_map[i].n = 0;
		region_map[i].power = v_new(0, 0, 0);
		region_map[i].kd = NAN;
	}
}

/*
** Allocated or reallocates memory for the GPU region map used in Progressive Photon Mapping. 
*/

void	cuda_malloc_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	// printf("cuda_malloc_region_map\n");
	size_t	size;

	if (r->scene->is_photon_mapping)
	{
		if (r->d_region_map)
			hipFree(r->d_region_map);
		size = sizeof(t_region) * tile.size * tile.size;
		gpu_errchk(hipMalloc(&(r->d_region_map), size));
	}
}