/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   photon_mapping.cu                                  :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 11:56:52 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/30 10:59:14 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "photon_mapping.h"
#include "hip/hip_runtime.h"
#include "../inc/cuda_call.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// TO DO add interaction with user to be able to stop

void	render_ppm(t_raytracing_tools *r)
{
	printf("render_ppm\n");
	int	photons_shot;
	
	photons_shot = 0;
	r->settings.photon_count_per_pass = 10;

	//malloc space for photons on GPU
	gpuErrchk(hipMalloc(&(r->h_d_scene->photon_list), sizeof(t_photon) * PHOTON_BOUNCE_MAX * r->settings.photon_count_per_pass));
	gpuErrchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	while (photons_shot < r->scene->photon_count)
	{
		//photon_mapping_pass
		photon_mapping_pass(r);
		//radiance estimation pass
		radiance_estimation_pass(r);
		//reset photon map // necessary? will be set to 0 inside the kernel anyway...
		photons_shot += r->settings.photon_count_per_pass;
	}
	hipFree(r->h_d_scene->photon_list);
}

// static void	reset_photon_map(t_raytracing_tools *r)
// {
// 	hipFree(r->h_d_scene->photon_list);
// 	gpuErrchk(hipMalloc(&(r->h_d_scene->photon_list), sizeof(t_photon) * PHOTON_BOUNCE_MAX * r->settings.photon_count_per_pass));
// 	gpuErrchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));	
// }