/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   photon_mapping_assist.cu                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/08 12:55:39 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/08 13:54:39 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "cuda_call.h"
#include <hip/hip_runtime.h>

/*
** Functions assiting PPM.
*/

void	perpare_memory(t_raytracing_tools *r)
{
	r->h_d_scene->photon_iteration = 1;
	(hipHostMalloc((void **)&r->rt_pixel_map, sizeof(t_color) *
		r->scene->res.y * r->scene->res.x));
	hipMemcpy(r->rt_pixel_map, r->d_pixel_map, sizeof(t_color) *
		r->scene->res.y * r->scene->res.x, hipMemcpyHostToHost);
	hipMalloc((void **)&(r->h_d_scene->photon_list), sizeof(t_photon) *
		PHOTON_BOUNCE_MAX * r->scene->photon_count_per_pass);
	hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene),
		hipMemcpyHostToDevice);
}

void	copy_1(t_raytracing_tools *r)
{
	hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene),
		hipMemcpyHostToDevice);
}

void	copy_2(t_raytracing_tools *r)
{
	hipMemcpy(r->d_pixel_map, r->rt_pixel_map, sizeof(t_color) *
		r->scene->res.x * r->scene->res.y, hipMemcpyHostToHost);
}

void	free_map_and_list(t_raytracing_tools *r)
{
	hipHostFree(r->rt_pixel_map);
	hipFree(r->h_d_scene->photon_list);
}
