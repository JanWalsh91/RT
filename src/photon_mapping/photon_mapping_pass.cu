/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   photon_mapping_pass.cu                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 12:16:47 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/07 11:35:08 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

static void		init_photon_group(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list);
static float	get_total_intensity(t_light *lights);

void	photon_mapping_pass(t_raytracing_tools *r)
{
	printf("photon_mapping_pass\n");
	t_photon		*init_photon_list;
	int 			photon_count;

	photon_count = r->scene->photon_count_per_pass;
	gpu_errchk(hipHostMalloc(&init_photon_list, sizeof(t_photon) * photon_count));
	init_photon_group(r, photon_count, init_photon_list);
	shoot_photon_wrapper(r, photon_count, init_photon_list);
	hipHostFree(init_photon_list);
}

static void		init_photon_group(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list)
{
	t_light		*l_ptr;
	float		total_intensity;
	float 		ratio;
	int			i;
	
	total_intensity = get_total_intensity(r->scene->lights);
	l_ptr = r->scene->lights;
	ratio = 0;
	i = -1;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
		{
			l_ptr = l_ptr->next;
			continue ;
		}
		ratio += photon_count * l_ptr->intensity / total_intensity;
		while (++i < ratio)
		{
			init_photon_list[i].pos = l_ptr->pos;
			init_photon_list[i].col = vec_to_col(l_ptr->col);
			init_photon_list[i].n = v_new(NAN, NAN, NAN);
		}
		l_ptr = l_ptr->next;
	}
}		

static float	get_total_intensity(t_light *lights)
{
	t_light		*l_ptr;
	float		total_intensity;
	
	l_ptr = lights;
	total_intensity = 0;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
		{
			l_ptr = l_ptr->next;
			continue ;
		}
		total_intensity += l_ptr->intensity;
		l_ptr = l_ptr->next;
	}
	return (total_intensity);
}

