#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   photon_mapping_pass.cu                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 12:16:47 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/01 16:55:49 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "photon_mapping.h"
#include "rt.cuh"
#include "../../inc/cuda_call.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
void			print_photons(t_kd_tree *tree);
static int		shoot_photon_group(t_raytracing_tools *r, size_t photon_count);
static void		init_photon_group(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list);
static float	get_total_intensity(t_light *lights);
static void		init_photon(t_photon *photon, t_light *light);
static void		shoot_photon_wrapper(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list);
__global__
static void		shoot_photon(t_scene *scene, t_photon *init_photon_list, int photon_count, float *rand_numbers);
__device__
static t_ray	init_kernel_photon(t_raytracing_tools *r, t_photon photon, float *rand_numbers);

void	photon_mapping_pass(t_raytracing_tools *r)
{
	printf("photon_mapping_pass\n");
	
	
	shoot_photon_group(r, r->scene->photon_count_per_pass);
}

static int	shoot_photon_group(t_raytracing_tools *r, size_t photon_count)
{
	// printf("shoot_photon_group\n");
	t_photon	*init_photon_list;
	hipError_t		code;

	if ((code = hipHostMalloc(&init_photon_list, sizeof(t_photon) * photon_count)))
	{
		printf("shoot_photon group malloc fail: %s, %d\n", hipGetErrorString(code), code);
		exit(1);
		 
		//if hipMalloc fails...
		// return (shoot_photon_group(r, photon_count / 2) +
		// shoot_photon_group(r, photon_count / 2));
	}
	init_photon_group(r, photon_count, init_photon_list);
	shoot_photon_wrapper(r, photon_count, init_photon_list);
	// hipHostFree(init_photon_list); // CAUSES INVALID DEVICE POINTER: try with hipHostFree
	return (photon_count);
}

static void		init_photon_group(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list)
{
	// printf("init_photon_group\n");
	t_light		*l_ptr;
	float		total_intensity;
	float 		ratio;
	int			i;
	
	total_intensity = get_total_intensity(r->scene->lights);
	l_ptr = r->scene->lights;
	ratio = 0;
	i = -1;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
			continue ;
		//get ratio of photons to shoot for one light
		ratio += photon_count * l_ptr->intensity / total_intensity;
		//while photon count within ratio
		while (++i < ratio)
			//init the photon
			init_photon((init_photon_list + i), l_ptr);
		l_ptr = l_ptr->next;
	}
}		

static float	get_total_intensity(t_light *lights)
{
	// printf("get_total_intensity\n");
	t_light		*l_ptr;
	float		total_intensity;
	
	l_ptr = lights;
	total_intensity = 0;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
			continue ;
		total_intensity += l_ptr->intensity;
		l_ptr = l_ptr->next;
	}
	return (total_intensity);
}

static void		init_photon(t_photon *photon, t_light *light)
{
	// printf("init_photon\n");
	photon->pos = light->pos;
	photon->col = vec_to_col(light->col);
	photon->n = v_new(NAN, NAN, NAN);
}


void			init_random_numbers(int nb, float *random_numbers)
{
	int i;

	i = -1;
	while (++i < nb)
	{
		random_numbers[i] = (rand() % 100) - 50;
		printf("%f\n", random_numbers[i]);
	}
}

/*
** Kernel wrapper and kernel for shooting photon
*/

static void		shoot_photon_wrapper(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list)
{
	printf("shoot_photon_wrapper\n");
	dim3 		blockSize;
	dim3 		gridSize;
	float		*h_rand_numbers;
	float		*d_rand_numbers;
	
	// srand(time(NULL));
	blockSize = dim3(BLOCK_DIM, 1, 1);
	gridSize = dim3(photon_count / BLOCK_DIM + ((photon_count % BLOCK_DIM) ? 1 : 0, 1));
	
	h_rand_numbers = (float *)malloc(sizeof(float) * r->scene->photon_count_per_pass * 3);
	hipMalloc(&d_rand_numbers, sizeof(float) * r->scene->photon_count_per_pass * 3);
	init_random_numbers(r->scene->photon_count_per_pass * 3, h_rand_numbers);
	hipMemcpy(d_rand_numbers, h_rand_numbers, sizeof(float) * r->scene->photon_count_per_pass * 3, hipMemcpyHostToDevice);
	shoot_photon<<<gridSize, blockSize>>>(r->d_scene, init_photon_list, photon_count, d_rand_numbers);
	hipFree(d_rand_numbers);
	free(h_rand_numbers);
	// printf("-------p: %f\n", p);
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) 
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	// gpuErrchk((hipDeviceSynchronize()));
}

__global__
static void			shoot_photon(t_scene *scene, t_photon *init_photon_list, int photon_count, float *rand_numbers)
{
	t_raytracing_tools	r;
	t_ray				photon;
	hiprandState			state;
	
	// printf("shoot_photon: time: %d\n", rand_i);
	r.pix.x = 0;
	r.pix.y = 0;
	r.scene = scene;
    r.idx = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (r.idx >= photon_count)
		return ;
	r.devStates = &state;
	// hiprand_init (r.idx + (rand_i % 50), 0, 0, r.devStates);
	hiprand_init (rand_numbers[0], r.idx % (int)rand_numbers[0], r.idx * rand_numbers[0], r.devStates);
	memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
	photon = init_kernel_photon(&r, init_photon_list[r.idx], rand_numbers);
	// if (r.idx == 0)
	// 	printf("photon [%i]: [%f, %f, %f]\n", r.idx, photon.dir.x, photon.dir.y, photon.dir.z);
	if (r.idx == 0)
		printf("photon cast primary ray\n");
	cast_primary_ray(&r, &photon);
}
 
__device__ 
static t_ray		init_kernel_photon(t_raytracing_tools *r, t_photon photon, float *rand_numbers)
{
	t_ray		new_ray; 
	// hiprandState localState;
	int 		i;
	
	//simplify rand nums here
	// printf("init_kernel_photon\n");
	// localState = *r->devStates;
	// new_ray.dir.x = hiprand_uniform(&localState);
	// (int)new_ray.dir.x % 2 ? new_ray.dir.x *= -1 : 0;
	// r->devStates = &localState;
	// new_ray.dir.y = hiprand_uniform(&localState);
	// (int)new_ray.dir.y % 2 ? new_ray.dir.y *= -1 : 0;
	// r->devStates = &localState;
	// new_ray.dir.z = hiprand_uniform(&localState);
	// (int)new_ray.dir.z % 2 ? new_ray.dir.z *= -1 : 0;
	// r->devStates = &localState;

	if (r->idx == 0)
		printf("99999999999Avant\n");
	new_ray.dir.x = rand_numbers[r->idx];
	new_ray.dir.y = rand_numbers[r->idx + 1];
	new_ray.dir.z = rand_numbers[r->idx + 2];
	__syncthreads();
	new_ray.dir = v_norm(new_ray.dir);
	if (r->idx == 0)
		printf("Apres : %f, %f, %f\n", new_ray.dir.x, new_ray.dir.y, new_ray.dir.z);
	// printf("init kernel photon: dir: [%f, %f, %f]\n", new_ray.dir.x, new_ray.dir.y, new_ray.dir.z);
	new_ray.type = R_DIRECT_PHOTON;
	new_ray.origin = photon.pos;
	new_ray.col = photon.col;
	new_ray.nhit = photon.n;
	new_ray.n_dir = 1;
	new_ray.depth = r->scene->ray_depth;
	new_ray.ior = r->scene->cameras->ior; ////////////
	r->ior_list[0] = r->scene->cameras->ior; ////////change for light ior!
	i = 0;
	while (i < r->scene->ray_depth && i < PHOTON_BOUNCE_MAX)
	{
		r->scene->photon_list[r->idx * PHOTON_BOUNCE_MAX + i].pos = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx * PHOTON_BOUNCE_MAX + i].dir = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx * PHOTON_BOUNCE_MAX + i].n = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx * PHOTON_BOUNCE_MAX + i].col = c_new(0, 0, 0);
		++i;
	}
	if (r->idx == 0)
		printf("done preping photon\n");
	return (new_ray);
}

void	print_photons(t_kd_tree *tree)
{
	// printf("PRINTF_PHOTONS\n");
	if (!tree)
	{
		printf("found nothing going up\n");
		return ;
	}
	printf("going left\n");
	print_photons(tree->left);
	printf("photon: [%p] [%f, %f, %f]\n", tree, tree->pos.x, tree->pos.y, tree->pos.z);
	printf("going right\n");
	print_photons(tree->right);
	printf("going up\n");
}