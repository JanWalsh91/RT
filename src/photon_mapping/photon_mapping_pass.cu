#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   photon_mapping_pass.cu                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 12:16:47 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/30 10:57:31 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "photon_mapping.h"
#include "rt.cuh"
#include "../../inc/cuda_call.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
void			print_photons(t_kd_tree *tree);
static int		shoot_photon_group(t_raytracing_tools *r, size_t photon_count);
static void		init_photon_group(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list);
static float	get_total_intensity(t_light *lights);
static void		init_photon(t_photon *photon, t_light *light);
static void		shoot_photon_wrapper(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list);
__global__
static void		shoot_photon(t_scene *scene, t_photon *init_photon_list, int photon_count, int rand_i);
__device__
static t_ray	init_kernel_photon(t_raytracing_tools *r, t_photon photon);

void	photon_mapping_pass(t_raytracing_tools *r)
{
	printf("photon_mapping_pass\n");
	
	srand(time(NULL));
	shoot_photon_group(r, r->scene->photon_count);
}

static int	shoot_photon_group(t_raytracing_tools *r, size_t photon_count)
{
	// printf("shoot_photon_group\n");
	t_photon	*init_photon_list;
	hipError_t		code;

	if ((code = hipHostMalloc(&init_photon_list, sizeof(t_photon) * photon_count)))
	{
		printf("shoot_photon group malloc fail: %s, %d\n", hipGetErrorString(code), code);
		exit(1);
		 
		//if hipMalloc fails...
		// return (shoot_photon_group(r, photon_count / 2) +
		// shoot_photon_group(r, photon_count / 2));
	}
	init_photon_group(r, photon_count, init_photon_list);
	shoot_photon_wrapper(r, photon_count, init_photon_list);
	// hipHostFree(init_photon_list); // CAUSES INVALID DEVICE POINTER: try with hipHostFree
	return (photon_count);
}

static void		init_photon_group(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list)
{
	// printf("init_photon_group\n");
	t_light		*l_ptr;
	float		total_intensity;
	float 		ratio;
	int			i;
	
	total_intensity = get_total_intensity(r->scene->lights);
	l_ptr = r->scene->lights;
	ratio = 0;
	i = -1;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
			continue ;
		//get ratio of photons to shoot for one light
		ratio += photon_count * l_ptr->intensity / total_intensity;
		//while photon count within ratio
		while (++i < ratio)
			//init the photon
			init_photon((init_photon_list + i), l_ptr);
		l_ptr = l_ptr->next;
	}
}		

static float	get_total_intensity(t_light *lights)
{
	// printf("get_total_intensity\n");
	t_light		*l_ptr;
	float		total_intensity;
	
	l_ptr = lights;
	total_intensity = 0;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
			continue ;
		total_intensity += l_ptr->intensity;
		l_ptr = l_ptr->next;
	}
	return (total_intensity);
}

static void		init_photon(t_photon *photon, t_light *light)
{
	// printf("init_photon\n");
	photon->pos = light->pos;
	photon->col = vec_to_col(light->col);
	photon->n = v_new(NAN, NAN, NAN);
}

/*
** Kernel wrapper and kernel for shooting photon
*/

static void		shoot_photon_wrapper(t_raytracing_tools *r, size_t photon_count, t_photon *init_photon_list)
{
	printf("shoot_photon_wrapper\n");
	dim3 		blockSize;
	dim3 		gridSize;
	
	blockSize = dim3(BLOCK_DIM, 1, 1);
	gridSize = dim3(photon_count / BLOCK_DIM + ((photon_count % BLOCK_DIM) ? 1 : 0, 1));
	shoot_photon<<<gridSize, blockSize>>>(r->d_scene, init_photon_list, photon_count, rand());
	C(1)
		hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();
if (errSync != hipSuccess) 
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	// gpuErrchk((hipDeviceSynchronize()));
}

__global__
static void			shoot_photon(t_scene *scene, t_photon *init_photon_list, int photon_count, int rand_i)
{
	t_raytracing_tools	r;
	t_ray				photon;
	hiprandState			state;
	
	// printf("shoot_photon: time: %d\n", rand_i);
	r.pix.x = 0;
	r.pix.y = 0;
	r.scene = scene;
    r.idx = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (r.idx >= photon_count)  
		return ; 
	r.devStates = &state;
	hiprand_init (r.idx + (rand_i % 50), 0, 0, r.devStates);
	memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
	photon = init_kernel_photon(&r, init_photon_list[r.idx]);
	// printf("photon [%i]: [%f, %f, %f]\n", r.idx, photon.dir.x, photon.dir.y, photon.dir.z);
	if (r.idx == 0)
		printf("photon cast primary ray\n");
	cast_primary_ray(&r, &photon);
	__syncthreads();
}
 
__device__ 
static t_ray		init_kernel_photon(t_raytracing_tools *r, t_photon photon)
{
	t_ray		new_ray; 
	hiprandState localState;
	int 		i;
	
	//simplify rand nums here
	// printf("init_kernel_photon\n");
	localState = *r->devStates;
	new_ray.dir.x = hiprand(&localState);
	// r->devStates = &localState;
	new_ray.dir.y = hiprand(&localState);
	// r->devStates = &localState;
	new_ray.dir.z = hiprand(&localState);
	r->devStates = &localState;
	new_ray.dir = v_norm(new_ray.dir);
	// printf("init kernel photon: dir: [%f, %f, %f]\n", new_ray.dir.x, new_ray.dir.y, new_ray.dir.z);
	new_ray.type = R_DIRECT_PHOTON;
	new_ray.origin = photon.pos;
	new_ray.col = photon.col;
	new_ray.nhit = photon.n;
	new_ray.n_dir = 1;
	new_ray.depth = r->scene->ray_depth;
	new_ray.ior = r->scene->cameras->ior; ////////////
	r->ior_list[0] = r->scene->cameras->ior; ////////change for light ior!
	i = 0;
	while (i < r->scene->ray_depth && i < PHOTON_BOUNCE_MAX)
	{
		r->scene->photon_list[r->idx + i].pos = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx + i].dir = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx + i].n = v_new(NAN, NAN, NAN);
		r->scene->photon_list[r->idx + i].col = c_new(0, 0, 0);
		++i;
	}
	if (r->idx == 0)
		printf("done preping photon\n");
	return (new_ray);
}

void	print_photons(t_kd_tree *tree)
{
	// printf("PRINTF_PHOTONS\n");
	if (!tree)
	{
		printf("found nothing going up\n");
		return ;
	}
	printf("going left\n");
	print_photons(tree->left);
	printf("photon: [%p] [%f, %f, %f]\n", tree, tree->pos.x, tree->pos.y, tree->pos.z);
	printf("going right\n");
	print_photons(tree->right);
	printf("going up\n");
}