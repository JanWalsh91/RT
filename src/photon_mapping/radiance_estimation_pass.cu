#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   radiance_estimation_pass.cu                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 12:16:58 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/30 10:56:06 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "photon_mapping.h"
#include "rt.cuh"
#include "../../inc/cuda_call.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void	radiance_estimation_pass(t_raytracing_tools *r)
{
	printf("radiance_estimation_pass\n");
	// dim3 		blockSize;
	// dim3 		gridSize;

	// //prepare kernel
	// blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	// gridSize = dim3(r->scene->res.x / BLOCK_DIM, r->scene->res.y / BLOCK_DIM);
	// estimate_region_radiance<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map, r->d_region_map);
}

// __global__
// void	estimate_region_radiance(t_scene *scene, t_color *pixel_map, t_region *region_map)
// {
// 	t_raytracing_tools	r;

// 	r.pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
// 	r.pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
// 	r.scene = scene;
//     r.idx = scene->res.x * r.pix.y + r.pix.x;
// }