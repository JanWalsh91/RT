#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   radiance_estimation_pass.cu                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 12:16:58 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/02 12:14:59 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "photon_mapping.h"
#include "rt.cuh"
#include "../../inc/cuda_call.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__device__
static void		update_photon_count(int *photon_count, float photons_added);
__device__
static void		update_radius(t_raytracing_tools *r, float *radius, float photon_count, float photons_added);
__device__
static t_vec3	add_accumulated_power(t_region *region, t_photon photon, float rad);
__device__
static void		udpate_accumulated_power(t_vec3 *pow, t_vec3 new_pow, float rad2);

__global__
void	estimate_region_radiance(t_scene *scene, t_color *pixel_map, t_region *region_map, t_tile tile)
{
	t_raytracing_tools	r;
	int					i;
	int					i2;
	int					photons_added;
	float				dist;
	t_vec3				power_added;
	
	r.pix.x = (tile.id.x * tile.size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (tile.id.y * tile.size) + (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene; 
    r.idx = scene->res.x * r.pix.y + r.pix.x;
	i2 = (r.pix.x % tile.size) + (r.pix.y % tile.size) * tile.size;
	photons_added = 0;
	power_added = v_new(0, 0, 0);
	if (r.pix.x >= scene->res.x || r.pix.y >= scene->res.y || v_isnan(region_map[i2].hit_pt))
		return ;
	// printf("iteration: %d\n", scene->photon_iteration);
	if (r.idx == 0 && scene->photon_iteration == 1)
	{
		t_vec3 normalized_power = v_scale(region_map[0].power, 1.0 / (float)(scene->photon_iteration * scene->photon_count_per_pass * M_PI * region_map[0].radius * region_map[0].radius));

		printf("\nregion intialization: iteration: [%d]\
			\n\tphotons added: [%d]\
			\n\thit_pt: [%f, %f, %f]\
			\n\tray_dir: [%f, %f, %f]\
			\n\tnormal: [%f, %f, %f]\
			\n\tkd: [%f]\
			\n\tradius: [%f]\
			\n\tphoton count: [%d]\
			\n\taccumulated power: [%f, %f, %f]\n\n",
			scene->photon_iteration,
			photons_added,
			region_map[0].hit_pt.x, region_map[0].hit_pt.y, region_map[0].hit_pt.z,
			region_map[0].ray_dir.x, region_map[0].ray_dir.y, region_map[0].ray_dir.z,
			region_map[0].normal.x, region_map[0].normal.y, region_map[0].normal.z,
			region_map[0].kd,
			region_map[0].radius,
			region_map[0].n,
			region_map[0].power.x, region_map[0].power.y, region_map[0].power.z
		);
	}
		// printf("---rad entering kernel: %f\n", region_map[0].radius);
	// printf("idx: [%d] i2: [%d]\n", r.idx, i2);
	i = -1;
	// printf("pcpp: %d, rad: %f\n", r.scene->photon_count_per_pass, region_map[i2].radius);
	while (++i < PHOTON_BOUNCE_MAX * r.scene->photon_count_per_pass)
	{
		// printf("POSITION DU PHOTON: %f, %f, %f\n", scene->photon_list[i].pos.x, scene->photon_list[i].pos.y, scene->photon_list[i].pos.z);
		if (!v_isnan(scene->photon_list[i].pos))
		{
			dist = v_length(v_sub(region_map[i2].hit_pt, scene->photon_list[i].pos));
			// printf("%d: dist: %f rad: %f\n", r.idx, dist, region_map[i2].radius);
			if (region_map[i2].radius > dist)
			{
				// if (r.pix.x == 100 && r.pix.y == 100)
				// {
				// 	printf("r.idx: %d, hit pt: [%f, %f, %f], photon: [%f, %f, %f], rad: %f dist: %f\n", r.idx, region_map[i2].hit_pt.x, region_map[i2].hit_pt.y, region_map[i2].hit_pt.z, scene->photon_list[i].pos.x, scene->photon_list[i].pos.y, scene->photon_list[i].pos.z, region_map[i2].radius, dist);
				// }
				
				power_added = v_add(power_added, add_accumulated_power(&region_map[i2], scene->photon_list[i], dist * dist));
				// printf("power: [%f, %f, %f]\n", power_added.x, power_added.y, power_added.z);
				++photons_added;
			}
		}
		__syncthreads();
	}
	if (photons_added)
		udpate_accumulated_power(&region_map[i2].power, power_added, region_map[i2].radius * region_map[i2].radius);
	// if (r.idx == 0)
	// {
	// 	// float r = region_map[0].radius;
	// 	printf("before update radius: rad: %f, photon_count: %d, photons_added: %d, %p\n", region_map[0].radius, region_map[0].n, photons_added, &region_map[0].radius);
	// }
	if (photons_added)
		update_radius(&r, &region_map[i2].radius, region_map[i2].n, photons_added);
	// if (r.idx == 0)
	// {
	// 	// printf("--- photons_added: %d, rad in kernel: %f --> %f, %p\n", photons_added, r, region_map[0].radius, &region_map[0].radius);
	// 	printf("before update radius: rad: %f, photon_count: %d, photons_added: %d, %p\n", region_map[0].radius, region_map[0].n, photons_added, &region_map[0].radius);
	// }
	update_photon_count(&region_map[i2].n, photons_added);
	//put in r->rt_pixel_map[index]
	t_color tmp = pixel_map[r.idx];
	// pixel_map[r.idx] = c_new(0, 110, 110);
	// printf("photon_iter: %lu, count per pass: %lu\n", scene->photon_iteration, scene->photon_count_per_pass);
	pixel_map[r.idx] = c_add(pixel_map[r.idx], vec_to_col(v_scale(region_map[i2].power, 1.0 / (float)(scene->photon_iteration * scene->photon_count_per_pass * M_PI * region_map[i2].radius * region_map[i2].radius))));
	// if (r.idx % 500 == 0)
	// printf("power: [%f, %f, %f]\n", region_map[i2].power.x, region_map[i2].power.y, region_map[i2].power.z);
	// printf("idx: [%d]. old color: [%d, %d, %d], new color: [%d, %d, %d]\n", r.idx, tmp.r, tmp.g, tmp.b, pixel_map[r.idx].r, pixel_map[r.idx].g, pixel_map[r.idx].b);
	if (r.idx == 0)
	{
		t_vec3 normalized_power = v_scale(region_map[0].power, 1.0 / (float)(scene->photon_iteration * scene->photon_count_per_pass * M_PI * region_map[0].radius * region_map[0].radius));

		printf("\nupdated region: iteration: [%d]\
			\n\tphoton count per pass: [%d]\
			\n\tphotons added: [%d]\
			\n\thit_pt: [%f, %f, %f]\
			\n\tray_dir: [%f, %f, %f]\
			\n\tnormal: [%f, %f, %f]\
			\n\tkd: [%f]\
			\n\tradius: [%f]\
			\n\tphoton count: [%d]\
			\n\taccumulated power: [%f, %f, %f]\
			\n\tnormalized_power: [%f, %f, %f]\n\n",
			scene->photon_iteration,
			scene->photon_count_per_pass,
			photons_added,
			region_map[0].hit_pt.x, region_map[0].hit_pt.y, region_map[0].hit_pt.z,
			region_map[0].ray_dir.x, region_map[0].ray_dir.y, region_map[0].ray_dir.z,
			region_map[0].normal.x, region_map[0].normal.y, region_map[0].normal.z,
			region_map[0].kd,
			region_map[0].radius,
			region_map[0].n,
			region_map[0].power.x, region_map[0].power.y, region_map[0].power.z,
			normalized_power.x, normalized_power.y, normalized_power.z
		);
	}
	__syncthreads();
}

__device__
static t_vec3	add_accumulated_power(t_region *region, t_photon photon, float rad2)
{
	// printf("update_accumulated_power\n");
	t_vec3 result;
	float k = 1; ///

	result = v_scale(col_to_vec(photon.col), 1 / (M_PI * rad2));
	result = v_scale(result, -v_dot(region->ray_dir, photon.n) * k * region->kd);
	// printf("add accumulated power: [%f, %f, %f]\n", result.x, result.y, result.z);
	return (result);
}

__device__
static void		udpate_accumulated_power(t_vec3 *pow, t_vec3 new_pow, float rad2)
{
	// printf("update_accumulated_power\n");
	// new_pow = v_scale(*pow, 1 / (M_PI * rad2));
	*pow = v_scale(v_add(*pow, new_pow), ((rad2 + 1) / (rad2)));
}

__device__
static void		update_radius(t_raytracing_tools *r, float *radius, float photon_count, float photons_added)
{
	// printf("update_radius\n");
	*radius *= sqrtf((photon_count + (2.0 / 3.0) * photons_added) / (photon_count + photons_added));
	// if (r->idx == 0)
	// 	printf("after : %f\n", *rad);
}

__device__
static void		update_photon_count(int *photon_count, float photons_added)
{
	// printf("update_photon_count\n");
	*photon_count += (2.0 / 3.0) * photons_added;
}

void			radiance_estimation_pass(t_raytracing_tools *r, t_tile tile)
{
	// printf("radiance_estimation_pass\n");
	dim3 		blockSize;
	dim3 		gridSize;
	int			size;

	// printf("a-----photon_iter: %d, count per pass: %d\n", r->scene->photon_iteration, r->scene->photon_count_per_pass);
	size = (tile.size / BLOCK_DIM) + ((tile.size % BLOCK_DIM) ? 1 : 0);
	blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	gridSize = dim3(size, size);
	estimate_region_radiance<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map, r->d_region_map, tile);
			hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();
if (errSync != hipSuccess) 
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	//add cuda debug
}
