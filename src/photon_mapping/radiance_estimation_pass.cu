#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   radiance_estimation_pass.cu                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 12:16:58 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/07 14:46:08 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__
static void		estimate_region_radiance(t_scene *scene, t_color *pixel_map,
				t_region *region_map, t_tile tile);
__device__
static void		update_region_values(t_raytracing_tools *r, int photons_added,
				t_vec3 power_added);
__device__
static t_vec3	add_accumulated_power(t_region *region, t_photon photon,
				float rad);

void			radiance_estimation_pass(t_raytracing_tools *r, t_tile tile)
{
	// printf("radiance_estimation_pass\n");
	dim3 		blockSize;
	dim3 		gridSize;
	int			size;

	size = (tile.size / BLOCK_DIM) + ((tile.size % BLOCK_DIM) ? 1 : 0);
	blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	gridSize = dim3(size, size);
	estimate_region_radiance<<<gridSize, blockSize>>>(r->d_scene,
		r->d_pixel_map, r->d_region_map, tile);
	cuda_check_kernel_errors();
}

__global__
static void		estimate_region_radiance(t_scene *scene, t_color *pixel_map,
				t_region *region_map, t_tile tile)
{
	t_raytracing_tools	r;
	int					i;
	int					photons_added;
	float				dist;
	t_vec3				power_added;
	
	r.pix.x = (tile.id.x * tile.size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (tile.id.y * tile.size) + (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene; 
    r.idx = scene->res.x * r.pix.y + r.pix.x;
	r.d_region_map = &region_map[(r.pix.x % tile.size) + (r.pix.y % tile.size) * tile.size];
	photons_added = 0;
	power_added = v_new(0, 0, 0);
	if (r.pix.x >= scene->res.x || r.pix.y >= scene->res.y || v_isnan(r.d_region_map->hit_pt))
		return ;
	i = -1;
	while (++i < PHOTON_BOUNCE_MAX * r.scene->photon_count_per_pass)
	{
		if (!v_isnan(scene->photon_list[i].pos))
		{
			dist = v_length(v_sub(r.d_region_map->hit_pt, scene->photon_list[i].pos));
			if (r.d_region_map->radius > dist && v_dot(r.d_region_map->normal, scene->photon_list[i].n) > 0.5)
			{
				power_added = v_add(power_added, add_accumulated_power(r.d_region_map, scene->photon_list[i], dist * dist));
				++photons_added;
			}
		}
	}
	update_region_values(&r, photons_added, power_added);
	pixel_map[r.idx] = c_add(pixel_map[r.idx], vec_to_col(v_scale(r.d_region_map->power, 1.0 /
	(float)(scene->photon_iteration * scene->photon_count_per_pass * M_PI * r.d_region_map->radius * r.d_region_map->radius))));
}

__device__
static void		update_region_values(t_raytracing_tools *r, int photons_added, t_vec3 power_added)
{
	float	old_rad2;
	float	alpha;

	alpha = 2.0 / 3.0;
	if (photons_added)
	{
		old_rad2 = r->d_region_map->radius * r->d_region_map->radius;
		r->d_region_map->radius *= sqrtf((r->d_region_map->n + alpha * photons_added) / (r->d_region_map->n + photons_added));
		r->d_region_map->power = v_scale(v_add(r->d_region_map->power, power_added), r->d_region_map->radius * r->d_region_map->radius / old_rad2);
	}
	r->d_region_map->n += alpha * photons_added;
}

__device__
static t_vec3	add_accumulated_power(t_region *region, t_photon photon, float dist2)
{
	// printf("update_accumulated_power\n");
	t_vec3 result;
	float k = 1500; ///

	result = v_scale(col_to_vec(photon.col), sqrtf(region->radius * region->radius - dist2) / (M_PI));
	result = v_scale(result, -v_dot(photon.dir, region->normal) * k * region->kd);
	// printf("add accumulated power: [%f, %f, %f]\n", result.x, result.y, result.z);
	return (result);
}




	// if (r.idx == 0 && scene->photon_iteration == 1)
	// {
	// 	t_vec3 normalized_power = v_scale(region_map[0].power, 1.0 / (float)(scene->photon_iteration * scene->photon_count_per_pass * M_PI * region_map[0].radius * region_map[0].radius));

	// 	printf("\nregion intialization: iteration: [%d]\
	// 		\n\tphotons added: [%d]\
	// 		\n\thit_pt: [%f, %f, %f]\
	// 		\n\tray_dir: [%f, %f, %f]\
	// 		\n\tnormal: [%f, %f, %f]\
	// 		\n\tkd: [%f]\
	// 		\n\tradius: [%f]\
	// 		\n\tphoton count: [%d]\
	// 		\n\taccumulated power: [%f, %f, %f]\n\n",
	// 		scene->photon_iteration,
	// 		photons_added,
	// 		region_map[0].hit_pt.x, region_map[0].hit_pt.y, region_map[0].hit_pt.z,
	// 		region_map[0].ray_dir.x, region_map[0].ray_dir.y, region_map[0].ray_dir.z,
	// 		region_map[0].normal.x, region_map[0].normal.y, region_map[0].normal.z,
	// 		region_map[0].kd,
	// 		region_map[0].radius,
	// 		region_map[0].n,
	// 		region_map[0].power.x, region_map[0].power.y, region_map[0].power.z
	// 	);
	// }














	// if (r.idx == 0)
	// {
	// 	t_vec3 normalized_power = v_scale(region_map[0].power, 1.0 / (float)(scene->photon_iteration * scene->photon_count_per_pass * M_PI * region_map[0].radius * region_map[0].radius));

	// 	printf("\nupdated region: iteration: [%d]\
	// 		\n\tphoton count per pass: [%d]\
	// 		\n\tphotons added: [%d]\
	// 		\n\thit_pt: [%f, %f, %f]\
	// 		\n\tray_dir: [%f, %f, %f]\
	// 		\n\tnormal: [%f, %f, %f]\
	// 		\n\tkd: [%f]\
	// 		\n\tradius: [%f]\
	// 		\n\tphoton count: [%d]\
	// 		\n\taccumulated power: [%f, %f, %f]\
	// 		\n\tnormalized_power: [%f, %f, %f]\n\n",
	// 		scene->photon_iteration,
	// 		scene->photon_count_per_pass,
	// 		photons_added,
	// 		region_map[0].hit_pt.x, region_map[0].hit_pt.y, region_map[0].hit_pt.z,
	// 		region_map[0].ray_dir.x, region_map[0].ray_dir.y, region_map[0].ray_dir.z,
	// 		region_map[0].normal.x, region_map[0].normal.y, region_map[0].normal.z,
	// 		region_map[0].kd,
	// 		region_map[0].radius,
	// 		region_map[0].n,
	// 		region_map[0].power.x, region_map[0].power.y, region_map[0].power.z,
	// 		normalized_power.x, normalized_power.y, normalized_power.z
	// 	);
	// }