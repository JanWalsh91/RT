/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   region_map.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/28 17:15:06 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/30 10:57:06 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "photon_mapping.h"
#include "hip/hip_runtime.h"
#include "../inc/cuda_call.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static void init_region_map(t_raytracing_tools *r, size_t size);

/*
** Allocated or reallocates memory for the CPU region map used in Progressive Photon Mapping. 
*/

void	malloc_region_map(t_raytracing_tools *r)
{
	printf("malloc_region_map\n");
	size_t size;

	r->update.photon_map = 2;
	if (r->scene->is_photon_mapping && r->update.photon_map == 2)
	{
		if (r->h_region_map)
			free(r->h_region_map);
		size = sizeof(t_region) * r->scene->res.x * r->scene->res.y;
		r->h_region_map = (t_region *)malloc(size);
		printf("%p\n", r->h_region_map);
		init_region_map(r, size);
		r->update.photon_map = 0;
	}
}

static void init_region_map(t_raytracing_tools *r, size_t size)
{
	int i;
	
	i = -1;
	while (++i < size)
	{
		r->h_region_map->radius = r->settings.photon_search_radius;
		r->h_region_map->n = 0;
		r->h_region_map->power = v_new(0, 0, 0);
		r->h_region_map->kd = NAN;
	}
}

/*
** Allocated or reallocates memory for the GPU region map used in Progressive Photon Mapping. 
*/

void	cuda_malloc_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	printf("cuda_malloc_region_map\n");
	size_t	size;

	if (r->scene->is_photon_mapping)
	{
		if (r->d_region_map)
			hipFree(r->d_region_map);
		size = sizeof(t_region) * tile.size * tile.size;
		gpuErrchk(hipMalloc(&(r->d_region_map), size));
	}
}

/*
** Sets tile region map variales to initial values
*/

void	refresh_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	t_region	empty;
	int			i;

	printf("refresh_region_map_tile\n");
	if (r->scene->is_photon_mapping)
	{
		empty.hit_pt = v_new(NAN, NAN, NAN);
		empty.ray_dir = v_new(NAN, NAN, NAN);
		empty.normal = v_new(NAN, NAN, NAN);
		empty.kd = NAN;
		i = -1;
		while (++i < tile.size * tile.size)
			gpuErrchk((hipMemcpy(&r->d_region_map[i], &empty, sizeof(t_region), hipMemcpyHostToDevice)));
	}
}

void	copy_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	//copy over row of a tile into global region map
	int i;
	int current_tile;
	if (r->scene->is_photon_mapping)
	{
		current_tile = tile.id.x * tile.size + tile.id.y * tile.size * tile.size;
		printf("%p, %p\n", &(r->h_region_map[current_tile + 0]), &(r->d_region_map[0]));
		printf("copy_region_map_tile: tileX: %d, tileY: %d, current_tile: %d\n", tile.id.x, tile.id.y, current_tile);
		i = -1;
		while (++i < tile.size)
			gpuErrchk((hipMemcpy(&(r->h_region_map[current_tile + i]), &(r->d_region_map[i]), sizeof(t_region), hipMemcpyDeviceToHost)));
	}
}