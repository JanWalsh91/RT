#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   region_map.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/28 17:15:06 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 09:50:18 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "photon_mapping.h"
#include "hip/hip_runtime.h"
#include "../inc/cuda_call.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

static void init_region_map(t_region *region_map, size_t size, float search_rad);

/*
** Allocated or reallocates memory for the CPU region map used in Progressive Photon Mapping. 
*/

//TODO ADD MALLOC PROTECTIONZ

void	malloc_region_map(t_raytracing_tools *r, t_tile tile)
{
	printf("malloc_region_map\n");
	size_t	size1;
	size_t	size2;
	int		i;

	r->update.photon_map = 2; ///////////
	if (r->scene->is_photon_mapping && r->update.photon_map == 2)
	{
		if (r->h_region_map)
			free(r->h_region_map);
		size1 = sizeof(t_region *) * tile.col * tile.row;
		size2 = sizeof(t_region) * tile.size * tile.size;
		r->h_region_map = (t_region **)malloc(size1);
		i = -1;
		// printf("----rad: %f\n", r->settings.photon_search_radius);
		// printf("region map: %p tile.col: %d tile.row: %d\n", r->h_region_map, tile.col, tile.row);
		while (++i < tile.max)
		{
			r->h_region_map[i] = (t_region *)malloc(size2);
			
			init_region_map(r->h_region_map[i], tile.size * tile.size, r->settings.photon_search_radius);
			// printf("region map[%d/%d]: %p\n", i, tile.max, r->h_region_map[i]);
		}
		r->update.photon_map = 0;
	}
}

static void init_region_map(t_region *region_map, size_t size, float search_rad)
{
	int i;
	
	i = -1;
	while (++i < size)
	{
		region_map[i].hit_pt = v_new(NAN, NAN, NAN);
		region_map[i].ray_dir = v_new(NAN, NAN, NAN);
		region_map[i].normal = v_new(NAN, NAN, NAN);
		region_map[i].radius = search_rad;
		region_map[i].n = 0;
		region_map[i].power = v_new(0, 0, 0);
		region_map[i].kd = NAN;
	}
}

/*
** Allocated or reallocates memory for the GPU region map used in Progressive Photon Mapping. 
*/

void	cuda_malloc_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	printf("cuda_malloc_region_map\n");
	size_t	size;

	if (r->scene->is_photon_mapping)
	{
		if (r->d_region_map)
			hipFree(r->d_region_map);
		size = sizeof(t_region) * tile.size * tile.size;
		gpuErrchk(hipMalloc(&(r->d_region_map), size));
	}
}

/*
** Sets tile region map variales to initial values
*/

void	refresh_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	t_region	empty;
	int			i;

	//does this need to be done on the host, or can it be done in the beginning of the kernel?
	// printf("refresh_region_map_tile\n");
	if (r->scene->is_photon_mapping)
	{
		empty.hit_pt = v_new(NAN, NAN, NAN);
		empty.ray_dir = v_new(NAN, NAN, NAN);
		empty.normal = v_new(NAN, NAN, NAN);
		empty.kd = NAN;
		i = -1;
		while (++i < tile.size * tile.size)
			gpuErrchk((hipMemcpy((r->d_region_map + i), &empty, sizeof(t_region), hipMemcpyHostToDevice)));
	}
}

void	copy_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	//copy over d_region_map over to corresponding h_region_map[i]
	int i;
	int current_tile;

	if (r->scene->is_photon_mapping)
	{
		current_tile = (tile.id.y) * tile.col + (tile.id.x);
		// printf("copy_region_map_tile: current tile: %d\n", current_tile);
		gpuErrchk((hipMemcpy(r->h_region_map[current_tile], r->d_region_map, sizeof(t_region) * tile.size * tile.size, hipMemcpyDeviceToHost)));
	}
}

void	get_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	//copy over h_region_map[i] over to d_region_map
	int i;
	int current_tile;

	if (r->scene->is_photon_mapping)
	{
		current_tile = (tile.id.y) * tile.col + (tile.id.x);
		// printf("get_region_map_tile: current tile: %d\n", current_tile);
		gpuErrchk((hipMemcpy(r->d_region_map, r->h_region_map[current_tile], sizeof(t_region) * tile.size * tile.size, hipMemcpyHostToDevice)));
	}
}

__device__
void	update_region_map(t_raytracing_tools *r, t_ray *cam_ray)
{
	if (r->scene->is_photon_mapping && !v_isnan(cam_ray->hit))
	{
		r->d_region_map->hit_pt = cam_ray->hit;
		r->d_region_map->ray_dir = cam_ray->dir;
		r->d_region_map->normal = v_scale(cam_ray->nhit, cam_ray->n_dir);
		r->d_region_map->kd = r->scene->objects[cam_ray->hit_obj].kd;
	}
}
