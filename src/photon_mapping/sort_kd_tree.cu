/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   sort_kd_tree.cu                                    :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 14:24:44 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/18 14:28:19 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "photon_mapping.h"
#include "hip/hip_runtime.h"

static t_kd_tree	*split_list(t_kd_tree *tree, int length);
static t_kd_tree	*get_next_smallest_by_dim(t_kd_tree **left, t_kd_tree **right, int dim);
static t_kd_tree	*split_list(t_kd_tree *tree, int length);
static int			get_length(t_kd_tree *root);
static t_kd_tree	*get_median(t_kd_tree *root, int length);
static t_kd_tree	*split_without_median(t_kd_tree *root, t_kd_tree **median);
static t_kd_tree	*merge_sort_by_dim(t_kd_tree *root, int dim);

__host__
void				print_photons2(t_kd_tree *tree);

/*
** Creates a sorted kd tree based off the linked list "root"
*/

void	sort_kd_tree(t_kd_tree **root, int dim, t_kd_tree **sorted)
{
	t_kd_tree	*right;
	t_kd_tree	*median;
	int			length;

	dim = (dim == 3) ? 0 : dim; 
	right = NULL;
	*root = merge_sort_by_dim(*root, dim);
	// printf("SORT RESULTS DIM %d %p: \n", dim, *root);
	// print_photons2(*root);
	length = get_length(*root);
	if (length == 1)
	{
		(*sorted) = *root;
		(*sorted)->right = NULL;
		(*sorted)->left = NULL;
		return ;
	}
	length = length / 2 + length % 2;
	median = get_median(*root, length); 
	right = split_without_median(*root, &median);
	// printf("root: %p, median: %p, right: %p\n", *root, median, right);
	*sorted = median;
	(*sorted)->right = NULL;
	(*sorted)->left = NULL;
	if (*root)
	{
		// printf("sort left: %p\n", *root);
		sort_kd_tree(root, dim + 1, &((*sorted)->left));
	}
	if (right)
	{
		// printf("sort right: %p\n", right);
		sort_kd_tree(&right, dim + 1, &((*sorted)->right));
	}
	// printf("done sorting\n");
}

static t_kd_tree		*merge_sort_by_dim(t_kd_tree *root, int dim)
{
	int			length;
	t_kd_tree	*tmp;
	t_kd_tree	*link;
	t_kd_tree	*sorted;


	// printf("merge_sort_by_dim: \n");
	// print_photons2(root);
	tmp = NULL;
	link = NULL;
	// C(1)
	length = get_length(root);
	// printf("length: [%d]\n", length);
	if (length > 1)
	{
		// C(2)
		tmp = split_list(root, length / 2);
		// printf("after split print root:\n");
		// print_photons2(root);
		// printf("after split print tmp:\n");
		// print_photons2(tmp);
		// printf("merge_sort_by_dim root\n");
		root = merge_sort_by_dim(root, dim);	
		// printf("merge_sort_by_dim tmp\n");
		tmp = merge_sort_by_dim(tmp, dim);
	}
	// C(3)
	// if (!root || !root->right)
	// {
	// 	printf("merge_sort_by_dim return : %p\n", root);
	// 	return (root);
	// }
	sorted = NULL;
	// C(4)
	while (root || tmp)
	{
		if (!sorted)
		{
			// C(30)
			sorted = get_next_smallest_by_dim(&root, &tmp, dim);
			link = sorted;
		}
		else
		{
			// C(31)
			link->right = get_next_smallest_by_dim(&root, &tmp, dim);
			link = link->right;
		}
	}
	link->right = NULL;
	// printf("merge_sort_by_dim return : %p\n", sorted);
	return (sorted);
}

static t_kd_tree	*get_next_smallest_by_dim(t_kd_tree **left, t_kd_tree **right, int dim)
{
	t_kd_tree	*tmp;
	int			r_length;
	int			l_length;

	l_length = get_length(*left);
	r_length = get_length(*right);
	tmp = NULL;
	// printf("get_next_smallest_by_dim: left: [%d] right: [%d]\n", l_length, r_length);
	// printf("compare left: %p (%f) ", *left, (l_length ? (*left)->pos.x : 0.0));
	// printf("with right: %p (%f)\n", *right, (r_length ? (*right)->pos.x : 0.0));
	if (!r_length && l_length)
	{
		// C(10)
		tmp = *left;
		// printf("check left length: %d\n", get_length(*left));
		*left = (*left)->right;
		// printf("check left length2: %d\n", get_length(*left));
		// printf("left address: %p\n", *left);
	}
	else if (!l_length && r_length)
	{
		// C(11)
		tmp = *right;
		// printf("check right length: %d\n", get_length(*right));
		*right = (*right)->right;
		// printf("check right length2: %d\n", get_length(*right));
		// printf("right address: %p\n", *right);
	}
	else if ((dim == 0 && (*left)->pos.x < (*right)->pos.x))
	{
		// C(12)
		tmp = *left;
		*left = (*left)->right;
	}
	else if ((dim == 0) && (*left)->pos.x >= (*right)->pos.x)
	{
		// C(13)
		tmp = *right;
		*right = (*right)->right;
	}
	else if (dim == 1 && (*left)->pos.y < (*right)->pos.y)
	{
		// C(14)
		tmp = *left;
		*left = (*left)->right;
	}
	else if (dim == 1 && (*left)->pos.y >= (*right)->pos.y)
	{
		// C(15)
		tmp = *right;
		*right = (*right)->right;
	}
	else if (dim == 2 && (*left)->pos.z < (*right)->pos.z)
	{
		// C(16)
		tmp = *left;
		*left = (*left)->right;
	}
	else if (dim == 2 && (*left)->pos.z >= (*right)->pos.z)
	{
		// C(17)
		tmp = *right;
		*right = (*right)->right;
	}
	// printf("get_next_smallest_by_dim return : %p\n", tmp);
	tmp->right = NULL;
	return (tmp);
}

static t_kd_tree	*split_list(t_kd_tree *tree, int length)
{
	int 		i;
	t_kd_tree	*tmp;
	t_kd_tree	*tmp2;

	tmp = tree;
	i = 0;
	while (++i < length)
	{
		tmp = tmp->right;
	}
	tmp2 = tmp->right;
	tmp->right = NULL;
	return (tmp2);
}

static int	get_length(t_kd_tree *root)
{
	int i;

	// printf("get_length: ");
	i = 0;
	while (root)
	{
			// printf("%p\n", root);
		++i;
		root = root->right;
	}
	// printf("%d\n", i);
	return (i);
}


static t_kd_tree	*get_median(t_kd_tree *root, int length)
{
	t_kd_tree	*median;
	int			i;

	i = 0;
	while (++i < length) 
		root = root->right;
	// printf("found median: [%p] [%f, %f, %f] (length: %d)\n", root, root->pos.x, root->pos.y, root->pos.z, length);
	return (root);
}

static t_kd_tree	*split_without_median(t_kd_tree *root, t_kd_tree **median)
{
	t_kd_tree	*right;
	// int			i;

	// printf("split_without_median: %p\n", *median);
	right = NULL;
	// i = 0;
	//iterate through list until link BEFORE the median
	// printf("root: %p\n", root);
	while (root != *median && root->right != *median)
	{
		root = root->right;
		// printf("root: %p\n", root);
	}
	// printf("stopped at: %p with i: %d\n", root, i);  
	if (root != *median)  
	{ 
		right = root->right->right;
		// printf("set right to %p\n", right);
	}
	else
		right = root->right;
	root->right = NULL;
	return (right);
}

__host__
void	print_photons2(t_kd_tree *tree)
{
	if (!tree)
		return ;
	// print_photons(tree->left);
	printf("photon: [%p] [%f, %f, %f]\n", tree, tree->pos.x, tree->pos.y, tree->pos.z);
	print_photons2(tree->right);
}