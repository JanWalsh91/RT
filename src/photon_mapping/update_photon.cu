#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   update_photon.cu                                   :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/10 15:50:15 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/02 15:16:36 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "photon_mapping.h"
#include "rt.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__
static void		save_photon(t_photon *photon_list, t_ray *ray, t_object *obj);
__device__
static float 	get_probability(float *p, t_vec3 o_col, float k, t_vec3 p_col);
__device__
static void		redirect_photon_diffuse(t_raytracing_tools *r, t_ray *ray);
__device__
static void		redirect_photon_specular(t_raytracing_tools *r, t_ray *ray);
__device__
static void		redirect_photon_transmit(t_raytracing_tools *r, t_ray *ray);
__device__
static int		fresnel_reflect(t_raytracing_tools *r, t_ray *ray);
__device__
static void		get_iors(float *n1, float *n2, t_raytracing_tools *r, t_ray *ray);
__device__
static void		scale_color(t_ray *ray, float p, float k, t_vec3 obj_col);

//put scale_color in redirect_... functions
__device__
t_color			update_photon(t_raytracing_tools *r, t_ray *ray)
{
	float	rand_f;
	float	tmp;
	float 	p;
	
	
	// printf("update_photon: type: [%d],  diffuse: [%f], reflection: [%f], transparency: [%f]\n",
	// ray->type,
	// r->scene->objects[ray->hit_obj].kd,
	// r->scene->objects[ray->hit_obj].reflection,
	// r->scene->objects[ray->hit_obj].transparency);
	
	if (r->scene->objects[ray->hit_obj].kd > 0.0 /*&& ray->type == R_DIRECT_PHOTON*/)
		save_photon(r->scene->photon_list + r->idx * PHOTON_BOUNCE_MAX, ray, &r->scene->objects[ray->hit_obj]);
	rand_f = hiprand_uniform(r->devStates);
	tmp = 0;
	p = NAN;
	ray->type = R_INDIRECT_PHOTON;
	// printf("update photon: rand_f: [%f]\n", rand_f);
	if ((tmp += get_probability(&p, r->scene->objects[ray->hit_obj].col, r->scene->objects[ray->hit_obj].kd / 3.0, col_to_vec(ray->col))) > rand_f)
	{
		// printf("[%d] cast as diffuse. Rand: [%f] tmp: [%f]\n", r->idx, rand_f, tmp);
		redirect_photon_diffuse(r, ray);
		scale_color(ray, p, r->scene->objects[ray->hit_obj].kd, r->scene->objects[ray->hit_obj].col);
		// printf("new dir: [%f, %f, %f]\n", ray->dir.x, ray->dir.y, ray->dir.z);
	}
	else if ((tmp += get_probability(&p, r->scene->objects[ray->hit_obj].col, r->scene->objects[ray->hit_obj].reflection / 3.0, col_to_vec(ray->col))) > rand_f)
	{
		// printf("[%d] cast as reflection. Rand: [%f] tmp: [%f]\n", r->idx, rand_f, tmp);
		redirect_photon_specular(r, ray);
		scale_color(ray, p, r->scene->objects[ray->hit_obj].reflection, r->scene->objects[ray->hit_obj].col);
	}
	else if ((tmp += get_probability(&p, r->scene->objects[ray->hit_obj].col, r->scene->objects[ray->hit_obj].transparency / 3.0, col_to_vec(ray->col))) > rand_f)
	{
		if (fresnel_reflect(r, ray))
		{
			// printf("[%d] cast as reflection due to fresnel. Rand: [%f] tmp: [%f]\n", r->idx, rand_f, tmp);
			redirect_photon_specular(r, ray);
			scale_color(ray, p, r->scene->objects[ray->hit_obj].reflection, r->scene->objects[ray->hit_obj].col);
		}
		else
		{
			// printf("[%d] cast as refraction. Rand: [%f] tmp: [%f]\n", r->idx, rand_f, tmp);
			redirect_photon_transmit(r, ray);
			scale_color(ray, p, r->scene->objects[ray->hit_obj].transparency, r->scene->objects[ray->hit_obj].col);
		}
	} 
	else
	{
		// printf("[%d] absorbed. Rand: [%f] tmp: [%f]\n", r->idx, rand_f, tmp);
		return (c_new(0, 0, 0));
	}
	// if energy high enough, shoot photon again: cast_primary_ray();
	if ((ray->col.r + ray->col.g + ray->col.b) > 10)
		return (cast_primary_ray(r, ray));
	else
		return (c_new(0, 0, 0));
	// printf("------cast again with type[%d]\n", ray->type);
}

__device__
static void		save_photon(t_photon *photon_list, t_ray *ray, t_object *obj)
{
	int i;

	i = 0;           
	// printf("0: [%f] - 1: [%f] - 3: [%f] - 3: [%f] - \n", photon_list[0].pos.x, photon_list[1].pos.x, photon_list[2].pos.x, photon_list[3].pos.x);                               
	while (!v_isnan(photon_list[i].pos) && i < PHOTON_BOUNCE_MAX)
		++i;
	// printf("save photon: at %d [%f, %f, %f]\n", i, photon_list[i].pos.x, photon_list[i].pos.y, photon_list[i].pos.z);
	// printf("found: [%d]: [%f, %f, %f]\n", i, photon_list[i].pos.x, photon_list[i].pos.y, photon_list[i].pos.z);
	photon_list[i].pos = ray->hit;
	photon_list[i].dir = ray->dir;
	photon_list[i].col = vec_to_col(get_object_color(obj, ray));
	photon_list[i].n = v_scale(ray->nhit, ray->n_dir);
	// printf("save photon[%d]: pos: [%f, %f, %f] dir: [%f, %f, %f] col: [%d, %d, %d]\n", i,
		// photon_list[i].pos.x, photon_list[i].pos.y, photon_list[i].pos.z,
		// photon_list[i].dir.x, photon_list[i].dir.y, photon_list[i].dir.z,
		// photon_list[i].col.r, photon_list[i].col.g, photon_list[i].col.b);
}

__device__
static float 	get_probability(float *p, t_vec3 o_col, float k, t_vec3 p_col)
{
	*p = (max(o_col.x * p_col.x, max(o_col.y * p_col.y, o_col.z * p_col.z)) * k / 255) / 
	max(p_col.x, max(p_col.y, p_col.z)); 
	return (*p);
}

__device__
static void			redirect_photon_diffuse(t_raytracing_tools *r, t_ray *ray)
{
	t_vec3	rand_dir;
	hiprandState state;
	ray->origin = v_add(ray->hit, v_scale(ray->nhit, ray->n_dir * BIAS));

	state = *r->devStates;
	rand_dir.x = ray->dir.x * (hiprand_uniform(&state) - 0.5);
	rand_dir.y = ray->dir.y * (hiprand_uniform(&state) - 0.5);
	rand_dir.z = ray->dir.z * (hiprand_uniform(&state) - 0.5);
	*r->devStates = state;
	rand_dir = v_norm(rand_dir);
	ray->dir = (v_dot(rand_dir, v_scale(ray->nhit, ray->n_dir)) < 0) ? v_scale(rand_dir, -1): rand_dir;
}

__device__
static void			redirect_photon_specular(t_raytracing_tools *r, t_ray *ray)
{
	ray->origin = v_add(ray->hit, v_scale(ray->nhit, ray->n_dir * BIAS));
	ray->dir = reflect(ray->dir, v_scale(ray->nhit, ray->n_dir));
}

__device__
static void			redirect_photon_transmit(t_raytracing_tools *r, t_ray *ray)
{
	float n1;
	float n2;
	
	update_ior(&n1, &n2, r, ray);
	ray->ior = n2;
	ray->origin = v_add(ray->hit, v_scale(ray->nhit, -ray->n_dir * BIAS)); 
	ray->dir = refract(ray->dir, v_scale(ray->nhit, ray->n_dir), n1, n2);
}



/*
** ray: incoming ray.
** p: probability of particular redirection (diffuse, refelcted, refracted)
** k: coefficient of corresponding redirection (diffuse, reflection, transparency)
** obj_col: color of object hit
** P_refl = P_inc * k / p
*/

__device__
static void			scale_color(t_ray *ray, float p, float k, t_vec3 obj_col)
{
	t_color p_inc;

	// printf("scale color input: p: [%f] k : [%f]\n", p, k);
	p_inc = ray->col;
	ray->col.r = p_inc.r * k * (obj_col.x / (p * 255.0));
	ray->col.g = p_inc.g * k * (obj_col.y / (p * 255.0));
	ray->col.b = p_inc.b * k * (obj_col.z / (p * 255.0));
	// printf("scale color: old: [%d, %d, %d] new: [%d, %d, %d]\n", p_inc.r, p_inc.g, p_inc.b, ray->col.r, ray->col.g, ray->col.b);
}

__device__
static int			fresnel_reflect(t_raytracing_tools *r, t_ray *ray)
{
	float	n1;
	float 	n2;
	float	f;

	if (!r->scene->is_fresnel)
		return (0);
	get_iors(&n1, &n2, r, ray);
	f = get_fresnel_ratio(ray->dir, v_scale(ray->nhit, ray->n_dir), n1, n2);
	return (f > hiprand_uniform_double(r->devStates));
}



/*
** Same as update_ior function, but only to retrieve the iors, not to update the lists and ray iors.
*/

__device__
static void		get_iors(float *n1, float *n2, t_raytracing_tools *r, t_ray *ray)
{
		int	i;
	float ret;

	if (ray->n_dir == 1)
	{
		i = -1;
		*n1 = ray->ior;
		while (r->ior_list[++i] > 0.001);
		// r->ior_list[i] = r->scene->objects[ray->hit_obj].ior;
		*n2 = r->scene->objects[ray->hit_obj].ior;
	}
	else
	{
		i = r->scene->ray_depth;
		while (i >= 0)
		{
			ret = r->ior_list[i] - r->scene->objects[ray->hit_obj].ior;
			if (ret < 0.001 && ret > -0.001)
				break;
			--i;
		}
		if (!(ret < 0.001 && ret > -0.001))
		{
			*n1 = ray->ior;
			i = 0;
			while (r->ior_list[i] > 0.001)
				++i;
			*n2 = r->ior_list[i - 1];
			return ;
		}
		// r->ior_list[i] = 0;
		// while (i < r->scene->ray_depth && r->ior_list[i] < 0.001)
		// {
		// 	r->ior_list[i] = r->ior_list[i + 1];
		// 	r->ior_list[i + 1] = 0;
		// 	++i;
		// }
		i = 0;
		while (r->ior_list[i] > 0.001)
			++i;
		*n2 = r->ior_list[i - 1];
		*n1 = ray->ior;
	}
	i = -1;
}