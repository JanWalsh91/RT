/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   update_region_map.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/28 17:15:06 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/05 15:19:20 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "hip/hip_runtime.h"
#include "../inc/cuda_call.h"

void	copy_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	//copy over d_region_map over to corresponding h_region_map[i]
	int current_tile;

	if (r->scene->is_photon_mapping)
	{
		current_tile = (tile.id.y) * tile.col + (tile.id.x);
		// printf("copy_region_map_tile: current tile: %d\n", current_tile);
		gpu_errchk((hipMemcpy(r->h_region_map[current_tile], r->d_region_map,
			sizeof(t_region) * tile.size * tile.size, hipMemcpyDeviceToHost)));
	}
}

void	get_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	//copy over h_region_map[i] over to d_region_map
	int current_tile;

	if (r->scene->is_photon_mapping)
	{
		current_tile = (tile.id.y) * tile.col + (tile.id.x);
		// printf("get_region_map_tile: current tile: %d\n", current_tile);
		gpu_errchk((hipMemcpy(r->d_region_map, r->h_region_map[current_tile],
			sizeof(t_region) * tile.size * tile.size, hipMemcpyHostToDevice)));
	}
}

__device__
void	update_region_map(t_raytracing_tools *r, t_ray *cam_ray)
{
	if (r->scene->is_photon_mapping && !v_isnan(cam_ray->hit))
	{
		r->d_region_map->hit_pt = cam_ray->hit;
		r->d_region_map->ray_dir = cam_ray->dir;
		r->d_region_map->normal = v_scale(cam_ray->nhit, cam_ray->n_dir);
		r->d_region_map->kd = r->scene->objects[cam_ray->hit_obj].kd;
	}
}
