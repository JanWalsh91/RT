#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/01 17:40:27 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"
#include "../../inc/cuda_call.h"

// static void	display_percentage(int num);

/*
** Updates a camera's pixel_map (color of image pixels).
*/


__host__ void	cuda_push_camera(t_camera **cameras_head, t_camera *new_camera)
{
	t_camera	*cam_ptr;

	if (new_camera)
	{
		if (!(*cameras_head))
			*cameras_head = new_camera;
		else
		{
			cam_ptr = *cameras_head;
			while (cam_ptr->next)
				cam_ptr = cam_ptr->next;
			cam_ptr->next = new_camera;
			new_camera->prev = cam_ptr;
		}
		new_camera->next = NULL;
	}
}

__host__ void	allocate_cameras(t_camera **d_cams, t_camera *h_cams, t_pt2 res)
{
	t_camera *p_cam;
	t_camera *new_cam;
	int		i;

	new_cam = NULL;
	*d_cams = NULL;
	p_cam = h_cams;
	while (p_cam)
	{
		hipMallocManaged(&new_cam, sizeof(t_camera));
		memcpy(new_cam, p_cam, sizeof(t_camera));
		i = -1;
		hipMallocManaged(&new_cam->pixel_map, sizeof(t_color *) * res.y);
		while (++i < res.y)
			hipMallocManaged(&new_cam->pixel_map[i], sizeof(t_color) * res.x);
		cuda_push_camera(d_cams, new_cam);
		p_cam = p_cam->next;
	}
}

__host__ void	cuda_push_light(t_light **lights_head, t_light *new_light)
{
	t_light	*light_ptr;

	if (new_light)
	{
		if (!(*lights_head))
			*lights_head = new_light;
		else
		{
			light_ptr = *lights_head;
			while (light_ptr->next)
				light_ptr = light_ptr->next;
			light_ptr->next = new_light;
		}
		new_light->next = NULL;
	}
}

__host__ void 	allocate_lights(t_light **d_lights, t_light *h_lights)
{
	t_light *p_light;
	t_light *new_light;

	new_light = NULL;
	*d_lights = NULL;
	p_light = h_lights;
	while (p_light)
	{
		hipMallocManaged(&new_light, sizeof(t_light));
		memcpy(new_light, p_light, sizeof(t_light));
		cuda_push_light(d_lights, new_light);
		p_light = p_light->next;
	}
}

__host__ void	cuda_push_object(t_object **objects_head, t_object *new_object)
{
	t_object	*obj_ptr;

	if (new_object)
	{
		if (!(*objects_head))
			*objects_head = new_object;
		else
		{
			obj_ptr = *objects_head;
			while (obj_ptr->next)
				obj_ptr = obj_ptr->next;
			obj_ptr->next = new_object;
		}
		new_object->next = NULL;
	}
}

__host__ void	allocate_objects(t_object **d_objs, t_object *h_objs)
{
	t_object *p_obj;
	t_object *new_obj;

	new_obj = NULL;
	*d_objs = NULL;
	p_obj = h_objs;
	while (p_obj)
	{
		hipMallocManaged(&new_obj, sizeof(t_object));
		memcpy(new_obj, p_obj, sizeof(t_object));
		cuda_push_object(d_objs, new_obj);
		p_obj = p_obj->next;
	}
}

__host__
void	cuda_push_scene(t_scene **scenes_head, t_scene *new_scene)
{
	t_scene	*scene_ptr;

	if (new_scene)
	{
		if (!*scenes_head)
			*scenes_head = new_scene;
		else
		{
			scene_ptr = *scenes_head;
			while (scene_ptr->next)
				scene_ptr = scene_ptr->next;
			scene_ptr->next = new_scene;
			new_scene->prev = scene_ptr;
		}
	}
}

__host__ void	allocate_scenes(t_raytracing_tools *d_r, t_raytracing_tools *h_r)
{
	t_scene	*p_scene;
	t_scene	*new_scene;

	new_scene = NULL;
	d_r->scenes = NULL;
	p_scene = h_r->scenes;
	while (p_scene)
	{
		//allocate mem
		hipMallocManaged(&new_scene, sizeof(t_scene));
		memcpy(new_scene, p_scene, sizeof(t_scene));
		if (!d_r->scenes)
			d_r->scenes = new_scene;
		//allocate cameras
		allocate_cameras(&new_scene->cameras, p_scene->cameras, p_scene->res);
		//allocate lights
		allocate_lights(&new_scene->lights, p_scene->lights);
		//allocate objects
		allocate_objects(&new_scene->objects, p_scene->objects);
		cuda_push_scene(&d_r->scenes, new_scene);
		p_scene = p_scene->next;
	}
}

__host__ t_raytracing_tools  *allocate_memory(t_raytracing_tools *h_r)
{
	C(11)
	t_raytracing_tools *d_r = NULL;

	hipMallocManaged(&d_r, sizeof(t_raytracing_tools));
	// printf("%d\n", d_r->pix.x);
	C(12)
	memcpy(d_r, h_r, sizeof(t_raytracing_tools));
	C(13)
	allocate_scenes(d_r, h_r);
	return (d_r);
}

__global__ void render_pixel(t_raytracing_tools *r)
{
	t_pt2	pixel;
	t_ray	cam_ray;

	pixel.x = blockIdx.x * blockDim.x + threadIdx.x;
	pixel.y = blockIdx.y * blockDim.y + threadIdx.y;

	r->pix.x = pixel.x;
	r->pix.y = pixel.y;
	cam_ray = init_camera_ray(pixel, r->scenes);
	r->scenes->cameras->pixel_map[pixel.y][pixel.x] = cast_primary_ray(r, &cam_ray);
}

__host__
void		render(t_raytracing_tools *r)
{
/*	t_ray	cam_ray;

	r->pix.y = -1;
	while (++r->pix.y < r->scenes->res.y)
	{
		r->pix.x = -1;
		display_percentage((int)((double)r->pix.y /
			r->scenes->res.y * (100)) + 1);
		while (++r->pix.x < r->scenes->res.x)
		{
			cam_ray = init_camera_ray(r->pix, r->scenes);
			r->scenes->cameras->pixel_map[r->pix.y][r->pix.x] =
				cast_primary_ray(r, &cam_ray);
		}
	}*/

	t_raytracing_tools *d_r;
	C(1)
	d_r = allocate_memory(r);
	C(2)
	dim3 block_size;
	dim3 grid_size;


	block_size = dim3(16, 16);
	grid_size = dim3(r->scenes->res.x / block_size.x + 1, r->scenes->res.y / block_size.y + 1);
	C(3)
	// print_scenes(d_r->scenes);
	render_pixel<<<grid_size, block_size>>>(d_r);
	hipDeviceSynchronize();
	// printf("%f\n", d_r->scenes->cameras->pos.y);
	int i;
	i = -1;


	C(4)
	hipMemcpy(r->scenes->cameras->pixel_map, d_r->scenes->cameras->pixel_map, sizeof(t_color *) * r->scenes->res.y, hipMemcpyDeviceToHost);
	// r->scenes->cameras->pixel_map = d_r->scenes->cameras->pixel_map;
	C(5)
	while (++i < r->scenes->res.y) ;
		//printf("%f\n", d_r->scenes->cameras->pixel_map[i][0].x);
		//memcpy(r->scenes->cameras->pixel_map[i], d_r->scenes->cameras->pixel_map[i], sizeof(t_color) * r->scenes->res.x);
		// r->scenes->cameras->pixel_map[i] = d_r->scenes->cameras->pixel_map[i];
	C(6)
}

// __device__
// static void	display_percentage(int num)
// {
// 	ft_printf("Loading...%i%%\r", num);
// }
