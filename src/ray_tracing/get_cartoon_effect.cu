#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_cartoon_effect.cu                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/07 20:55:19 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/08 12:26:29 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../../inc/cuda_call.h"

/*
** cartoontools: 
** x: bin_size
** y: tolerance
** z: radius_filter
*/
__global__
void	cartoonize(t_scene *scene, t_color *pixel_map, t_vec3 c);
__device__
void	create_edges(t_scene *scene, t_color *pixel_map, t_vec3 c, t_pt2 pix);
__device__
void	reset_values(int *f, t_dpt2 comp[3]);
__device__
void	add_colors(t_dpt2 comp[3], t_color *pixel_map, int i1, int i2);
__device__
bool	high_contrast(t_dpt2 comp[3], float tolerance);
__device__
void	set_black(t_color *pix, int x);

void	get_cartoon_effect(t_raytracing_tools *r, t_vec3 cartoon_tools)
{
	dim3			block_size;
	dim3			grid_size;
	
	block_size = dim3(32, 32, 1);
	grid_size = dim3(r->scene->res.x / 32 + 1, r->scene->res.y / 32 + 1);
	cartoonize<<<grid_size, block_size>>>(r->d_scene, r->d_pixel_map, cartoon_tools);
	cuda_check_kernel_errors();
}
__global__
void	cartoonize(t_scene *scene, t_color *pixel_map, t_vec3 c)
{
	t_pt2	pix;
	int		idx;

	pix.x = blockDim.x * blockIdx.x + threadIdx.x;
	pix.y = blockDim.y * blockIdx.y + threadIdx.y;
	idx = scene->res.x * pix.y + pix.x;
	if (pix.x >= scene->res.x || pix.y >= scene->res.y)
		return ;
	pixel_map[idx].r = pixel_map[idx].r - pixel_map[idx].r % (int)c.x;
	pixel_map[idx].g = pixel_map[idx].g - pixel_map[idx].g % (int)c.x;
	pixel_map[idx].b = pixel_map[idx].b - pixel_map[idx].b % (int)c.x;
	// pixel_map[idx] = c_new(255, 0, 0);
	// C(1)
	__syncthreads();
	// C(2)
	// create_edges(scene, pixel_map, c, pix);
}

__device__
bool	high_contrast2(t_dpt2 comp[3], float tolerance)
{
	if (fabs(comp[0].x - comp[0].y) > tolerance ||
		fabs(comp[1].x - comp[1].y) > tolerance ||
		fabs(comp[2].x - comp[2].y) > tolerance)
		{
			// printf("fabs 1 : %f vs %f\n", fabs(comp[0].x - comp[0].y), tolerance);
			// printf("fabs 2 : %f vs %f\n", fabs(comp[1].x - comp[1].y), tolerance);
			// printf("fabs 3 : %f vs %f\n", fabs(comp[2].x - comp[2].y), tolerance);
		return (true);
		}
	return (false);
}

__device__
void	create_edges(t_scene *scene, t_color *pixel_map, t_vec3 c, t_pt2 pix)
{
	int		idx;
	int 	f;
	t_dpt2	comp[3];
	bool	is_black;

	reset_values(&f, comp);
	idx = scene->res.x * pix.y + pix.x;
	is_black = 0;
	while (++f < c.z)
		if (pix.x - f > 0 && pix.x + f < scene->res.x)
			add_colors(comp, pixel_map, idx - f, idx + f);
	if (high_contrast(comp, c.y))
		is_black = 1;
	reset_values(&f, comp);
	f = 0;
	while (++f < c.z && !is_black)
		if (pix.y - f > 0 && pix.y + f < scene->res.y)
			add_colors(comp, pixel_map, idx - f * scene->res.x, idx + f * scene->res.x);
	if (high_contrast(comp, c.y))
	{
		// printf("set black: [%d, %d]\n", pix.x, pix.y);
		is_black = 1;
	}
	__syncthreads();
	return (set_black(&pixel_map[idx], is_black));
}

__device__
void	reset_values(int *f, t_dpt2 comp[3])
{
	*f = 0;
	comp[0].x = 0;
	comp[0].y = 0;
	comp[1].x = 0;
	comp[1].y = 0;
	comp[2].x = 0;
	comp[2].y = 0;
}

__device__
void	add_colors(t_dpt2 comp[3], t_color *pixel_map, int i1, int i2)
{
	comp[0].x += (float)pixel_map[i1].r;
	comp[0].y += (float)pixel_map[i2].r;
	comp[1].x += (float)pixel_map[i1].g;
	comp[1].y += (float)pixel_map[i2].g;
	comp[2].x += (float)pixel_map[i1].b;
	comp[2].y += (float)pixel_map[i2].b;
}

__device__
bool	high_contrast(t_dpt2 comp[3], float tolerance)
{
	if (comp[0].x - comp[0].y < -tolerance ||
		comp[1].x - comp[1].y < -tolerance ||
		comp[2].x - comp[2].y < -tolerance)
	{
			// printf("high contraSt\n");
		return (true);
	}
	if (comp[1].x - comp[1].y < -tolerance)
	{
		// return (true);
	}
	return (false);
}


__device__
void	set_black(t_color *pix, int x)
{
	if (x)
		*pix = c_new(0, 0, 0);
}