#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_cone_intersection.cu                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:30:04 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/05 12:08:04 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

__device__
static void	get_finite_cone_intersection(t_ray *ray,
	t_object *object, t_intersection_tools *i);
static bool	lower_than_min(double r, t_intersection_tools *i, t_object *obj,
	t_ray *ray);
static bool	higher_than_max(double r, t_intersection_tools *i, t_object *obj,
	t_ray *ray);

/*
** Checks for a ray-cone intersection.
*/

__device__
bool		get_cone_intersection(t_raytracing_tools *r, t_ray *ray,
			int index)
{
	t_intersection_tools i;

	i.d1 = tan(r->scenes->objects[index].angle);
	i.v1 = v_sub(ray->origin, r->scenes->objects[index].pos);
	i.q.x = v_dot(ray->dir, ray->dir) - (1.0 + i.d1 * i.d1) *
		pow(v_dot(ray->dir, r->scenes->objects[index].dir), 2.0);
	i.q.y = 2 * (v_dot(ray->dir, i.v1) - (1.0 + i.d1 * i.d1) *
		v_dot(ray->dir, r->scenes->objects[index].dir) * v_dot(i.v1, r->scenes->objects[index].dir));
	i.q.z = v_dot(i.v1, i.v1) - (1.0 + i.d1 * i.d1) * pow(v_dot(i.v1,
		r->scenes->objects[index].dir), 2.0);
	if (!solve_quadratic(i.q, &i.r1, &i.r2) || (i.r1 < 0 && i.r2 < 0))
		return (false);
	if (i.r2 < i.r1)
		ft_swapd(&i.r1, &i.r2);
	get_finite_cone_intersection(ray, &r->scenes->objects[index], &i);
	(i.r1 < 0 || isnan(i.r1)) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0 || isnan(i.r1))
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0;
	if (ray->type == R_PRIMARY && r->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_CONE;
	}
	return (true);
}

__device__
static void	get_finite_cone_intersection(t_ray *ray,
	t_object *obj, t_intersection_tools *i)
{
	bool	r1_too_low;
	bool	r1_too_high;
	bool	r2_too_low;
	bool	r2_too_high;

	r1_too_low = lower_than_min(i->r1, i, obj, ray) || i->r1 == -1;
	r1_too_high = higher_than_max(i->r1, i, obj, ray) || i->r1 == -1;
	r2_too_low = lower_than_min(i->r2, i, obj, ray) || i->r2 == -1;
	r2_too_high = higher_than_max(i->r2, i, obj, ray) || i->r2 == -1;
	if (r1_too_low || r1_too_high)
	{
		i->r1 = NAN;
		if (r2_too_low || r2_too_high)
			i->r2 = NAN;
	}
	else if (!r1_too_low && !r1_too_high)
		i->r2 = NAN;
}

__device__
static bool	lower_than_min(double r, t_intersection_tools *i, t_object *obj,
	t_ray *ray)
{
	if (r > 0)
	{
		if (v_dot(obj->dir, v_sub(v_add(ray->origin,
			v_scale(ray->dir, r)), obj->pos)) < 0.0)
			return (true);
	}
	else
		return (true);
	return (false);
}

__device__
static bool	higher_than_max(double r, t_intersection_tools *i, t_object *obj,
	t_ray *ray)
{
	if (r > 0)
	{
		if (v_dot(obj->dir, v_sub(v_add(ray->origin, v_scale(ray->dir, r)),
			v_add(obj->pos, v_scale(obj->dir, obj->height)))) > 0.0)
			return (true);
	}
	else
		return (true);
	return (false);
}
