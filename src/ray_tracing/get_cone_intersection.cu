#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_cone_intersection.cu                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:30:04 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/08 12:34:18 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

__device__
static void	get_finite_cone_intersection(t_ray *ray,
	t_object *object, t_intersection_tools *i);
__device__
static bool	lower_than_min(float r, t_intersection_tools *i, t_object *obj,
	t_ray *ray);
__device__
static bool	higher_than_max(float r, t_intersection_tools *i, t_object *obj,
	t_ray *ray);

/*
** Checks for a ray-cone intersection.
*/

__device__
bool		get_cone_intersection(t_raytracing_tools *r, t_ray *ray,
			int index)
{
	t_intersection_tools i;

	i.d1 = tan(r->scene->objects[index].angle);
	i.v1 = v_sub(ray->origin, r->scene->objects[index].pos);
	i.q.x = v_dot(ray->dir, ray->dir) - (1.0 + i.d1 * i.d1) *
		pow(v_dot(ray->dir, r->scene->objects[index].dir), 2.0);
	i.q.y = 2 * (v_dot(ray->dir, i.v1) - (1.0 + i.d1 * i.d1) *
		v_dot(ray->dir, r->scene->objects[index].dir) * v_dot(i.v1, r->scene->objects[index].dir));
	i.q.z = v_dot(i.v1, i.v1) - (1.0 + i.d1 * i.d1) * pow(v_dot(i.v1,
		r->scene->objects[index].dir), 2.0);
	if (!solve_quadratic(i.q, &i.r1, &i.r2) || (i.r1 < 0 && i.r2 < 0))
		return (false);
	if (i.r2 < i.r1)
		ft_swapf(&i.r1, &i.r2);
	get_finite_cone_intersection(ray, &r->scene->objects[index], &i);
	(i.r1 < 0 || isnan(i.r1)) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0 || isnan(i.r1))
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0;
	if (ray->type == R_PRIMARY && r->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_CONE;
	}
	return (true);
}

__device__
static void	get_finite_cone_intersection(t_ray *ray,
	t_object *obj, t_intersection_tools *i)
{
	bool	r1_too_low;
	bool	r1_too_high;
	bool	r2_too_low;
	bool	r2_too_high;

	r1_too_low = lower_than_min(i->r1, i, obj, ray) || i->r1 == -1;
	r1_too_high = higher_than_max(i->r1, i, obj, ray) || i->r1 == -1;
	r2_too_low = lower_than_min(i->r2, i, obj, ray) || i->r2 == -1;
	r2_too_high = higher_than_max(i->r2, i, obj, ray) || i->r2 == -1;
	if (r1_too_low || r1_too_high)
	{
		i->r1 = NAN;
		if (r2_too_low || r2_too_high)
			i->r2 = NAN;
	}
	else if (!r1_too_low && !r1_too_high)
		i->r2 = NAN;
}

__device__
static bool	lower_than_min(float r, t_intersection_tools *i, t_object *obj,
	t_ray *ray)
{
	if (r > 0)
	{
		if (v_dot(obj->dir, v_sub(v_add(ray->origin,
			v_scale(ray->dir, r)), obj->pos)) < 0.0)
			return (true);
	}
	else
		return (true);
	return (false);
}

__device__
static bool	higher_than_max(float r, t_intersection_tools *i, t_object *obj,
	t_ray *ray)
{
	if (r > 0)
	{
		if (v_dot(obj->dir, v_sub(v_add(ray->origin, v_scale(ray->dir, r)),
			v_add(obj->pos, v_scale(obj->dir, obj->height)))) > 0.0)
			return (true);
	}
	else
		return (true);
	return (false);
}

__device__
t_pt2	get_uv_cone(t_object *obj, t_ray *ray, t_pt2 *dim)
{
	t_pt2	coord;
	t_vec3	hit_center;

	hit_center = v_scale(ray->nhit, -1);
	coord.x = (0.5 + (atan2f(hit_center.z, hit_center.x) / (float)(2 * M_PI))) * dim->x;
	coord.y = (sqrtf(v_dist(ray->hit, obj->pos) * v_dist(ray->hit, obj->pos) - (obj->rad * obj->rad)) / (sqrtf(obj->height * obj->height + obj->rad * obj->rad))) * dim->y;
	coord.x %= dim->x;
	coord.y %= dim->y;

	return (coord);
}
