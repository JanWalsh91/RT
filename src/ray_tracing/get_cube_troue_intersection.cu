#include "hip/hip_runtime.h"
#include "rt.h"

__device__
bool		get_torus_intersection(t_raytracing_tools *r, t_ray *ray,
				int index)
{
	t_quartic	qua;
	t_object	*obj = &r->scene->objects[index];
	t_vec4im	sol;
	float		res;

	qua.a = (powf(ray->dir.x, 4)) + (powf(ray->dir.y, 4) + (pow(ray->dir.z, 4))); 
	qua.b =	(4 * ((powf(ray->dir.x, 3) * ray->origin.x) + (powf(ray->dir.y, 3) * ray->origin.y) +
		(powf(ray->dir.y, 3) * ray->origin.y)));
	qua.c = (6 * ((powf(ray->dir.x, 2) * powf(ray->origin.x,2)) + (powf(ray->dir.y, 2) * powf(ray->origin.y, 2)) + (powf(ray->dir.y, 2) * powf(ray->origin.y, 2))) - 
		(5 (powf(ray->dir.x, 2) + powf(ray->dir.y, 2) + powf(ray->dir.z, 2)))); 
	qua.d = (4 * ((powf(ray->origin.x, 3) * ray->dir.x) + (powf(ray->origin.y, 3) * ray->dir.y) + (powf(ray->origin.y, 3) * ray->dir.y)) -
		(10 * (ray->dir.x * ray->origin.x) + (ray->dir.y * ray->origin.y) + (ray->dir.y * ray.origin.y)));
	qua.e = (powf(ray->origin.x, 4) + powf(ray->origin.y, 4) + powf(ray->origin.z, 4) - (5 (powf(ray->origin.x, 2) + powf(ray->origin.y, 2) + powf(ray->origin.z, 2))));
	

	if(!solve_quartic(&qua, &sol);
		return(false);
	if (isnan(sol.x.i) || isnan(sol.y.i))
		res = choose_between_four_roots(sol.w.r, sol.x.r, sol.y.r, sol.z.r);
	else
		res = choose_between_two_roots(sol.w.r, sol.z.r);
	if (res < 0)
		return (false);
	if (r->pix.y == 600 && r->pix.x == 600)
	{
	 r->t > res ? ray->t = res : 0;
	if (ray->type == R_PRIMARY && r->t > res)
	{
		ray->hit_obj = index;
		ray->hit_type = T_CUBETROUE;
	}
	return (true);
}