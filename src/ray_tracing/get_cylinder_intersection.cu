#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_cylinder_intersection.cu                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:27:49 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/05 12:08:04 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

static void	get_finite_cylinder_intersection(t_raytracing_tools *r, t_ray *ray,
	int index, t_intersection_tools *i);

/*
** Checks for a cylinder-ray intersection.
*/

__device__
bool		get_cylinder_intersection(t_raytracing_tools *r, t_ray *ray,
	int index)
{
	t_intersection_tools	i;

	i.v3 = v_sub(ray->origin, r->scenes->objects[index].pos);
	i.v1 = v_sub(ray->dir, v_scale(r->scenes->objects[index].dir, v_dot(ray->dir, r->scenes->objects[index].dir)));
	i.q.x = v_dot(i.v1, i.v1);
	i.v2 = v_sub(i.v3, v_scale(r->scenes->objects[index].dir, v_dot(i.v3, r->scenes->objects[index].dir)));
	i.q.y = 2 * v_dot(i.v1, i.v2);
	i.q.z = v_dot(i.v2, i.v2) - pow(r->scenes->objects[index].rad, 2);
	if (!solve_quadratic(i.q, &i.r1, &i.r2))
		return (false);
	if (i.r2 < i.r1)
		ft_swapd(&i.r1, &i.r2);
	get_finite_cylinder_intersection(r, ray, index, &i);
	(i.r1 < 0 || isnan(i.r1)) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0 || isnan(i.r1))
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0;
	if (ray->type == R_PRIMARY && r->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_CYLINDER;
	}
	return (true);
}

__device__
static void	get_finite_cylinder_intersection(t_raytracing_tools *r, t_ray *ray,
	int index, t_intersection_tools *i)
{
	if (i->r1 > 0)
	{
		i->p = v_add(ray->origin, v_scale(ray->dir, i->r1));
		if (v_dot(r->scenes->objects[index].dir, v_sub(i->p, r->scenes->objects[index].pos)) < 0 ||
			v_dot(r->scenes->objects[index].dir, v_sub(i->p, v_add(r->scenes->objects[index].pos,
			v_scale(r->scenes->objects[index].dir, r->scenes->objects[index].height)))) > 0)
			i->r1 = NAN;
	}
	if (i->r2 > 0)
	{
		i->p = v_add(ray->origin, v_scale(ray->dir, i->r2));
		if (v_dot(r->scenes->objects[index].dir, v_sub(i->p, r->scenes->objects[index].pos)) < 0 ||
			v_dot(r->scenes->objects[index].dir, v_sub(i->p, v_add(r->scenes->objects[index].pos,
			v_scale(r->scenes->objects[index].dir, r->scenes->objects[index].height)))) > 0)
			i->r2 = NAN;
	}
}
