#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_diffuse.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/05 15:18:12 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/19 13:45:44 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the diffuse value of the color at hitpoint.
*/

__device__
t_color	get_diffuse(t_scene *scene, t_ray *primary_ray,
		t_ray *shadow_ray, t_light *light)
{
	t_color	new_col;
	double	r2;

	r2 = (!v_isnan(light->pos)) ?
		pow(v_length(v_sub(shadow_ray->origin, light->pos)), 2.0) :
		1 / (4 * M_PI);
	new_col = vec_to_col(v_scale(v_mult(light->col, v_scale(scene->objects[primary_ray->hit_obj].col,
		light->intensity / (4 * M_PI * r2))), ft_clampd(v_dot(shadow_ray->dir,
		v_scale(primary_ray->nhit, primary_ray->n_dir)), 0, 1)));
	new_col = c_scale(new_col, scene->objects[primary_ray->hit_obj].kd);
	// new_col = c_clamp(new_col, 0, 255);
	return (new_col);
}
