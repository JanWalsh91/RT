#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_flares.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/23 10:18:02 by tgros             #+#    #+#             */
/*   Updated: 2017/06/05 16:28:26 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"

static int		get_light_count(t_light *light);
__global__
static void		draw_one_flare(t_light_flare_tools *tools, t_scene *scene,
				t_color *pixel_map);

void			get_flares(t_raytracing_tools *r)
{
	int					i;
	t_light_flare_tools *tools;
	size_t				shift;
	int					light_count;

	r->t = INFINITY;
	light_count = get_light_count(r->scene->lights);
	if (!light_count)
		return ;
	hipMalloc(&tools, sizeof(t_light_flare_tools) * light_count);
	init_light_flares_wrapper(light_count, r, tools);
	i = -1;
	shift = 0;
	while (++i < light_count)
	{
		dim3 blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
		dim3 gridSize = dim3(r->scene->res.x / BLOCK_DIM + 1,
			r->scene->res.y / BLOCK_DIM + 1);
		draw_one_flare<<<gridSize, blockSize>>>(tools + shift,
			r->d_scene, r->d_pixel_map);
		cuda_check_kernel_errors();
		shift++;
	}
}

static int		get_light_count(t_light *light)
{
	int				i;
	t_light			*l_ptr;

	i = 0;
	l_ptr = light;
	while (l_ptr)
	{
		++i;
		l_ptr = l_ptr->next;
	}
	return (i);
}

__global__
static void		draw_one_flare(t_light_flare_tools *tools, t_scene *scene,
				t_color *pixel_map)
{
	t_pt2	pix;
	float	rad;
	t_color	col;

	pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (pix.x < tools->pos.x - tools->max_rad ||
		pix.x > tools->pos.x + tools->max_rad ||
		pix.y < tools->pos.y - tools->max_rad ||
		pix.y > tools->pos.y + tools->max_rad ||
		pix.x < 0 || pix.x >= scene->res.x || pix.y < 0 ||
		pix.y >= scene->res.y || !tools->is_valid)
		return ;
	rad = sqrt(powf((tools->pos.x - pix.x), 2) +
		powf((tools->pos.y - pix.y), 2));
	col = pixel_map[pix.y * scene->res.x + pix.x];
	col = c_add(col, c_scale(vec_to_col(tools->light->col),
		((tools->max_rad - rad) /
		(tools->max_rad * rad)) * tools->light->kflare));
	pixel_map[pix.y * scene->res.x + pix.x] = col;
}
