#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_flares.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/23 10:18:02 by tgros             #+#    #+#             */
/*   Updated: 2017/05/24 14:53:59 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"


int		get_light_count(t_light *light)
{
	int 	i;
	t_light	*l_ptr;

	i = 0;
	l_ptr = light;
	while (l_ptr)
	{
		++i;
		l_ptr = l_ptr->next;
	}
	return (i);
}

__device__
void	update_light_ltw(t_light_flare_tools *tools, t_vec3 camera_dir)
{
	t_vec3	forward;
	t_vec3	right;
	t_vec3	up;

	forward = v_norm(camera_dir);
	if (v_dot(forward, v_new(0, 1, 0)) > 0.9999 ||
		v_dot(forward, v_new(0, 1, 0)) < -0.9999)
		right = v_new(1, 0, 0);
	else
		right = v_norm(v_cross(v_new(0, 1, 0), forward));
	up = v_norm(v_cross(forward, right));
	m_new_identity(&tools->ltw);
	tools->ltw[0][0] = right.x;
	tools->ltw[0][1] = right.y;
	tools->ltw[0][2] = right.z;
	tools->ltw[1][0] = up.x;
	tools->ltw[1][1] = up.y;
	tools->ltw[1][2] = up.z;
	tools->ltw[2][0] = forward.x;
	tools->ltw[2][1] = forward.y;
	tools->ltw[2][2] = forward.z;
	// tools->ltw[3][0] = tools->pos.x;
	// tools->ltw[3][1] = tools->pos.y;
	// tools->ltw[3][2] = tools->pos.z;
}
__device__
void	d_swap(float *f1, float *f2)
{
	float tmp;
	tmp = *f1;
	*f1 = *f2;
	*f2 = tmp;
}

__device__
void	invert_matrix(t_matrix m, t_matrix *invert)
{ 
    t_matrix mat;
    for (unsigned column = 0; column < 4; ++column) { 
        // Swap row in case our pivot point is not working
        if (m[column][column] == 0) { 
            unsigned big = column; 
            for (unsigned row = 0; row < 4; ++row) 
                if (fabs(m[row][column]) > fabs(m[big][column])) big = row; 
            // Print this is a singular matrix, return identity ?
            if (big == column) ; 
            // Swap rows                               
            else for (unsigned j = 0; j < 4; ++j) { 
                d_swap(&m[column][j], &m[big][j]); 
                d_swap(&mat[column][j], &mat[big][j]); 
            } 
        } 
        // Set each row in the column to 0  
        for (unsigned row = 0; row < 4; ++row) { 
            if (row != column) { 
                float coeff = m[row][column] / m[column][column]; 
                if (coeff != 0) { 
                    for (unsigned j = 0; j < 4; ++j) { 
                        m[row][j] -= coeff * m[column][j]; 
                        mat[row][j] -= coeff * mat[column][j]; 
                    } 
                    // Set the element to 0 for safety
                    m[row][column] = 0; 
                } 
            } 
        } 
    } 
    // Set each element of the diagonal to 1
    for (unsigned row = 0; row < 4; ++row) { 
        for (unsigned column = 0; column < 4; ++column) { 
            mat[row][column] /= m[row][row]; 
        } 
    }
	invert = &mat;
} 

__device__
bool	get_view_pane_intersection(t_ray *ray, t_camera *cam)
{
	float d1;
	t_vec3	v1;
	float r1;

	d1 = v_dot(v_scale(cam->dir, -1), ray->dir);
	if (d1 > -1e-20)
		return (false);
	v1 = v_sub(v_add(cam->pos, cam->dir), cam->pos);
	r1 = v_dot(v1, v_scale(cam->dir, -1)) / d1;
	if (r1 < 0)
		return (false);
	ray->t = r1;
	printf("T: %f\n", ray->t);
	return (true);
}

__device__
t_pt2	get_pane_coords(t_ray *ray, t_camera *cam, t_scene *scene)
{
	t_pt2	coord;
	t_vec3	ortho_x;
	t_vec3	ortho_y;
	t_vec3	proj_dir;
	float	dot_norm;

	proj_dir = v_sub(ray->dir, v_scale(cam->dir, -1));
	dot_norm = v_dot(v_new(0, 1, 0), v_scale(cam->dir, -1)); 
	ortho_x = v_cross(v_new(0, 1, 0), v_scale(cam->dir, -1));
	ortho_y = v_cross(v_scale(cam->dir, -1), ortho_x);
	ortho_x = v_norm(ortho_x);
	ortho_y = v_norm(ortho_y);
	ray->hit = v_add(ray->origin, v_scale(ray->dir, ray->t));
	coord.x = (int)(v_dot(v_scale(ortho_x, -1), proj_dir) * scene->res.x * 0.5 / scene->image_aspect_ratio / cam->scale + scene->res.x * 0.5);
	coord.y = (int)(v_dot(v_scale(ortho_y, -1), proj_dir) * scene->res.y * 0.5 / cam->scale + scene->res.y / 2);
	// printf("coords: [%d, %d]\n", coord.x, coord.y);
	return (coord);
}

__global__
void	init_light_flares(t_scene *scene, t_light_flare_tools *tools)
{
	t_raytracing_tools	r;
	t_ray				ray;
	int					id;
	int					y;

	id = blockIdx.x;
	r.scene = scene;
	ray.t = INFINITY;
	ray.type = R_PRIMARY;
	r.t = INFINITY;
	tools[id].is_valid = 1;
	ray.origin = scene->cameras->pos;
	ray.dir = v_norm(v_sub(scene->lights[id].pos, scene->cameras->pos));
	if (!get_view_pane_intersection(&ray, scene->cameras))
	{
		tools[id].is_valid = 0;
		return ;
	}
	tools[id].pos = get_pane_coords(&ray, scene->cameras, scene);
	if (tools[id].pos.x < 0 || tools[id].pos.x >= scene->res.x || tools[id].pos.y < 0 || tools[id].pos.y >= scene->res.y)
	{
		tools[id].is_valid = 0;
		return ;
	}
	tools[id].dist = v_length(v_sub(scene->lights[id].pos, ray.origin));
	tools[id].max_rad = powf(scene->lights[id].intensity, 4) / (sqrtf(tools[id].dist));
	tools[id].light = &scene->lights[id];
	tools[id].light->kflare = 1.0;
	if (tools[id].pos.x >= 0 && tools[id].pos.y >= 0 && tools[id].pos.x < scene->res.x && tools[id].pos.y < scene->res.y)
	{
		y = -1;
		while (scene->objects[++y].type != T_INVALID_TOKEN)
		{
			if (intersects(&r, &ray, y) &&
				ray.t < tools[id].dist && ray.t > 0.0)
			{
				tools[id].is_valid = 0;
				return ;
			}
		}
	}
	else
		tools[id].is_valid = 0;
}

__global__
void	draw_one_flare(t_light_flare_tools *tools, t_scene *scene, t_color *pixel_map)
{
	t_pt2	pix;
	float	rad;
	t_color	col;

	pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (pix.x < tools->pos.x - tools->max_rad || pix.x > tools->pos.x + tools->max_rad ||
		pix.y < tools->pos.y - tools->max_rad || pix.y > tools->pos.y + tools->max_rad ||
		pix.x < 0 || pix.x >= scene->res.x || pix.y < 0 || pix.y >= scene->res.y || !tools->is_valid)
			return ;
	rad = sqrt(powf((tools->pos.x - pix.x), 2) + powf((tools->pos.y - pix.y), 2));
	col = pixel_map[pix.y * scene->res.x + pix.x];
	col = c_add(col, c_scale(vec_to_col(tools->light->col), (1 / powf(rad, 2) * (tools->max_rad - rad)) * tools->light->kflare));
	pixel_map[pix.y * scene->res.x + pix.x] = col;
}

void	add_lens_flare(t_raytracing_tools *r, t_color *pixel_map)
{
	int					i;
	t_light_flare_tools *tools;
	size_t				shift;

	r->t = INFINITY;
	int light_count = get_light_count(r->scene->lights);
	hipMalloc(&tools, sizeof(t_light_flare_tools) * light_count);
	init_light_flares<<<light_count, 1>>>(r->d_scene, tools);
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("1 Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("1 Async kernel error: %s\n", hipGetErrorString(errAsync));
	if (errSync != hipSuccess || errAsync != hipSuccess)
		exit(-1);
	i = -1;
	shift = 0;
	while (++i < light_count)
	{
		dim3 blockSize 	= dim3(BLOCK_DIM, BLOCK_DIM, 1);
		dim3 gridSize	= dim3(r->scene->res.x / BLOCK_DIM + 1, r->scene->res.y / BLOCK_DIM + 1);
		draw_one_flare<<<gridSize, blockSize>>>(tools + shift, r->d_scene, r->d_pixel_map);
		errSync  = hipGetLastError();
		errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("2 Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("2 Async kernel error: %s\n", hipGetErrorString(errAsync));
		if (errSync != hipSuccess || errAsync != hipSuccess)
			exit(-1);
		shift++;
	}
}

void 	lens_flare_wrapper(t_raytracing_tools *r)
{
	printf("lens_flare_wrapper\n");
	add_lens_flare(r, r->d_pixel_map);
	printf("end lens_flare_wrapper\n");
	hipError_t errAsync = hipDeviceSynchronize();
	if (errAsync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errAsync));
	printf("end lens_flare_wrapper");
}
