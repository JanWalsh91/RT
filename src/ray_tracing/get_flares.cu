#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_flares.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/23 10:18:02 by tgros             #+#    #+#             */
/*   Updated: 2017/05/25 10:24:48 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"


int		get_light_count(t_light *light)
{
	int 	i;
	t_light	*l_ptr;

	i = 0;
	l_ptr = light;
	while (l_ptr)
	{
		++i;
		l_ptr = l_ptr->next;
	}
	return (i);
}

__device__
bool	get_view_pane_intersection(t_ray *ray, t_camera *cam)
{
	float d1;
	t_vec3	v1;
	float r1;

	d1 = v_dot(v_scale(cam->dir, -1), ray->dir);
	if (d1 > -1e-20)
		return (false);
	v1 = v_sub(v_add(cam->pos, cam->dir), cam->pos);
	r1 = v_dot(v1, v_scale(cam->dir, -1)) / d1;
	if (r1 < 0)
		return (false);
	ray->t = r1;
	printf("T: %f\n", ray->t);
	return (true);
}

__device__
t_pt2	get_pane_coords(t_ray *ray, t_camera *cam, t_scene *scene)
{
	t_pt2	coord;
	t_vec3	ortho_x;
	t_vec3	ortho_y;
	t_vec3	proj_dir;

	proj_dir = v_sub(ray->dir, v_scale(cam->dir, -1));
	ortho_x = v_cross(v_new(0, 1, 0), v_scale(cam->dir, -1));
	ortho_y = v_cross(v_scale(cam->dir, -1), ortho_x);
	ortho_x = v_norm(ortho_x);
	ortho_y = v_norm(ortho_y);
	ray->hit = v_add(ray->origin, v_scale(ray->dir, ray->t));
	coord.x = (int)(v_dot(v_scale(ortho_x, -1), proj_dir) * scene->res.x * 0.5 / scene->image_aspect_ratio / cam->scale + scene->res.x * 0.5);
	coord.y = (int)(v_dot(v_scale(ortho_y, -1), proj_dir) * scene->res.y * 0.5 / cam->scale + scene->res.y / 2);
	// printf("coords: [%d, %d]\n", coord.x, coord.y);
	return (coord);
}

__global__
void	init_light_flares(t_scene *scene, t_light_flare_tools *tools)
{
	t_raytracing_tools	r;
	t_ray				ray;
	int					id;
	int					y;

	id = blockIdx.x;
	r.scene = scene;
	ray.t = INFINITY;
	ray.type = R_PRIMARY;
	r.t = INFINITY;
	tools[id].is_valid = 1;
	ray.origin = scene->cameras->pos;
	ray.dir = v_norm(v_sub(scene->lights[id].pos, scene->cameras->pos));
	if (!get_view_pane_intersection(&ray, scene->cameras))
	{
		tools[id].is_valid = 0;
		return ;
	}
	tools[id].pos = get_pane_coords(&ray, scene->cameras, scene);
	if (tools[id].pos.x < 0 || tools[id].pos.x >= scene->res.x || tools[id].pos.y < 0 || tools[id].pos.y >= scene->res.y)
	{
		tools[id].is_valid = 0;
		return ;
	}
	tools[id].dist = v_length(v_sub(scene->lights[id].pos, ray.origin));
	tools[id].max_rad = powf(scene->lights[id].intensity, 3) / tools[id].dist;
	tools[id].light = &scene->lights[id];
	if (tools[id].pos.x >= 0 && tools[id].pos.y >= 0 && tools[id].pos.x < scene->res.x && tools[id].pos.y < scene->res.y)
	{
		y = -1;
		while (scene->objects[++y].type != T_INVALID_TOKEN)
		{
			if (intersects(&r, &ray, y) &&
				ray.t < tools[id].dist && ray.t > 0.0)
			{
				tools[id].is_valid = 0;
				return ;
			}
		}
	}
	else
		tools[id].is_valid = 0;
}

__global__
void	draw_one_flare(t_light_flare_tools *tools, t_scene *scene, t_color *pixel_map)
{
	t_pt2	pix;
	float	rad;
	t_color	col;

	pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (pix.x < tools->pos.x - tools->max_rad || pix.x > tools->pos.x + tools->max_rad ||
		pix.y < tools->pos.y - tools->max_rad || pix.y > tools->pos.y + tools->max_rad ||
		pix.x < 0 || pix.x >= scene->res.x || pix.y < 0 || pix.y >= scene->res.y || !tools->is_valid)
			return ;
	rad = sqrt(powf((tools->pos.x - pix.x), 2) + powf((tools->pos.y - pix.y), 2));
	col = pixel_map[pix.y * scene->res.x + pix.x];
	col = c_add(col, c_scale(vec_to_col(tools->light->col), ((tools->max_rad - rad) / (tools->max_rad * rad)) * tools->light->kflare));
	pixel_map[pix.y * scene->res.x + pix.x] = col;
}

void	add_lens_flare(t_raytracing_tools *r, t_color *pixel_map)
{
	int					i;
	t_light_flare_tools *tools;
	size_t				shift;

	r->t = INFINITY;
	int light_count = get_light_count(r->scene->lights);
	hipMalloc(&tools, sizeof(t_light_flare_tools) * light_count);
	init_light_flares<<<light_count, 1>>>(r->d_scene, tools);
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("1 Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("1 Async kernel error: %s\n", hipGetErrorString(errAsync));
	if (errSync != hipSuccess || errAsync != hipSuccess)
		exit(-1);
	i = -1;
	shift = 0;
	while (++i < light_count)
	{
		dim3 blockSize 	= dim3(BLOCK_DIM, BLOCK_DIM, 1);
		dim3 gridSize	= dim3(r->scene->res.x / BLOCK_DIM + 1, r->scene->res.y / BLOCK_DIM + 1);
		draw_one_flare<<<gridSize, blockSize>>>(tools + shift, r->d_scene, r->d_pixel_map);
		errSync  = hipGetLastError();
		errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("2 Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("2 Async kernel error: %s\n", hipGetErrorString(errAsync));
		if (errSync != hipSuccess || errAsync != hipSuccess)
			exit(-1);
		shift++;
	}
}

void 	lens_flare_wrapper(t_raytracing_tools *r)
{
	printf("lens_flare_wrapper\n");
	add_lens_flare(r, r->d_pixel_map);
	printf("end lens_flare_wrapper\n");
	hipError_t errAsync = hipDeviceSynchronize();
	if (errAsync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errAsync));
	printf("end lens_flare_wrapper");
}
