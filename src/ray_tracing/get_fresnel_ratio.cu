#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_fresnel_ratio.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 14:25:09 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/01 12:37:47 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the fresnel ratio.
*/

__device__
static float	get_fresnel_ratio2(float cosi, float etai, float etat, float sint);

__device__
float			get_fresnel_ratio(t_vec3 ray_dir, t_vec3 normal, float ior)
{
	float	cosi;
	float	etai;
	float	etat;
	float	sint;

	cosi = ft_clampf(v_dot(ray_dir, normal), -1, 1);
	etai = 1;
	etat = ior;
	if (cosi  > 0)
		ft_swapf(&etai, &etat);
	sint = etai / etat * sqrtf(1 - cosi * cosi > 0.0 ? 1 - cosi * cosi : 0.0);
	if (sint >= 1) 
		return (1);
	else
		return (get_fresnel_ratio2(cosi, etai, etat, sint));
	
}

__device__
static float	get_fresnel_ratio2(float cosi, float etai, float etat, float sint)
{
	float	cost;
	float 	Rs;
	float	Rp;

	cost = sqrtf((1 - sint * sint > 0 ? 1 - sint * sint : 0.0)); 
    cosi = cosi < 0 ? -cosi : cosi; 
    Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost)); 
    Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost)); 
	return ((Rs * Rs + Rp * Rp) / 2); 
}