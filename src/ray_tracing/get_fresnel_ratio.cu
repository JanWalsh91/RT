#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_fresnel_ratio.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 14:25:09 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/15 15:28:37 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the fresnel ratio.
*/

__device__
static double	get_fresnel_ratio2(double cosi, double etai, double etat, double sint);

__device__
double			get_fresnel_ratio(t_vec3 ray_dir, t_vec3 normal, double ior)
{
	double	cosi;
	double	etai;
	double	etat;
	double	sint;

	cosi = ft_clampd(v_dot(ray_dir, normal), -1, 1);
	etai = 1;
	etat = ior;
	if (cosi  > 0)
		ft_swapd(&etai, &etat);
	sint = etai / etat * __dsqrt_rn(1 - cosi * cosi > 0.0 ? 1 - cosi * cosi > 0.0 : 0.0));
	if (sint >= 1) 
		return (1);
	else
		return (get_fresnel_ratio2(cosi, etai, etat, sint));
	
}

__device__
static double	get_fresnel_ratio2(double cosi, double etai, double etat, double sint)
{
	double	cost;
	double 	Rs;
	double	Rp;

	cost = __dsqrt_rn((1 - sint * sint > 0 ? 1 - sint * sint : 0.0)); 
    cosi = cosi < 0 ? -cosi : cosi; 
    Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost)); 
    Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost)); 
	return ((Rs * Rs + Rp * Rp) / 2); 
}