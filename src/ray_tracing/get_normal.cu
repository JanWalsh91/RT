#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_normal.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 16:05:39 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/02 17:28:17 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"

static void	get_sphere_normal(t_ray *ray, t_object *obj);
static void	get_plane_normal(t_ray *ray, t_object *obj);
static void	get_cylinder_normal(t_ray *ray, t_object *obj);
static void	get_cone_normal(t_ray *ray, t_object *obj);

/*
** Updates the normal at hitpoint (nhit and n_dir).
*/

__device__
void		get_normal(t_ray *ray, t_object *obj)
{
	if (ray->hit_type == T_SPHERE)
		get_sphere_normal(ray, obj);
	if (ray->hit_type == T_PLANE || ray->hit_type == T_DISK)
		get_plane_normal(ray, obj);
	if (ray->hit_type == T_CYLINDER)
		get_cylinder_normal(ray, obj);
	if (obj->type == T_CONE)
		get_cone_normal(ray, obj);
	ray->n_dir = v_dot(ray->nhit, ray->dir) < 0 ? 1 : -1;
}

__device__
static void	get_sphere_normal(t_ray *ray, t_object *obj)
{
	ray->nhit = (v_sub(ray->hit, obj->pos));
	ray->nhit = v_norm(ray->nhit);
}

__device__
static void	get_plane_normal(t_ray *ray, t_object *obj)
{
	ray->nhit = v_norm(obj->dir);
}

__device__
static void	get_cylinder_normal(t_ray *ray, t_object *obj)
{
	t_vec3	x;
	double	m;

	x = v_sub(ray->origin, obj->pos);
	m = v_dot(ray->dir, v_scale(obj->dir, ray->t)) + v_dot(x, obj->dir);
	x = v_add(obj->pos, v_scale(obj->dir, m));
	ray->nhit = v_norm(v_sub(ray->hit, x));
}

__device__
static void	get_cone_normal(t_ray *ray, t_object *obj)
{
	t_vec3	x;

	x = v_sub(ray->hit, obj->pos);
	ray->nhit = v_sub(x, v_scale(obj->dir, (v_length(x) / cos(obj->angle))));
	ray->nhit = v_norm(ray->nhit);
}
