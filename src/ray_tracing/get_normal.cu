#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_normal.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 16:05:39 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/18 14:35:44 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

__device__
static void	get_sphere_normal(t_ray *ray, t_object *obj);
__device__
static void	get_plane_normal(t_ray *ray, t_object *obj);
__device__
static void	get_cylinder_normal(t_ray *ray, t_object *obj);
__device__
static void	get_cone_normal(t_ray *ray, t_object *obj);

__device__
t_vec3		get_normal_at_normal_map(t_object *obj, t_ray *ray)
{
	t_vec3	tangente;
	t_vec3	bitangente;
	t_vec3	color;
	t_vec3	res;
	t_pt2	coord;

	tangente = v_norm(v_cross(ray->dir, ray->nhit));
	bitangente = v_norm(v_cross(ray->nhit, tangente));
	coord = get_uv_coord(obj, ray, &obj->normal_map_dim);
	color = col_to_vec(obj->normal_map[obj->normal_map_dim.x * coord.y + coord.x]);
	color.x = (color.x / 255.0f) * 2.0 - 1;
	color.y = (color.y / 255.0f) * 2.0 - 1;
	color.z = (color.z / 255.0f) * 2.0 - 1;
	res.x = (tangente.x * color.x + bitangente.x * color.y + ray->nhit.x * color.z);
	res.y = (tangente.y * color.x + bitangente.y * color.y + ray->nhit.y * color.z);
	res.z = (tangente.z * color.x + bitangente.z * color.y + ray->nhit.z * color.z);
	return (res);
}

/*
** Updates the normal at hitpoint (nhit and n_dir).
*/

__device__
void		get_normal(t_ray *ray, t_object *obj)
{
	// if (ray->type > 1)
	// 	printf("photon: get_normal. nhit: [%f, %f, %f]\n", ray->nhit.x, ray->nhit.y, ray->nhit.z);
	if (ray->hit_type == T_SPHERE)
		get_sphere_normal(ray, obj);
	if (ray->hit_type == T_PLANE || ray->hit_type == T_DISK)
		get_plane_normal(ray, obj);
	if (ray->hit_type == T_CYLINDER)
		get_cylinder_normal(ray, obj);
	if (obj->type == T_CONE)
		get_cone_normal(ray, obj);
	ray->n_dir = v_dot(ray->nhit, ray->dir) < 0 ? 1 : -1;
}

__device__
static void	get_sphere_normal(t_ray *ray, t_object *obj)
{
	ray->nhit = (v_sub(ray->hit, obj->pos));
	ray->nhit = v_norm(ray->nhit);
	if (obj->normal_map)
		ray->nhit = get_normal_at_normal_map(obj, ray);
}

__device__
static void	get_plane_normal(t_ray *ray, t_object *obj)
{
	ray->nhit = v_norm(obj->dir);
	if (obj->normal_map)
		ray->nhit = get_normal_at_normal_map(obj, ray);
}

__device__
static void	get_cylinder_normal(t_ray *ray, t_object *obj)
{
	t_vec3	x;
	float	m;

	x = v_sub(ray->origin, obj->pos);
	m = v_dot(ray->dir, v_scale(obj->dir, ray->t)) + v_dot(x, obj->dir);
	x = v_add(obj->pos, v_scale(obj->dir, m));
	ray->nhit = v_norm(v_sub(ray->hit, x));
	if (obj->normal_map)
		ray->nhit = get_normal_at_normal_map(obj, ray);
}

__device__
static void	get_cone_normal(t_ray *ray, t_object *obj)
{
	t_vec3	x;

	x = v_sub(ray->hit, obj->pos);
	ray->nhit = v_sub(x, v_scale(obj->dir, (v_length(x) / cos(obj->angle))));
	ray->nhit = v_norm(ray->nhit);
	if (obj->normal_map)
		ray->nhit = get_normal_at_normal_map(obj, ray);
}
