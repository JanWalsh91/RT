#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_obj_intersection.cu                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:26:41 by talemari          #+#    #+#             */
/*   Updated: 2017/05/18 15:28:58 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../../inc/objparser.h"
#include <stdio.h>
/*
** Checks for a object-sphere intersection.
*/

__device__
static void		print_triangles_dev(t_obj *obj)
{
	t_obj_triangle		*tmp2;
	t_list				*tmp;

	tmp = obj->triangle;	
	printf("Printing triangles\n");
	while (tmp != NULL)
	{
		printf("-----TRIANGLE----- %p\n", tmp);
		tmp2 = (t_obj_triangle *)tmp->content;
		printf("v1 = %f, %f, %f\n", obj->vertex[tmp2->v.x].x, obj->vertex[tmp2->v.x].y, obj->vertex[tmp2->v.x].z);
		printf("v2 = %f, %f, %f\n", obj->vertex[tmp2->v.y].x, obj->vertex[tmp2->v.y].y, obj->vertex[tmp2->v.y].z);
		printf("v3 = %f, %f, %f\n", obj->vertex[tmp2->v.z].x, obj->vertex[tmp2->v.z].y, obj->vertex[tmp2->v.z].z);
		tmp = tmp->next;
	}
}

__device__
bool	get_sphere_intersection_obj(t_raytracing_tools *r, t_ray *ray, int index)
{
	t_intersection_tools	i;

	i.v1 = v_sub(ray->origin, r->scene->objects[index].pos);
	i.q.x = v_dot(ray->dir, ray->dir);
	i.q.y = 2 * v_dot(i.v1, ray->dir);
	i.q.z = v_dot(i.v1, i.v1) - pow(r->scene->objects[index].rad, 2);
	if (!solve_quadratic(i.q, &i.r1, &i.r2))
		return (false);
	if (i.r2 < i.r1)
		ft_swapf(&i.r1, &i.r2);
	(i.r1 < 0) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0)
		return (false);
	return (true);
}

__device__
bool	get_obj_intersection(t_raytracing_tools *r, t_ray *ray, int index)
{
	t_triangle		triangle;
	t_obj_triangle	*lst_triangle;
	t_obj			*o;
	t_list			*tmp;
	float			t;

	// if (r->idx != 0)
	// 	return (0);
	if (!(get_sphere_intersection_obj(r, ray, index)))
		return (false);
	t = INFINITY;
	o = r->scene->objects[index].obj;
	tmp = o->triangle;
	//printf("Check\n");
	//if (r->pix.x == 0 && r->pix.y == 0)
		//print_triangles_dev(o);
	while (tmp)
	{
		//printf("Triangle check\n");
	
		lst_triangle = (t_obj_triangle *)tmp->content;
		triangle.v1 = o->vertex[lst_triangle->v.x];
		triangle.v2 = o->vertex[lst_triangle->v.y];
		triangle.v3 = o->vertex[lst_triangle->v.z];
		if (get_triangle_intersection(r, &triangle, ray, index))
		{
			if (ray->t < t)
			{
				t = ray->t;
			}
		}
		//if (r->pix.x == 600 && r->pix.y < 200)
			//printf("addr %p\n", o->triangle->next);
		tmp = tmp->next;
	}
	if (t != INFINITY)
	{
		ray->t = t;
		return (true);
	}
	// if (r->idx == 0)
	// printf("Exit obj_intersection\n");
	return (false);
}
