#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_obj_intersection.cu                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:26:41 by talemari          #+#    #+#             */
/*   Updated: 2017/05/18 15:28:58 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../../inc/objparser.h"
/*
** Checks for a object-sphere intersection.
*/

__device__
bool	get_obj_intersection(t_obj *o, t_ray *ray, int index, t_vec3 *norm)
{
	t_triangle		triangle;
	t_obj_triangle	*lst_triangle;

	while (o->triangle)
	{
		lst_triangle = (t_obj_triangle *)o->triangle->content;
		triangle.v1 = o->vertex[lst_triangle->v.x];
		triangle.v2 = o->vertex[lst_triangle->v.y];
		triangle.v3 = o->vertex[lst_triangle->v.z];
		if (get_triangle_intersection(&triangle, ray, index, norm))
			return (true);
		o->triangle = o->triangle->next;
	}
	return (false);
}
