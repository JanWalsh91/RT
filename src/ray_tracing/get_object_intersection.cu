#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_obj_intersection.cu                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:26:41 by talemari          #+#    #+#             */
/*   Updated: 2017/05/18 15:28:58 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../../inc/objparser.h"
#include <stdio.h>
/*
** Checks for a object-sphere intersection.
*/

__device__
static void		print_triangles_dev(t_obj *obj)
{
	t_obj_triangle		*tmp2;
	t_list				*tmp;

	tmp = obj->triangle;	
	printf("Printing triangles\n");
	while (tmp != NULL)
	{
		printf("-----TRIANGLE----- %p\n", tmp);
		tmp2 = (t_obj_triangle *)tmp->content;
		printf("v1 = %f, %f, %f\n", obj->vertex[tmp2->v.x].x, obj->vertex[tmp2->v.x].y, obj->vertex[tmp2->v.x].z);
		printf("v2 = %f, %f, %f\n", obj->vertex[tmp2->v.y].x, obj->vertex[tmp2->v.y].y, obj->vertex[tmp2->v.y].z);
		printf("v3 = %f, %f, %f\n", obj->vertex[tmp2->v.z].x, obj->vertex[tmp2->v.z].y, obj->vertex[tmp2->v.z].z);
		tmp = tmp->next;
	}
}

__device__
bool	get_sphere_intersection_obj(t_raytracing_tools *r, t_ray *ray, int index)
{
	t_intersection_tools	i;

	i.v1 = v_sub(ray->origin, r->scene->objects[index].pos);
	i.q.x = v_dot(ray->dir, ray->dir);
	i.q.y = 2 * v_dot(i.v1, ray->dir);
	i.q.z = v_dot(i.v1, i.v1) - pow(r->scene->objects[index].rad, 2);
	if (!solve_quadratic(i.q, &i.r1, &i.r2))
		return (false);
	if (i.r2 < i.r1)
		ft_swapf(&i.r1, &i.r2);
	(i.r1 < 0) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0)
		return (false);
	return (true);
}

__device__
bool	get_triangle_intersection_obj(t_raytracing_tools *r, t_triangle *t, t_ray *ray)
{
	t_intersection_tools	i;
	float					tmp;

	i.p = v_sub(t->v2, t->v1);
	i.q = v_sub(t->v3, t->v1);
	i.v1 = v_cross(ray->dir, i.q);
	i.d1 = v_dot(i.p, i.v1);
	if (i.d1 < 1e-8 && i.d1 > -1e-8)
		return (false);
	i.d2 = 1 / i.d1;
	i.v2 = v_sub(ray->origin, t->v1);
	i.r1 = v_dot(i.v1, i.v2) * i.d2;
	if (i.r1 < 0 || i.r1 > 1)
		return (false);
	i.v3 = v_cross(i.v2, i.p);
	i.r2 = v_dot(ray->dir, i.v3) * i.d2;
	if (i.r2 < 0 || i.r1 + i.r2 > 1)
		return (false);
	tmp = v_dot(i.q, i.v3) * i.d2;
	if (tmp < ray->t)
	{
		ray->t = tmp;
		if (v_dot(ray->nhit, ray->dir) > 0)
			ray->n_dir = -1;
	}
	return (true);
}

__device__
bool	get_obj_intersection(t_raytracing_tools *r, t_ray *ray, int index)
{
	t_triangle		triangle;
	t_obj_triangle	*lst_triangle;
	t_obj			*o;
	t_list			*tmp;
	float			t;
	float			raytmp;

	// if (r->idx != 0)
	// 	return (0);
	raytmp = ray->t;
	printf("obj inter\n");
	if (!(get_sphere_intersection_obj(r, ray, index)))
		return (false);
	t = INFINITY;
	ray->t = raytmp;
	o = r->scene->objects[index].obj;
	tmp = o->triangle;
	//printf("Check\n");
	//if (r->pix.x == 0 && r->pix.y == 0)
		//print_triangles_dev(o);
	while (tmp)
	{
		//printf("Triangle check\n");
	
		lst_triangle = (t_obj_triangle *)tmp->content;
		triangle.v1 = o->vertex[lst_triangle->v.x];
		triangle.v2 = o->vertex[lst_triangle->v.y];
		triangle.v3 = o->vertex[lst_triangle->v.z];
		if (get_triangle_intersection_obj(r, &triangle, ray))
		{
			if (ray->t < t)
			{
				t = ray->t;
				ray->nhit = v_cross(v_sub(triangle.v2, triangle.v1),
						v_sub(triangle.v3, triangle.v1));
			}
		}
		//if (r->pix.x == 600 && r->pix.y < 200)
			//printf("addr %p\n", o->triangle->next);
		tmp = tmp->next;
	}
	if (t != INFINITY)
	{
		ray->t = t;
		ray->hit_obj = index;
		ray->hit_type = T_OBJ;
		return (true);
	}
	// if (r->idx == 0)
	// printf("Exit obj_intersection\n");
	return (false);
}
