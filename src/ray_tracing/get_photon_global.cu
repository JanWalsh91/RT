#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_photon_global.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/16 13:34:30 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/26 21:41:17 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "photon_mapping.h"

__device__
void			knn_search(t_vec3 *hit_pos, t_kd_tree *photon_kd_tree, t_selected_photon *photons, int dim, int k);
__device__
static float	get_dist_by_dim(t_vec3 *hit_pos, t_vec3 photon_pos, int dim);
__device__
static float	get_dist2(t_vec3 hit_pos, t_vec3 photon_pos);
__device__
static int		add_to_list(t_selected_photon *photons, t_kd_tree *photon, float dist2, int k);
__device__
static void		push_list(t_selected_photon *list, t_selected_photon tmp, int i, int k);
__device__
static int		more_points_can_be_found(float dist, t_selected_photon *list, int k);
__device__
static void		print_selected_photons(t_selected_photon *list, int k);
__device__
float			get_max_radius(t_vec3 *hit, t_selected_photon *photons);

/*
** Gathers k photons from the photon map closest to the hitpoint and averages a color
** value based on incoming radiosity.
*/

__device__
t_color	get_photon_global(t_raytracing_tools *r, t_ray *ray)
{
	t_selected_photon *photons;
	int dim = 0;
	int k = 10;
	int i = -1;
	float r2;
	//malloc list of photons based on r->settings->k + 1

	// printf("get_photon_global\n");
	// printf("-----%p\n", r->scene->photon_kd_tree);
	// printf("in kernel selected_photons: [%p]\n", r->scene->selected_photons);
	// printf("in kernel selected_photons: [%p]\n", r->scene->selected_photons[1]);
	if (r->idx == 0)
		C(1)
	// __syncthreads();
	photons = r->scene->selected_photons + r->idx;
	// printf("-----%p\n", r->scene->se	lected_photons);
	// printf("-----%p, pix num: %d, idx: %d\n", r->scene->selected_photons[r->idx], r->scene->res.x * r->scene->res.y, r->idx);
	if (r->idx == 0)
		C(2)
	// __syncthreads();
	while (++i < k)
	{
		// printf("-----%p\n", photons);
		photons[i].photon = NULL;
	}
	if (r->idx == 0)
		C(3)
	// __syncthreads();
	// printf("hit pos: %p [%f, %f, %f]\n", &ray->hit, ray->hit.x, ray->hit.y, ray->hit.z);

	knn_search(&ray->hit, r->scene->photon_kd_tree, photons, dim, k);

	// printf("IM GONNA PRINT SOME PHOTONS\n");
	// print_selected_photons(photons, k);
	t_vec3 sum;
	i = -1;
	sum = v_new(0, 0, 0);
	r2 = get_max_radius(&ray->hit, photons);
	if (r->idx == 0)
		C(4)
	while (++i < k && photons[i].photon)
	{
		sum = v_add(sum, v_scale(v_mult(col_to_vec(photons[i].photon->col), v_scale(col_to_vec(photons[i].photon->col), 1.0 / (0.001 + 4 * M_PI * (r2 - photons[i].dist2)))),
		ft_clampf(v_dot(photons[i].photon->n, v_scale(ray->nhit, ray->n_dir)), 0, 1)));

		// sum.x += photons[i].photon->col.r;
		// sum.y += photons[i].photon->col.g;
		// sum.z += photons[i].photon->col.b;
		if (r->idx == 1)
		printf("adding photon: %d, %d, %d\n", photons[i].photon->col.r, photons[i].photon->col.g, photons[i].photon->col.b);
	}
	if (r->idx == 0)
		C(5)
	if (r->idx == 1)
	{
		// printf("photon count: %d\n", r->scene->photon_count);
		printf("sum result: [%f, %f, %f] photoncount: %d\n", sum.x, sum.y, sum.z, r->scene->photon_count);
	}
	if (r->scene->photon_count)
		sum = v_scale(sum, 1.0 / 10);
	if (r->idx == 1)
	{
		// printf("photon count: %d\n", r->scene->photon_count);
		printf("sum result: [%f, %f, %f]\n", sum.x, sum.y, sum.z);
	}
	__syncthreads();
	//Take sum of power of all photons corrected by Lambertian Shading, divide by number of photons shot
	//Convert to t_color and return.
	return (vec_to_col(sum));
}

__device__
float	get_max_radius(t_vec3 *hit, t_selected_photon *photons)
{
	int i;
	float r2;

	r2 = 0;
	i = -1;
	while (++i < 10 && photons[i].photon)
	{
		if (r2 < photons[i].dist2)
			r2 = photons[i].dist2;
	}
	return (r2);
}

/*
** knn = k Nearest Neighbor search
** Gathers k closest photons to the hit point.
*/

__device__
void		knn_search(t_vec3 *hit_pos, t_kd_tree *photon_kd_tree, t_selected_photon *photons, int dim, int k)
{
	int		dir;
	float	dist;

	// printf("knn_search\n");
	// print_selected_photons(photons, k);
	if (!photon_kd_tree)
	{
		// printf("no photon map\n");
		return ;
	}
	dim = dim == 3 ? 0 : dim;
	//compare current node.dim distance with pos.dim
	dist = get_dist_by_dim(hit_pos, photon_kd_tree->pos, dim);
	// printf("dist: %f\n", dist);
	dir = (dist < 0) ? 2 : 1;
	// printf("dir: %d\n", dir);
	//go either left or right (closest first)
	dir == 1 ? knn_search(hit_pos, photon_kd_tree->right, photons, dim + 1, k) :
				knn_search(hit_pos, photon_kd_tree->left, photons, dim + 1, k);
	// printf("hit pos: %p [%f, %f, %f]\n", hit_pos, hit_pos->x, hit_pos->y, hit_pos->z);
	// //check to see if you can add it to photon list
	add_to_list(photons, photon_kd_tree, get_dist2(*hit_pos, photon_kd_tree->pos), k);

	// //if k points have been found and the other branch cannot have points closer to any of the current bests
	// 	//do not go down other branch
	if (more_points_can_be_found(dist, photons, k))
		dir == 1 ? knn_search(hit_pos, photon_kd_tree->left, photons, dim + 1, k) :
			knn_search(hit_pos, photon_kd_tree->right, photons, dim + 1, k);
}

__device__
static float	get_dist_by_dim(t_vec3 *hit_pos, t_vec3 photon_pos, int dim)
{
	// printf("get_dist_by_dim\n");
	if (dim == 0)
		return (hit_pos->x - photon_pos.x);
	if (dim == 1)
		return (hit_pos->y - photon_pos.y);
	if (dim == 2)
		return (hit_pos->z - photon_pos.z);
	else
		return (INFINITY);
}

__device__
static float	get_dist2(t_vec3 hit_pos, t_vec3 photon_pos)
{
	// printf("get_dist2\n");
	float length;

	length = v_dot(hit_pos, photon_pos);
	length = length < 0 ? -length : length;
	return (length);
}

__device__
static int		add_to_list(t_selected_photon *photons, t_kd_tree *photon, float dist2, int k)
{
	// printf("add_to_list\n");
	int 				i;
	t_selected_photon	tmp;

	i = -1;
	while (++i <= k)
	{
		if (photons[i].photon == NULL || photons[i].dist2 > dist2)
		{
			tmp.photon = photon;
			tmp.dist2 = dist2;
			push_list((photons), tmp, i, k);
			return (1);
		}
	}
	return (0);
}

__device__
static void		push_list(t_selected_photon *list, t_selected_photon tmp, int i, int k)
{
	// printf("push_list\n");
	t_selected_photon tmp2;
	

	//if at end of list or at NULL photon, store photon at last position.
	if (i == k - 1 || list[i].photon == NULL)
	{
		list[i] = tmp;
		return ;
	}
	//else save that photon
	tmp2 = list[i];
	//push that photon 
	push_list((list), tmp2, i + 1, k);
	//then store the previous photon 
	list[i].photon = tmp.photon;
	list[i].dist2 = tmp.dist2;
}

__device__
static int	more_points_can_be_found(float dist, t_selected_photon *list, int k)
{
	// printf("more_points_can_be_found ");
	int i;

	if (!list[k - 1].photon) //last photon does not exist. is list[k] the last photon?!
	{
		// printf("1\n");
		return (1);
	}
	i = -1;
	//check dist of all photons against dist from plane
	while (++i < k)
	{
		//if a photon in the selected photon list is farther from the hit point that the plane, there are more solutions
		if (dist * dist < list[i].dist2)
		{
			// printf("1\n");
			return (1);
		}
	}
	// printf("0\n");
	return (0);
}

__device__
static void	print_selected_photons(t_selected_photon *list, int k)
{
	int i = -1;
	while (++i < k && list && list[i].photon)
		printf("k: %d, selected photon: %p, [%f, %f, %f]\n",
		k, &list[i], list[i].photon->pos.x, list[i].photon->pos.y, list[i].photon->pos.z);
}