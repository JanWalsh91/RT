#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_plane_intersection.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:25:30 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/04 14:04:17 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Checks for a ray-plane intersection.
*/

__device__
bool	get_plane_intersection(t_raytracing_tools *r, t_ray *ray, int index)
{
	t_intersection_tools	i;

	i.n_dir = 1;
	if (v_dot(r->scene->objects[index].dir, ray->dir) > 0)
		i.n_dir = -1;
	i.d1 = v_dot(v_scale(r->scene->objects[index].dir, i.n_dir), ray->dir);
	if (i.d1 > -1e-20)
		return (false);
	i.v1 = v_sub(r->scene->objects[index].pos, ray->origin);
	i.r1 = v_dot(i.v1, v_scale(r->scene->objects[index].dir, i.n_dir)) / i.d1;
	if (i.r1 < 0)
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0;
	if (ray->type == R_PRIMARY && r->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_PLANE;
	}
	return (true);
}

__device__
t_pt2	get_uv_plane(t_object *obj, t_ray *ray)
{
	t_pt2	coord;
	t_vec3	ortho_x;
	t_vec3	ortho_y;
	t_vec3	vec;

	// vec.x = ray->nhit.z < 0.001 && ray->nhit.z > -0.001 ? ray->nhit.z : ray->nhit.y;
	// vec.y = 0;
	// vec.z = ray->nhit.x < 0.001 && ray->nhit.x > -0.001 ? -ray->nhit.y: -ray->nhit.x;

	// ortho_x = v_norm(vec);
	// ortho_y = v_cross(ray->nhit, ortho_x);

	// coord.x = abs((int)((v_dot(ortho_x, ray->hit)) * (obj->texture_dim.x)) % obj->texture_dim.x);
	// coord.y = abs((int)((v_dot(ortho_y, ray->hit)) * (obj->texture_dim.y)) % obj->texture_dim.y);

	// coord.x = (int)v_dot(ortho_x, ray->hit) % obj->texture_dim.x;
	// coord.y = (int)v_dot(ortho_y, ray->hit) % obj->texture_dim.y;



	// hit_center = v_scale(ray->nhit, -1);
	// coord.x = (0.5 + (atan2f(hit_center.z, hit_center.x) / (float)(2 * M_PI))) * obj->texture_dim.x;
	// coord.y = (0.5 - (asinf(hit_center.y) / M_PI)) * obj->texture_dim.y;


	// coord.x = ray->hit.x;
	// coord.y = ray->hit.z;


	// cross entre n et z

	float	vaidotte = v_dot(v_new(0, 1, 0), ray->nhit);

	ortho_x = v_cross(ray->nhit, vaidotte < 0.001 && vaidotte > -0.001 ? v_new(0, 1, 0) : v_new(1, 0, 0));
	ortho_x = v_dot(ortho_x, v_new(0, 0, 1)) > 0 ? ortho_x : v_scale(ortho_x, -1);
	ortho_y = v_cross(ray->nhit, ortho_x);

	coord.x = (int)abs(v_dot(ortho_x, ray->hit)) % obj->texture_dim.x;
	coord.y = (int)abs(v_dot(ortho_y, ray->hit)) % obj->texture_dim.y;

	// si ortho

	// printf("%f, %f\n", v_dot(ortho_x, ray->hit), v_dot(ortho_y, ray->hit));
	// printf("%d, %d\n\n", coord.x, coord.y);

	if (coord.x >= obj->texture_dim.x || coord.y >= obj->texture_dim.y || coord.x < 0 || coord.y < 0)
	{
		coord.x = -1;
		coord.y = -1;
	}

	return (coord);
}
