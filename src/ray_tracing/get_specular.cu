#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_specular.cu                                    :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/11 14:13:51 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/19 13:46:06 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the specular value of the color at hitpoint.
*/

__device__
t_color	get_specular(t_scene *scene, t_ray *primary_ray, t_ray *shadow_ray, t_light *light)
{
	t_color	new_col;
	double	specular_intensity;
	t_vec3	reflection;
	double	directional_light_factor;

	directional_light_factor = v_isnan(light->pos) ? 1000 : 1;
	reflection = reflect(primary_ray->dir, v_scale(primary_ray->nhit,
		primary_ray->n_dir));
	specular_intensity = pow((ft_clampd(v_dot(reflection,
		shadow_ray->dir), 0, 1)), scene->objects[primary_ray->hit_obj].specular_exp);
	specular_intensity *= light->intensity * scene->objects[primary_ray->hit_obj].ks *
		directional_light_factor;
	new_col = c_scale(vec_to_col(light->col), specular_intensity);
	// new_col = v_clamp(new_col, 0, 255);
	return (new_col);
}
