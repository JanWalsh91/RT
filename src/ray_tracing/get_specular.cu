#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_specular.cu                                    :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/11 14:13:51 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/21 16:56:46 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the specular value of the color at hitpoint.
*/

__device__
t_color	get_specular(t_scene *scene, t_ray *primary_ray, t_ray *shadow_ray, t_light *light)
{
	t_color	new_col;
	float	specular_intensity;
	t_vec3	reflection;
	float 	r2;
	 
	r2 = v_isnan(light->dir) ? v_dist(primary_ray->hit, light->pos) : 1;
	reflection = reflect(primary_ray->dir, v_scale(primary_ray->nhit,
		primary_ray->n_dir));
	specular_intensity = pow((ft_clampf(v_dot(reflection,
		shadow_ray->dir), 0, 1)), scene->objects[primary_ray->hit_obj].specular_exp);
	specular_intensity *= (light->intensity / r2) * scene->objects[primary_ray->hit_obj].ks;
	new_col = c_scale(vec_to_col(light->col), specular_intensity);
	// new_col = v_clamp(new_col, 0, 255);
	return (new_col);
}
