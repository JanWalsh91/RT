#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_sphere_intersection.cu                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:26:41 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/04 10:37:05 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"

/*
** Checks for a ray-sphere intersection.
*/

__device__
bool	get_sphere_intersection(t_raytracing_tools *r, t_ray *ray, int index)
{
	t_intersection_tools	i;

	i.v1 = v_sub(ray->origin, r->scenes->objects[index].pos);
	i.q.x = v_dot(ray->dir, ray->dir);
	i.q.y = 2 * v_dot(i.v1, ray->dir);
	i.q.z = v_dot(i.v1, i.v1) - pow(r->scenes->objects[index].rad, 2);
	if (!solve_quadratic(i.q, &i.r1, &i.r2))
		return (false);
	if (i.r2 < i.r1)
		ft_swapd(&i.r1, &i.r2);
	(i.r1 < 0) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0)
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0;
	if (ray->type == R_PRIMARY && r->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_SPHERE;
	}
	return (true);
}
