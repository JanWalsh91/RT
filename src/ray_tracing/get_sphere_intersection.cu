#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_sphere_intersection.cu                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:26:41 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/02 18:25:35 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"

/*
** Checks for a ray-sphere intersection.
*/

__device__
bool	get_sphere_intersection(t_scene *scene, t_ray *ray, int index)
{
	t_intersection_tools	i;

	i.v1 = v_sub(ray->origin, scene->objects[index].pos);
	i.q.x = v_dot(ray->dir, ray->dir);
	i.q.y = 2 * v_dot(i.v1, ray->dir);
	i.q.z = v_dot(i.v1, i.v1) - pow(scene->objects[index].rad, 2);
	if (!solve_quadratic(i.q, &i.r1, &i.r2))
		return (false);
	if (i.r2 < i.r1)
		ft_swapd(&i.r1, &i.r2);
	(i.r1 < 0) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0)
		return (false);
	scene->t > i.r1 ? ray->t = i.r1 : 0;
	if (ray->type == R_PRIMARY && scene->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_SPHERE;
	}
	return (true);
}
