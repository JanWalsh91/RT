#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_torus_intersection.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/09 13:52:47 by tgros             #+#    #+#             */
/*   Updated: 2017/05/27 14:31:43 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"

__device__
float	choose_between_two_roots(float a, float b)
{
	if (a < 0)
	{
		if (b > 0)
			return(b);
	}
	else if (b < 0)
	{
		if (a > 0)
			return(a);
	}
	else if (a < 0 && b < 0)
		return(-1);
	else if (a > b)
		return(b);
	return(a);
}

__device__
float	choose_between_four_roots(float a, float b, float c, float d)
{
	float e;
	float f;

	e = choose_between_two_roots(a, b);
	f = choose_between_two_roots(c, d);
	if (e > 0 && f > 0)	
		return ((e < f) ? e : f);
	else if (e > 0 && f < 0)
		return (e);
	else if (f > 0 && e < 0)
		return (f);
	else
		return (-1);
}

// void	get_torus_quartic()
// {
// 	qua.m = v_dot(ray->dir, ray->dir);
// 	qua.n = v_dot(ray->dir, v_sub(ray->origin, obj->pos));
// 	qua.o = v_dot(v_sub(ray->origin, obj->pos), v_sub(ray->origin, obj->pos));
// 	qua.p = v_dot(ray->dir, obj->dir);
// 	qua.q = v_dot(v_sub(ray->origin, obj->pos), obj->dir);

// 	// printf("%f, %f, %f\n", obj->dir.x, obj->dir.y, obj->dir.z);

// 	qua.a = qua.m * qua.m;
// 	qua.b = 4 * qua.m * qua.n;
// 	qua.c = 4 * (qua.m * qua.m) + 2 * qua.m * qua.o - 2 * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad) * qua.m + 4 * (obj->rad_torus * obj->rad_torus) * qua.p * qua.p;
// 	qua.d = 4 * qua.n * qua.o - 4 * (obj->rad_torus * obj->rad_torus + obj->rad *
// 		obj->rad) * qua.n + 8 * obj->rad_torus * obj->rad_torus * qua.p * qua.q;
// 	qua.e = qua.o * qua.o - 2 * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad) * qua.o + 4 * (obj->rad_torus * obj->rad_torus) * (qua.q * qua.q) +
// 	((obj->rad_torus * obj->rad_torus + obj->rad * obj->rad) * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad));
// }

__device__
bool		get_torus_intersection(t_raytracing_tools *r, t_ray *ray,
				int index)
{
	t_quartic	qua;
	t_object	*obj = &r->scene->objects[index];
	t_vec4im	sol;
	float		res;
	float		rad_torus2;
	float		rad2;

	// obj->dir.x = 0;
	// obj->dir.y = 0;
	// obj->dir.z = 1;
	 //obj->rad_torus = 0.000001;
	// obj->rad = 0.001;
	qua.m = v_dot(ray->dir, ray->dir);
	qua.n = v_dot(ray->dir, v_sub(ray->origin, obj->pos));
	qua.o = v_dot(v_sub(ray->origin, obj->pos), v_sub(ray->origin, obj->pos));
	qua.p = v_dot(ray->dir, obj->dir);
	qua.q = v_dot(v_sub(ray->origin, obj->pos), obj->dir);
	rad2 = powf(obj->rad, 2.0);
	rad_torus2 = powf(obj->rad_torus, 2.0);
	// printf("%f, %f, %f\n", obj->dir.x, obj->dir.y, obj->dir.z);

	//printf("Rad torus = %f, rad = %f\n", obj->rad_torus, obj->rad);


   // a = m^2
   // b = 4*m*n
   // c = 4*m^2 + 2*m*o - 2*(R^2+r^2)*m + 4*R^2*p^2
   // d = 4*n*o - 4*(R^2+r^2)*n + 8*R^2*p*q
   // e = o^2 - 2*(R^2+r^2)*o + 4*R^2*q^2 + (R^2-r^2)^2
	qua.a = powf(qua.m, 2.0);
	qua.b = (4.0 * qua.m * qua.n);
	qua.c = (4.0 * qua.a + (2.0 * qua.m * qua.o)) - (2.0 * rad_torus2 + rad2 *
		qua.m)+ (4.0 * rad_torus2 * qua.p * qua.p);
	qua.d = (4.0 * qua.n * qua.o) - (4.0 * (rad_torus2 + rad2) * qua.n)+ (8.0 * rad_torus2 * qua.p * qua.q);
	qua.e = (qua.o * qua.o) - (2.0 * (rad_torus2 + rad2) * qua.o) + (4.0 * rad_torus2 * qua.q * qua.q) +
	((rad_torus2 + rad2) * (rad_torus2 + rad2));
	if (!solve_quartic(&qua, &sol))
	{
		//printf("Return false\n");
	 	return (false);
	}
	if (isnan(sol.w.r) && isnan(sol.x.r) && isnan(sol.y.r) && isnan(sol.z.r))
		return(false);
	res = INFINITY;
	// if (r->pix.y == 1 && r->pix.x == 1)
	// {
	//  printf("sol quartic\nX1 : %f, %f\nX2 : %f, %f\nX3 : %f. %f\nX4 : %f, %f\n", sol.w.r, sol.w.i,
	// 	 sol.x.r, sol.x.i, sol.y.r, sol.y.i, sol.z.r, sol.z.i);
	// // printf("resultat : %f\n",res);
	// }
	if ((isnan(sol.x.i) || isnan(sol.y.i)) && (!isnan(sol.w.r) && !isnan(sol.x.r) && !isnan(sol.y.r) && !isnan(sol.z.r)))
	{
		printf("sol quartic\nX1 : %f, %f\nX2 : %f, %f\nX3 : %f. %f\nX4 : %f, %f\n", sol.w.r, sol.w.i,
	 	 sol.x.r, sol.x.i, sol.y.r, sol.y.i, sol.z.r, sol.z.i);
		res = choose_between_four_roots(sol.w.r, sol.x.r, sol.y.r, sol.z.r);
	}
	else 
		res = choose_between_two_roots(sol.w.r, sol.z.r);
	if (res < 0 || isnan(res))
		return (false);
	 r->t > res ? ray->t = res : 0;
	if (ray->type == R_PRIMARY && r->t > res)
	{
		ray->hit_obj = index;
		ray->hit_type = T_TORUS;
	}
	//printf("mnopq: %f, %f, %f, %f, %f : sol %f, %f, %f, %f\n", qua.m, qua.n, qua.o,qua.p,qua.q, sol.w, sol.x, sol.y, sol.z);
	return (true);
}
