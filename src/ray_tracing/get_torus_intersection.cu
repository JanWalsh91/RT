#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_torus_intersection.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/09 13:52:47 by tgros             #+#    #+#             */
/*   Updated: 2017/05/26 17:09:45 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"

__device__
float	choose_between_two_roots(float a, float b)
{
	if (a < 0)
	{
		if (b > 0)
			return(b);
	}
	else if (b < 0)
	{
		if (a > 0)
			return(a);
	}
	else if (a < 0 && b < 0)
		return(-1);
	else if (a > b)
		return(b);
	return(a);
}

__device__
float	choose_between_four_roots(float a, float b, float c, float d)
{
	float e;
	float f;

	e = choose_between_two_roots(a, b);
	f = choose_between_two_roots(c, d);
	if (e > 0 && f > 0)	
		return ((e < f) ? e : f);
	else if (e > 0 && f < 0)
		return (e);
	else if (f > 0 && e < 0)
		return (f);
	else
		return (-1);
}

// void	get_torus_quartic()
// {
// 	qua.m = v_dot(ray->dir, ray->dir);
// 	qua.n = v_dot(ray->dir, v_sub(ray->origin, obj->pos));
// 	qua.o = v_dot(v_sub(ray->origin, obj->pos), v_sub(ray->origin, obj->pos));
// 	qua.p = v_dot(ray->dir, obj->dir);
// 	qua.q = v_dot(v_sub(ray->origin, obj->pos), obj->dir);

// 	// printf("%f, %f, %f\n", obj->dir.x, obj->dir.y, obj->dir.z);

// 	qua.a = qua.m * qua.m;
// 	qua.b = 4 * qua.m * qua.n;
// 	qua.c = 4 * (qua.m * qua.m) + 2 * qua.m * qua.o - 2 * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad) * qua.m + 4 * (obj->rad_torus * obj->rad_torus) * qua.p * qua.p;
// 	qua.d = 4 * qua.n * qua.o - 4 * (obj->rad_torus * obj->rad_torus + obj->rad *
// 		obj->rad) * qua.n + 8 * obj->rad_torus * obj->rad_torus * qua.p * qua.q;
// 	qua.e = qua.o * qua.o - 2 * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad) * qua.o + 4 * (obj->rad_torus * obj->rad_torus) * (qua.q * qua.q) +
// 	((obj->rad_torus * obj->rad_torus + obj->rad * obj->rad) * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad));
// }

__device__
bool		get_torus_intersection(t_raytracing_tools *r, t_ray *ray,
				int index)
{
	t_quartic	qua;
	t_object	*obj = &r->scene->objects[index];
	t_vec4im	sol;
	float		res;

	obj->dir.x = 0;
	obj->dir.y = 0;
	obj->dir.z = -1;
	obj->rad_torus = 0.000001;
	obj->rad = 0.001;
	qua.m = v_dot(ray->dir, ray->dir);
	qua.n = v_dot(ray->dir, v_sub(ray->origin, obj->pos));
	qua.o = v_dot(v_sub(ray->origin, obj->pos), v_sub(ray->origin, obj->pos));
	qua.p = v_dot(ray->dir, obj->dir);
	qua.q = v_dot(v_sub(ray->origin, obj->pos), obj->dir);

	// printf("%f, %f, %f\n", obj->dir.x, obj->dir.y, obj->dir.z);

	// printf("Rad torus = %f, rad = %f\n", obj->rad_torus, obj->rad);

	qua.a = qua.m * qua.m;
	qua.b = 4 * qua.m * qua.n;
	qua.c = (4 * (qua.m * qua.m)) + (2 * qua.m * qua.o) - (2 * (obj->rad_torus * obj->rad_torus + obj->rad * obj->rad)) *
	qua.m + (4 * (obj->rad_torus * obj->rad_torus) * qua.p * qua.p);
	qua.d = (4 * qua.n * qua.o) - 4 * ((obj->rad_torus * obj->rad_torus) + (obj->rad * obj->rad)) * qua.n + 8 * obj->rad_torus * obj->rad_torus * qua.p * qua.q;
	qua.e = qua.o * qua.o - 2 * (obj->rad_torus * obj->rad_torus + obj->rad * obj->rad) * qua.o + (4 * (obj->rad_torus * obj->rad_torus) * (qua.q * qua.q)) + ((obj->rad_torus * obj->rad_torus + obj->rad * obj->rad) * (obj->rad_torus * obj->rad_torus + obj->rad * obj->rad));



	if (!solve_quartic(&qua, &sol))
	{
		//printf("Return false\n");
	 	return (false);
	}
	if (isnan(sol.x.i) || isnan(sol.y.i))
		res = choose_between_four_roots(sol.w.r, sol.x.r, sol.y.r, sol.z.r);
	else
		res = choose_between_two_roots(sol.w.r, sol.z.r);
	if (r->pix.x == 1 && r->pix.y == 1)
	{
		printf("Coucou\n");
	}
	if (res < 0)
	{
		//printf("Return false\n");
		return (false);
	}

	if (r->pix.y == 600 && r->pix.x == 1)
	{
	// printf("sol quartic\nX1 : %f, %f\nX2 : %f, %f\nX3 : %f. %f\nX4 : %f, %f\n", sol.w.r, sol.w.i,
		// sol.x.r, sol.x.i, sol.y.r, sol.y.i, sol.z.r, sol.z.i);
	// printf("resultat : %f\n",res);
	}
	 r->t > res ? ray->t = res : 0;
	if (ray->type == R_PRIMARY && r->t > res)
	{
		ray->hit_obj = index;
		ray->hit_type = T_TORUS;
	}
	//printf("mnopq: %f, %f, %f, %f, %f : sol %f, %f, %f, %f\n", qua.m, qua.n, qua.o,qua.p,qua.q, sol.w, sol.x, sol.y, sol.z);
	return (true);
}
