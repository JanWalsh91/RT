#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_torus_intersection.cu                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/09 13:52:47 by tgros             #+#    #+#             */
/*   Updated: 2017/05/22 16:21:34 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"

__device__
float	choose_between_two_roots(float a, float b)
{
	if (a < 0)
	{
		if (b > 0)
			return(b);
	}
	else if (b < 0)
	{
		if (a > 0)
			return(a);
	}
	else if (a < 0 && b < 0)
		return(-1);
	else if (a > b)
		return(b);
	return(a);
}

__device__
float	choose_between_four_roots(float a, float b, float c, float d)
{
	float e;
	float f;

	e = choose_between_two_roots(a, b);
	f = choose_between_two_roots(c, d);
	if (e > 0 && f > 0)	
		return ((e < f) ? e : f);
	else if (e > 0 && f < 0)
		return (e);
	else if (f > 0 && e < 0)
		return (f);
	else
		return (-1);
}

// void	get_torus_quartic()
// {
// 	qua.m = v_dot(ray->dir, ray->dir);
// 	qua.n = v_dot(ray->dir, v_sub(ray->origin, obj->pos));
// 	qua.o = v_dot(v_sub(ray->origin, obj->pos), v_sub(ray->origin, obj->pos));
// 	qua.p = v_dot(ray->dir, obj->dir);
// 	qua.q = v_dot(v_sub(ray->origin, obj->pos), obj->dir);

// 	// printf("%f, %f, %f\n", obj->dir.x, obj->dir.y, obj->dir.z);

// 	qua.a = qua.m * qua.m;
// 	qua.b = 4 * qua.m * qua.n;
// 	qua.c = 4 * (qua.m * qua.m) + 2 * qua.m * qua.o - 2 * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad) * qua.m + 4 * (obj->rad_torus * obj->rad_torus) * qua.p * qua.p;
// 	qua.d = 4 * qua.n * qua.o - 4 * (obj->rad_torus * obj->rad_torus + obj->rad *
// 		obj->rad) * qua.n + 8 * obj->rad_torus * obj->rad_torus * qua.p * qua.q;
// 	qua.e = qua.o * qua.o - 2 * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad) * qua.o + 4 * (obj->rad_torus * obj->rad_torus) * (qua.q * qua.q) +
// 	((obj->rad_torus * obj->rad_torus + obj->rad * obj->rad) * (obj->rad_torus * obj->rad_torus +
// 		obj->rad * obj->rad));
// }

__device__
bool		get_torus_intersection(t_raytracing_tools *r, t_ray *ray,
				int index)
{
	t_quartic	qua;
	t_object	*obj = &r->scene->objects[index];
	t_vec4im	sol;
	float		res;

	obj->dir.x = 0;
	obj->dir.y = 0;
	obj->dir.z = 1;
	obj->rad_torus = 1;
	obj->rad = 10;
 //	m = D|D, n = D|X, o = X|X, p = D|V, q = X|V
	qua.m = v_dot(ray->dir, ray->dir);
	qua.n = v_dot(ray->dir, v_sub(ray->origin, obj->pos));
	qua.o = v_dot(v_sub(ray->origin, obj->pos), v_sub(ray->origin, obj->pos));
	qua.p = v_dot(ray->dir, obj->dir);
	qua.q = v_dot(v_sub(ray->origin, obj->pos), obj->dir);

	// printf("%f, %f, %f\n", obj->dir.x, obj->dir.y, obj->dir.z);

	// printf("Rad torus = %f, rad = %f\n", obj->rad_torus, obj->rad);


   // a = m^2
   // b = 4*m*n
   // c = 4*m^2 + 2*m*o - 2*(R^2+r^2)*m + 4*R^2*p^2
   // d = 4*n*o - 4*(R^2+r^2)*n + 8*R^2*p*q
   // e = o^2 - 2*(R^2+r^2)*o + 4*R^2*q^2 + (R^2-r^2)^2
	qua.a = qua.m * qua.m;
	qua.b = 4 * qua.m * qua.n;
	qua.c = (4 * (qua.m * qua.m)) + (2 * qua.m * qua.o) - (2 * (obj->rad_torus * obj->rad_torus + obj->rad * obj->rad)) *
		qua.m + (4 * (obj->rad_torus * obj->rad_torus) * qua.p * qua.p);
	qua.d = (4 * qua.n * qua.o) - 4 * ((obj->rad_torus * obj->rad_torus) + (obj->rad * obj->rad)) * qua.n + 8 * obj->rad_torus * obj->rad_torus * qua.p * qua.q;
	qua.e = qua.o * qua.o - 2 * (obj->rad_torus * obj->rad_torus + obj->rad * obj->rad) * qua.o +
	(4 * (obj->rad_torus * obj->rad_torus) * (qua.q * qua.q)) +
	((obj->rad_torus * obj->rad_torus + obj->rad * obj->rad) * (obj->rad_torus * obj->rad_torus + obj->rad * obj->rad));

	if (r->pix.x == 600)
	{
		//printf("rpixy : %d ray dir: %f %f %f abcde : %f %f %f %f %f\n", r->pix.y, ray->dir.x, ray->dir.y, ray->dir.z, qua.a, qua.b, qua.c, qua.d, qua.e);
	}

	if (!solve_quartic(&qua, &sol))
	{
		//printf("Return false\n");
	 	return (false);
	}
	if (r->pix.y == 600 && r->pix.x == 600)
	{
	 printf("sol quartic\nX1 : %f, %f\nX2 : %f, %f\nX3 : %f. %f\nX4 : %f, %f\n", sol.w.r, sol.w.i,
		 sol.x.r, sol.x.i, sol.y.r, sol.y.i, sol.z.r, sol.z.i);
	// printf("resultat : %f\n",res);
	}
	if (isnan(sol.x.i) || isnan(sol.y.i))
		res = choose_between_four_roots(sol.w.r, sol.x.r, sol.y.r, sol.z.r);
	else
		res = choose_between_two_roots(sol.w.r, sol.z.r);
	if (r->pix.x == 1 && r->pix.y == 1)
	{
		//printf("Coucou\n");
	}
	if (res < 0)
	{
		//printf("Return false\n");
		return (false);
	}

	if (r->pix.y == 600 && r->pix.x == 600)
	{
	 printf("sol quartic\nX1 : %f, %f\nX2 : %f, %f\nX3 : %f. %f\nX4 : %f, %f\n", sol.w.r, sol.w.i,
		 sol.x.r, sol.x.i, sol.y.r, sol.y.i, sol.z.r, sol.z.i);
	// printf("resultat : %f\n",res);
	}
	 r->t > res ? ray->t = res : 0;
	if (ray->type == R_PRIMARY && r->t > res)
	{
		ray->hit_obj = index;
		ray->hit_type = T_TORUS;
	}
	//printf("mnopq: %f, %f, %f, %f, %f : sol %f, %f, %f, %f\n", qua.m, qua.n, qua.o,qua.p,qua.q, sol.w, sol.x, sol.y, sol.z);
	return (true);
}
