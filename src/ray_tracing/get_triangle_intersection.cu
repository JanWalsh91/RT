#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_triangle_intersection.cu                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:26:41 by talemari          #+#    #+#             */
/*   Updated: 2017/05/18 15:28:58 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../../inc/objparser.h"
/*
** Checks for a ray-sphere intersection.
*/

__device__
static void		print_vec2(t_vec3 vec)
{
	printf("x : %f, y : %f, z : %f\n", vec.x, vec.y, vec.z);
}

__device__
bool	get_triangle_intersection(t_raytracing_tools *r, t_triangle *t, t_ray *ray, int index)
{
	float		det;
	float		invdet;
	float		u;
	float		v;
	float		tmp;
	t_vec3		edge1;
	t_vec3		edge2;
	t_vec3		s1;
	t_vec3		s2;
	t_vec3		s3;

	if (r->idx == 0)
		printf("get triangle itnersection\n");
	//if (r->pix.x == 600 && r->pix.y < 5)
	//{
	
	edge1 = v_sub(t->v2, t->v1);
	edge2 = v_sub(t->v3, t->v1);
	if (r->idx == 0)
	{
		printf("edge: %f, %f, %f edge2: %f, %f, %f\n", edge1.x, edge1.y, edge1.z, edge2.x, edge2.y, edge2.z);
		print_vec2(t->v1);
		print_vec2(t->v2);
		print_vec2(t->v3);
	}
	__syncthreads();
	s1 = v_cross(ray->dir, edge2);
	det = v_dot(edge1, s1);
	// printf("det = %f\n", det);
	if (r->idx == 0)
		C(10)
	__syncthreads();
	if (det > -0.0001 && det < 0.0001)
	{
		if (r->idx == 0)
			C(2)
				
		return (false);
	}
	
		// printf("idx: %d, det: %f\n", r->idx, det);
		
	__syncthreads();
	invdet = 1 / det;
	s2 = v_sub(ray->origin, t->v1);
	u = v_dot(s2, s1) * invdet;
	if (u < 0 || u > 1)
	{
		if (r->idx == 0)
			C(3)	
		return (false);
	}
	if (r->idx == 0)
		C(12)
	__syncthreads();
	s3 = v_cross(ray->dir, edge1);
	v = v_dot(ray->dir, s3) * invdet;
	if (v < 0 || (u + v) > 1)
	{
		if (r->idx == 0)
			C(4)	
		return (false);
	}
	if (r->idx == 0)
		C(13)
	__syncthreads();	
	tmp = v_dot(edge2, s3) * invdet;
	if ((tmp < 0) || (tmp > ray->t) || ray->type == R_SHADOW)
	{
		if (r->idx == 0)
			C(5)	
		return (false);
	}
	if (r->idx == 0)
		C(14)
	__syncthreads();
	ray->t = tmp - 0.005;
	ray->hit_obj = index;
	ray->hit_type = T_TRIANGLE;
	ray->nhit = v_cross(edge2, edge1);
	//}
	if (r->idx == 0)
		C(1)
	__syncthreads();
	return (true);
}
