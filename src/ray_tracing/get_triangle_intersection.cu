#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_triangle_intersection.cu                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:26:41 by talemari          #+#    #+#             */
/*   Updated: 2017/05/18 15:28:58 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../../inc/objparser.h"
/*
** Checks for a ray-sphere intersection.
*/

__device__
bool	get_triangle_intersection(t_triangle *t, t_ray *ray, int index, t_vec3 *norm)
{
	float		det;
	float		invdet;
	float		u;
	float		v;
	float		tmp;
	t_vec3		edge1;
	t_vec3		edge2;
	t_vec3		s1;
	t_vec3		s2;
	t_vec3		s3;

	edge1 = v_sub(t->v2, t->v1);
	edge2 = v_sub(t->v3, t->v1);
	s1 = v_cross(ray->dir, edge2);
	det = v_dot(edge1, s1);
	if (det > -0.0001 && det < 0.0001)
		return (false);
	invdet = 1 / det;
	s2 = v_sub(ray->origin, t->v1);
	u = v_dot(s2, s1) * invdet;
	if (u < 0 || u > 1)
		return (false);
	s3 = v_cross(ray->dir, edge1);
	v = v_dot(ray->dir, s3) * invdet;
	if (v < 0 || (u + v) > 1)
		return (false);
	tmp = v_dot(edge2, s3) * invdet;
	if ((tmp < 0) || (tmp > ray->t) || ray->type == R_SHADOW)
		return (false);
	ray->t = tmp - 0.005;
	ray->hit_obj = index;
	ray->hit_type = T_TRIANGLE;
	*norm = v_cross(edge2, edge1);
	return (true);
}
