#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   in_shadow.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/05 13:13:23 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/19 14:24:06 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Throws a shadow ray for each light in the scene.
** Returns true if there is an intersection between a light and the origin of
** the ray, else returns false.
*/
__device__
static void	filter_for_transparency(t_vec3 *dim_light, t_vec3 obj_col, float k);
__device__
static void	filter_color(float *dim, float obj_col, float k);

__device__
int		in_shadow(t_raytracing_tools *r, t_ray *primary_ray,
		t_ray *shadow_ray, t_light *light, t_vec3 *dim_light)
{
	int			i;
	float		max;
	int			is_transparent;

	is_transparent = 0;
	r->t = INFINITY;
	shadow_ray->t = INFINITY;
	shadow_ray->type = R_SHADOW;
	shadow_ray->origin = v_add(primary_ray->hit,
		v_scale(primary_ray->nhit, BIAS * primary_ray->n_dir));
	if (!v_isnan(light->pos))
		max = v_length(v_sub(light->pos, shadow_ray->origin));
	else
		max = INFINITY;
	if (!v_isnan(light->pos))
		shadow_ray->dir = v_sub(light->pos, shadow_ray->origin);
	else
		shadow_ray->dir = v_scale(light->dir, -1);
	shadow_ray->dir = v_norm(shadow_ray->dir);
	i = -1;
	while (r->scene->objects[++i].type != T_INVALID_TOKEN)
	{
		if (intersects(r, shadow_ray, i) &&
			shadow_ray->t < max && shadow_ray->t > 0.0) 
		{
			if (r->scene->objects[i].transparency > 0.01)
			{
				filter_for_transparency(dim_light, 
					get_object_color(&r->scene->objects[i], primary_ray),
					r->scene->objects[i].transparency);
				is_transparent = 1;
			}
			else
				return (2);
		}
	}
	return (is_transparent);
}

__device__
static void	filter_for_transparency(t_vec3 *dim_light, t_vec3 obj_col, float k)
{
	filter_color(&dim_light->x, obj_col.x, k);
	filter_color(&dim_light->y, obj_col.y, k);
	filter_color(&dim_light->z, obj_col.z, k);
}

__device__
static void	filter_color(float *dim, float obj_col, float k)
{
	*dim *= (1 - (255 - obj_col) / 255 * (1 - k)) * k;				
}