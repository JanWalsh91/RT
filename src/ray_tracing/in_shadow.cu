#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   in_shadow.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/05 13:13:23 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/04 11:47:47 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"

/*
** Throws a shadow ray for each light in the scene.
** Returns true if there is an intersection between a light and the origin of
** the ray, else returns false.
*/

__device__
bool	in_shadow(t_raytracing_tools *r, t_ray *primary_ray,
		t_ray *shadow_ray, t_light *light)
{
	int			i;
	double		max;

	r->t = INFINITY;
	shadow_ray->type = R_SHADOW;
	shadow_ray->origin = v_add(primary_ray->hit,
		v_scale(primary_ray->nhit, BIAS * primary_ray->n_dir));
	if (!v_isnan(light->pos))
		max = v_length(v_sub(light->pos, shadow_ray->origin));
	else
		max = INFINITY;
	if (!v_isnan(light->pos))
		shadow_ray->dir = v_sub(light->pos, shadow_ray->origin);
	else
		shadow_ray->dir = v_scale(light->dir, -1);
	shadow_ray->dir = v_norm(shadow_ray->dir);
	i = -1;
	while (r->scenes->objects[++i].type != T_INVALID_TOKEN)
	{
		if (intersects(r, shadow_ray, i) &&
			shadow_ray->t < max && shadow_ray->t > 0)
			return (true);
	}
	return (false);
}
