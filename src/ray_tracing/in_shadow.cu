#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   in_shadow.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/05 13:13:23 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/18 16:43:02 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Throws a shadow ray for each light in the scene.
** Returns true if there is an intersection between a light and the origin of
** the ray, else returns false.
*/
__device__
static void	filter_for_transparency(t_vec3 *dim_light, t_vec3 obj_col, float k);
__device__
static void	filter_color(float *dim, float obj_col, float k);

__device__
int		in_shadow(t_raytracing_tools *r, t_ray *primary_ray,
		t_ray *shadow_ray, t_light *light, t_vec3 *dim_light)
{
	int			i;
	float		max;
	int			is_transparent;

	is_transparent = 0;
	r->t = INFINITY;
	shadow_ray->t = INFINITY;
	shadow_ray->type = R_SHADOW;
	shadow_ray->origin = v_add(primary_ray->hit,
		v_scale(primary_ray->nhit, BIAS * primary_ray->n_dir));
	// shadow_ray->t = 0.f; ??
	if (!v_isnan(light->pos))
		max = v_length(v_sub(light->pos, shadow_ray->origin));
	else
		max = INFINITY;
	if (!v_isnan(light->pos))
		shadow_ray->dir = v_sub(light->pos, shadow_ray->origin);
	else
		shadow_ray->dir = v_scale(light->dir, -1);
	shadow_ray->dir = v_norm(shadow_ray->dir);
	i = -1;
	while (r->scene->objects[++i].type != T_INVALID_TOKEN)
	{
		if (intersects(r, shadow_ray, i) &&
			shadow_ray->t < max && shadow_ray->t > 0.0) 
		{
			// if (r->pix.x == 485 && r->pix.y == 577)
				// printf("Intersect avec : %d\n", r->scene->objects[i].type);
			// if (r->pix.x == 527 && r->pix.y == 303)
			// {
				// printf("Intersect avec : %d et t = %f\n", r->scene->objects[i].type, shadow_ray->t);
			// }
			if (r->scene->objects[i].transparency > 0.01)
			{
				// if (r->pix.x == 485 && r->pix.y == 577)
					// printf("Transparency: [%f]\n", r->scene->objects[i].transparency);
				// if (r->pix.x == 527 && r->pix.y == 303)
					// printf("Transparency: [%f]\n", r->scene->objects[i].transparency);
				filter_for_transparency(dim_light, get_object_color(&r->scene->objects[i], primary_ray), r->scene->objects[i].transparency);
				is_transparent = 1;
			}
			else
				return (2);
		}
	}
	return (is_transparent);
}

__device__
static void	filter_for_transparency(t_vec3 *dim_light, t_vec3 obj_col, float k)
{
	filter_color(&dim_light->x, obj_col.x, k);
	filter_color(&dim_light->y, obj_col.y, k);
	filter_color(&dim_light->z, obj_col.z, k);
}

__device__
static void	filter_color(float *dim, float obj_col, float k)
{
	*dim *= (1 - (255 - obj_col) / 255 * (1 - k)) * k;				
}