#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   in_shadow.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/05 13:13:23 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/26 11:15:28 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Throws a shadow ray for each light in the scene.
** Returns true if there is an intersection between a light and the origin of
** the ray, else returns false.
*/

__device__
int		in_shadow(t_raytracing_tools *r, t_ray *primary_ray,
		t_ray *shadow_ray, t_light *light, t_color *dim_light)
{
	int			i;
	float		max;
	int			is_transparent;

	is_transparent = 0;
	r->t = INFINITY;
	shadow_ray->type = R_SHADOW;
	shadow_ray->origin = v_add(primary_ray->hit,
		v_scale(primary_ray->nhit, BIAS * primary_ray->n_dir));
	if (!v_isnan(light->pos))
		max = v_length(v_sub(light->pos, shadow_ray->origin));
	else
		max = INFINITY;
	if (!v_isnan(light->pos))
		shadow_ray->dir = v_sub(light->pos, shadow_ray->origin);
	else
		shadow_ray->dir = v_scale(light->dir, -1);
	shadow_ray->dir = v_norm(shadow_ray->dir);
	i = -1;
	// while (r->scene->objects[++i].type != T_INVALID_TOKEN)
	// {
	// 	if (intersects(r, shadow_ray, i) &&
	// 		shadow_ray->t < max && shadow_ray->t > 0)
	// 		return (true);
	// }
	while (r->scene->objects[++i].type != T_INVALID_TOKEN)
	{
		if (intersects(r, shadow_ray, i) &&
			shadow_ray->t < max && shadow_ray->t > 0)
		{
			if (r->scene->objects[i].transparency > 0.01)
			{
				*dim_light = c_min(c_scale(vec_to_col(r->scene->objects[i].col), r->scene->objects[i].transparency),
				*dim_light);
				is_transparent = 1;
			}
			else
				return (2);
		}
	}
	return (is_transparent);
}
