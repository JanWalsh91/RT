/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   intersects.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/06 12:11:23 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/07 20:27:38 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Checks for a ray-object intersection.
*/

__device__
bool	intersects(t_raytracing_tools *r, t_ray *ray, int index)
{
	if (r->scene->objects[index].type == T_SPHERE)
		return (get_sphere_intersection(r, ray, index));
	else if (r->scene->objects[index].type == T_PLANE)
		return (get_plane_intersection(r, ray, index));
	else if (r->scene->objects[index].type == T_DISK)
		return (get_disk_intersection(r, ray, index));
	else if (r->scene->objects[index].type == T_CYLINDER)
		return (get_cylinder_intersection(r, ray, index));
	else if (r->scene->objects[index].type == T_CONE)
		return (get_cone_intersection(r, ray, index));
	else if (r->scene->objects[index].type == T_PARABOLOID)
		return (get_paraboloid_intersection(r, ray, index));
	else if (r->scene->objects[index].type == T_OBJ)
		return (get_obj_intersection(r, ray, index));
	return (false);
}
