#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   refract.cu                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 15:37:56 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/03 15:10:11 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Given a ray direction and normal at hit point, returns the direction of the
** refracted ray.
*/

__device__
t_vec3	refract(t_vec3 ray_dir, t_vec3 nhit, float ray_ior, float new_ior)
{
	float	cosi;
	float	etai;
	float	etat;
	float	eta;
	float	k;
	t_vec3	n;

	cosi = ft_clampf(v_dot(ray_dir, nhit), -1, 1);
	etai = ray_ior;
	etat = new_ior;
	n = nhit;
	if (cosi < 0)
		cosi = -cosi;
	else
	{
		ft_swapf(&etai, &etat);
		n = v_scale(nhit, -1);
	}
	eta = etai / etat;
	k = 1 - eta * eta * (1 - cosi * cosi);
	//if k < 0, there is no refracted ray;
	return (k < 0 ? v_new(NAN, NAN, NAN) : v_add(v_scale(ray_dir, eta), v_scale(n, (eta * cosi - sqrtf(k))))); 
}
// Vec3f refract(const Vec3f &I, const Vec3f &N, const float &ior) 
// { 
//     float cosi = clamp(-1, 1, I.dotProduct(N)); 
//     float etai = 1, etat = ior; 
//     Vec3f n = N; 
//     if (cosi < 0) { cosi = -cosi; } else { std::swap(etai, etat); n= -N; } 
//     float eta = etai / etat; 
//     float k = 1 - eta * eta * (1 - cosi * cosi); 
//     return k < 0 ? 0 : eta * I + (eta * cosi - sqrtf(k)) * n; 
// } 