#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   refract.cu                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 15:37:56 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/18 13:24:47 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Given a ray direction and normal at hit point, returns the direction of the
** refracted ray.
*/

__device__
t_vec3	refract(t_vec3 ray_dir, t_vec3 nhit, float ray_ior, float new_ior)
{
	float	cosi;
	float	etai;
	float	etat;
	float	eta;
	float	k;
	t_vec3	n;

	cosi = ft_clampf(v_dot(ray_dir, nhit), -1, 1);
	etai = ray_ior;
	etat = new_ior;
	n = nhit;
	if (cosi < 0)
		cosi = -cosi;
	else
	{
		ft_swapf(&etai, &etat);
		n = v_scale(nhit, -1);
	}
	eta = etai / etat;
	k = 1 - eta * eta * (1 - cosi * cosi);
	return (k < 0 ? v_new(NAN, NAN, NAN) : v_add(v_scale(ray_dir, eta), v_scale(n, (eta * cosi - sqrtf(k))))); 
}
