#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   refract.cu                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 15:37:56 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/26 11:30:00 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Given a ray direction and normal at hit point, returns the direction of the
** refracted ray.
*/

__device__
t_vec3	refract(t_vec3 ray_dir, t_vec3 nhit, float ior)
{
	float	cosi;
	float	etai;
	float	etat;
	float	eta;
	float	k;
	t_vec3	n;

	cosi = ft_clampf(v_dot(ray_dir, nhit), -1, 1);
	etai = 1;
	etat = ior;
	n = nhit;
	if (cosi < 0)
		cosi = -cosi;
	else
	{
		ft_swapf(&etai, &etat);
		n = v_scale(nhit, -1);
	}
	eta = etai / etat;
	k = 1 - eta * eta * (1 - cosi * cosi);
	//if k < 0, there is no refracted ray;
	return (k < 0 ? v_new(0, 0, 0) : v_add(v_scale(ray_dir, eta), v_scale(n, (eta * cosi - __dsqrt_rn(k))))); 
}
// Vec3f refract(const Vec3f &I, const Vec3f &N, const float &ior) 
// { 
//     float cosi = clamp(-1, 1, I.dotProduct(N)); 
//     float etai = 1, etat = ior; 
//     Vec3f n = N; 
//     if (cosi < 0) { cosi = -cosi; } else { std::swap(etai, etat); n= -N; } 
//     float eta = etai / etat; 
//     float k = 1 - eta * eta * (1 - cosi * cosi); 
//     return k < 0 ? 0 : eta * I + (eta * cosi - sqrtf(k)) * n; 
// } 