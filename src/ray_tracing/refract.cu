#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   refract.cu                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 15:37:56 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/17 12:24:08 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Given a ray direction and normal at hit point, returns the direction of the
** refracted ray.
*/

__device__
t_vec3	refract(t_vec3 ray_dir, t_vec3 nhit, double ior)
{
	double	cosi;
	double	etai;
	double	etat;
	double	eta;
	float	k;
	t_vec3	n;

	cosi = ft_clampd(v_dot(ray_dir, nhit), -1, 1);
	etai = 1;
	etat = ior;
	n = nhit;
	if (cosi < 0)
		cosi = -cosi;
	else
	{
		ft_swapd(&etai, &etat);
		n = v_scale(nhit, -1);
	}
	eta = etai / etat;
	k = 1 - eta * eta * (1 - cosi * cosi);
	//if k < 0, there is no refracted ray;
	return (k < 0 ? v_new(0, 0, 0) : v_add(v_scale(ray_dir, eta), v_scale(n, (eta * cosi - __dsqrt_rn(k))))); 
}
// Vec3f refract(const Vec3f &I, const Vec3f &N, const float &ior) 
// { 
//     float cosi = clamp(-1, 1, I.dotProduct(N)); 
//     float etai = 1, etat = ior; 
//     Vec3f n = N; 
//     if (cosi < 0) { cosi = -cosi; } else { std::swap(etai, etat); n= -N; } 
//     float eta = etai / etat; 
//     float k = 1 - eta * eta * (1 - cosi * cosi); 
//     return k < 0 ? 0 : eta * I + (eta * cosi - sqrtf(k)) * n; 
// } 