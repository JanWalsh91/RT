#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/21 12:20:33 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/cuda_call.h"

#include <pthread.h>

/*
** Updates a camera's pixel_map (color of image pixels).
*/
#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void	printte_matrix(t_matrix m)
{
	int i;
	int	y;

	y = -1;
	while (++y < 4)
	{
		i = -1;
		while (++i < 4)
			printf("[%f]", m[y][i]);
		printf("\n");
	}
}

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map/*, t_pt2 *loading*/)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	int					idx;

	r.pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene;
	// printf("%d %d\n", blockIdx.x, blockIdx.y);
	// printf("%d %d\n", threadIdx.x, threadIdx.y);	

    idx = scene->res.x * r.pix.y + r.pix.x;

	if (idx < 1)
	{
		// printf("Coucou\n");
		printf("%f\n", scene->lights[1].col.x);
		// *loading = 20;
	}


	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		cam_ray = init_camera_ray(&r);	
		d_pixel_map[idx] = filter(cast_primary_ray(&r, &cam_ray), scene->cameras->filter);
		// d_pixel_map[idx] = cast_primary_ray(&r, &cam_ray);
	}
	//loading->x = atomicAdd(&(loading->x), 1);
	//__syncthreads();
}



t_object	*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	array = (t_object *)malloc(sizeof(t_object) * (size + 1)); // malloc error
	array[size].type = T_INVALID_TOKEN;
	object = head;
	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object)); // stack memcpy ?
		object = object->next;
	}
	return (array);
}

size_t			get_object_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}

t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light	*head;
	t_light	*array;

	size = 0;
	head = light;
	while (light)
	{
		++size;
		light = light->next;
	}
	array = (t_light *)malloc(sizeof(t_light) * (size + 1)); // malloc error
	// bzero(array, sizeof(t_light) * (size + 1));
	array[size].col = v_new(NAN, NAN, NAN);
	light = head;
	size = -1;
	// printf("COLOR: %f\n", array[size].col.x);
	while (light)
	{
		array[++size].col = v_new(NAN, NAN, NAN);
		memcpy(&array[size], light, sizeof(t_light));
		light = light->next;
		// printf("%p\n", light);
	}
	// printf("COLOR: %f\n", array[size].col.x);
	return (array);
}

size_t			get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	// printf("%f\n", lights[size].col.x);
	while (!v_isnan(lights[size].col))
		++size;
	// C(3)
	return ((size + 1) * sizeof(t_light));
}

		#include <time.h>

// void		*loading_bar(void	*pt)
// {
// 	t_pt2 *max = (t_pt2*) pt;
// 	// usleep(1000000);

// 	while (max->x < max->y / 2) ;
// 		printf("loading : %d\n", max->x);
// 	return NULL;
// }

void		render(t_scene *scene)
{
	t_color		*d_pixel_map;
	t_color 	*h_pixel_map;
	t_scene		*h_scene_to_array;
	t_scene		*h_d_scene;
	t_scene		*d_scene;
	dim3		block_size;
	dim3		grid_size;
	clock_t		start;
	clock_t		stop;

	size_t       available;
  	size_t       total;

	hipDeviceSetLimit(hipLimitStackSize, 1024 * scene->ray_depth);
	// Preparation des deux structures temporarires
	start = clock();
	if (!(h_scene_to_array = (t_scene *)malloc(sizeof(t_scene))))
		exit(0); // malloc error
	stop = clock();
	printf("1. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	

	start = clock();
	if (!(h_d_scene = (t_scene *)malloc(sizeof(t_scene))))
		exit(0); // malloc error
	stop = clock();
	printf("2. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	
	start = clock();
	if (!(memcpy(h_scene_to_array, scene, sizeof(t_scene))))
		exit(0);
	stop = clock();
	printf("3. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	
	start = clock();
	memcpy(h_d_scene, scene, sizeof(t_scene));
	stop = clock();
	printf("4. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);

	// Creation des tableaux 1D pour les objets et lumieres
	start = clock();
	h_scene_to_array->objects = list_to_array_objects(scene->objects);
	stop = clock();
	printf("5. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	start = clock();
	h_scene_to_array->lights = list_to_array_lights(scene->lights);
	stop = clock();
	printf("6. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	
	start = clock();
	h_scene_to_array->cameras = (t_camera *)malloc(sizeof(t_camera));
	stop = clock();
	printf("7. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	
	// t_vec3	*truc;

	start = clock();
	// hipMalloc(&truc, sizeof(t_vec3));
	gpuErrchk(hipSetDevice(0));
	// hipFree(0);
	stop = clock();
	printf("80. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);


	start = clock();
	memcpy(h_scene_to_array->cameras, scene->cameras, sizeof(t_camera));
	stop = clock();
	printf("8. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	
	// Allocation de la memoire GPU
	start = clock();
	gpuErrchk(hipMalloc(&(h_d_scene->lights), get_lights_array_length(h_scene_to_array->lights)));
	
	stop = clock();
	printf("10. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	start = clock();
	gpuErrchk(hipMalloc(&(h_d_scene->objects), get_object_array_length(h_scene_to_array->objects)));
	stop = clock();
	printf("9. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	start = clock();
	gpuErrchk(hipMalloc(&(h_d_scene->cameras), sizeof(t_camera)));
	stop = clock();
	printf("11. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	start = clock();
	gpuErrchk(hipMalloc(&d_scene, sizeof(t_scene)));
	stop = clock();
	printf("12. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);

	//printf("Pointer to camera fov, host: %p\n", &(h_d_scene->cameras[0].fov));

	// Copie des tableaux du CPU vers le GPU, en passant par la structure contenant des pointeurs sur GPU
	start = clock();
	gpuErrchk(hipMemcpy(h_d_scene->cameras, h_scene_to_array->cameras, sizeof(t_camera), hipMemcpyHostToDevice));
	stop = clock();
	printf("13. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	start = clock();
	gpuErrchk((hipMemcpy(h_d_scene->objects, h_scene_to_array->objects, get_object_array_length(h_scene_to_array->objects), hipMemcpyHostToDevice)));
	stop = clock();
	printf("14. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	start = clock();
	gpuErrchk(hipMemcpy(h_d_scene->lights, h_scene_to_array->lights, get_lights_array_length(h_scene_to_array->lights), hipMemcpyHostToDevice));
	stop = clock();
	printf("15. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);

	// Copie de la structure finale sur le GPU, contenant les pointeurs GPU
	start = clock();
	gpuErrchk(hipMemcpy(d_scene, h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	stop = clock();
	printf("16. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	// Pixel map
	start = clock();
	h_pixel_map = (t_color *)malloc(sizeof(t_color) * scene->res.y * scene->res.x);
	stop = clock();
	printf("17. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
  	hipMemGetInfo(&available, &total);
 	printf("available memory: [%'lu]\ntotal memory:   [%'lu]\n", available, total);
	start = clock();
	printf("Pixelmap size : %lu\n", sizeof(t_color) * scene->res.y * scene->res.x);
	printf("t_color size : %d\n", scene->res.y);
	gpuErrchk((hipMalloc(&d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x)));
	// hipHostMalloc(&h_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x);
	stop = clock();
	hipMemGetInfo(&available, &total);
 	printf("available memory: [%'lu]\ntotal memory:   [%'lu]\n", available, total);
	printf("18. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);

	dim3 blockSize 	= dim3(BLOCK_DIM, BLOCK_DIM, 1);
	dim3 gridSize	= dim3(scene->res.x / BLOCK_DIM + 1, scene->res.y / BLOCK_DIM + 1);


	printf("gridsize: [%d][%d][%d] blocksize: [%d][%d][%d]\n", gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);
	



	start = clock();
	render_pixel<<<gridSize, blockSize>>>(d_scene, d_pixel_map/*, progress*/);
	gpuErrchk( hipPeekAtLastError() );
	// gpuErrchk( hipDeviceSynchronize() ); // Debug
	gpuErrchk((hipDeviceSynchronize()));
	stop = clock();
	printf("19. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);

	//   printf("!!!%d \n", progress->x);
	//   pthread_join(p0, NULL);
 
	start = clock();
	gpuErrchk(hipMemcpy(h_pixel_map, d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x, hipMemcpyDeviceToHost));
	stop = clock();
	printf("20. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);


	//printf("h_pix_map: %f\n", h_pixel_map[0].z);
	start = clock();
	memcpy(scene->cameras->pixel_map, h_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x);
	
	// in a different thread:
	// export_image(h_pixel_map, scene->res.y, scene->res.x, "image.bmp");

	stop = clock();
	printf("21. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	  //printf("scene->cameras->pixel_map: %f\n", scene->cameras->pixel_map[0].z);


	start = clock();
	hipFree(h_d_scene->cameras);
	hipFree(h_d_scene->lights);
	hipFree(h_d_scene->objects);
	hipFree(d_scene);
	hipFree(d_pixel_map);
	stop = clock();
	printf("22. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);

	start = clock();
	free(h_scene_to_array->objects);
	free(h_scene_to_array->lights);
	free(h_scene_to_array->cameras);
	free(h_scene_to_array);
	free(h_d_scene);
	free(h_pixel_map);
	stop = clock();
	printf("23. Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);


	  int nb;

	  hipGetDeviceCount(&nb);
	  printf("Nb device : %d\n", nb);
}
