#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/22 12:16:17 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"

#include <pthread.h>

/*
** Updates a camera's pixel_map (color of image pixels).
*/
#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void	printte_matrix(t_matrix m)
{
	int i;
	int	y;

	y = -1;
	while (++y < 4)
	{
		i = -1;
		while (++i < 4)
			printf("[%f]", m[y][i]);
		printf("\n");
	}
}

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map/*, t_pt2 *loading*/)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	int					idx;

	r.pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene;
	// printf("%d %d\n", blockIdx.x, blockIdx.y);
	// printf("%d %d\n", threadIdx.x, threadIdx.y);	

    idx = scene->res.x * r.pix.y + r.pix.x;

	if (idx < 1)
	{
		// printf("Coucou\n");
		printf("%f\n", scene->lights[1].col.x);
		// *loading = 20;
	}


	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		cam_ray = init_camera_ray(&r);	
		d_pixel_map[idx] = filter(cast_primary_ray(&r, &cam_ray), scene->cameras->filter);
		// d_pixel_map[idx] = cast_primary_ray(&r, &cam_ray);
	}
	//loading->x = atomicAdd(&(loading->x), 1);
	//__syncthreads();
}



t_object	*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	array = (t_object *)malloc(sizeof(t_object) * (size + 1)); // malloc error
	array[size].type = T_INVALID_TOKEN;
	object = head;
	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object)); // stack memcpy ?
		object = object->next;
	}
	return (array);
}

size_t			get_object_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}

t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light	*head;
	t_light	*array;

	size = 0;
	head = light;
	while (light)
	{
		++size;
		light = light->next;
	}
	array = (t_light *)malloc(sizeof(t_light) * (size + 1)); // malloc error
	// bzero(array, sizeof(t_light) * (size + 1));
	array[size].col = v_new(NAN, NAN, NAN);
	light = head;
	size = -1;
	// printf("COLOR: %f\n", array[size].col.x);
	while (light)
	{
		array[++size].col = v_new(NAN, NAN, NAN);
		memcpy(&array[size], light, sizeof(t_light));
		light = light->next;
		// printf("%p\n", light);
	}
	// printf("COLOR: %f\n", array[size].col.x);
	return (array);
}

size_t			get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	// printf("%f\n", lights[size].col.x);
	while (!v_isnan(lights[size].col))
		++size;
	// C(3)
	return ((size + 1) * sizeof(t_light));
}

		#include <time.h>

// void		*loading_bar(void	*pt)
// {
// 	t_pt2 *max = (t_pt2*) pt;
// 	// usleep(1000000);

// 	while (max->x < max->y / 2) ;
// 		printf("loading : %d\n", max->x);
// 	return NULL;
// }

void		render(t_scene *scene)
{
	t_color		*d_pixel_map;
	t_color 	*h_pixel_map;
	t_scene		*h_scene_to_array;
	t_scene		*h_d_scene;
	t_scene		*d_scene;
	dim3		block_size;
	dim3		grid_size;


	size_t       available;
  	size_t       total;

	hipDeviceSetLimit(hipLimitStackSize, 1024 * scene->ray_depth);

	// Preparation des deux structures temporarires
	if (!(h_scene_to_array = (t_scene *)malloc(sizeof(t_scene))))
		exit(0); // malloc error
	if (!(h_d_scene = (t_scene *)malloc(sizeof(t_scene))))
		exit(0); // malloc error
	if (!(memcpy(h_scene_to_array, scene, sizeof(t_scene))))
		exit(0);
	memcpy(h_d_scene, scene, sizeof(t_scene));

	// Creation des tableaux 1D pour les objets et lumieres
	h_scene_to_array->objects = list_to_array_objects(scene->objects);
	h_scene_to_array->lights = list_to_array_lights(scene->lights);
	h_scene_to_array->cameras = (t_camera *)malloc(sizeof(t_camera));
	gpuErrchk(hipSetDevice(0));
	memcpy(h_scene_to_array->cameras, scene->cameras, sizeof(t_camera));
	
	// Allocation de la memoire GPU
	gpuErrchk(hipMalloc(&(h_d_scene->lights), get_lights_array_length(h_scene_to_array->lights)));
	gpuErrchk(hipMalloc(&(h_d_scene->objects), get_object_array_length(h_scene_to_array->objects)));
	gpuErrchk(hipMalloc(&(h_d_scene->cameras), sizeof(t_camera)));
	gpuErrchk(hipMalloc(&d_scene, sizeof(t_scene)));

	// Copie des tableaux du CPU vers le GPU, en passant par la structure contenant des pointeurs sur GPU
	gpuErrchk(hipMemcpy(h_d_scene->cameras, h_scene_to_array->cameras, sizeof(t_camera), hipMemcpyHostToDevice));
	gpuErrchk((hipMemcpy(h_d_scene->objects, h_scene_to_array->objects, get_object_array_length(h_scene_to_array->objects), hipMemcpyHostToDevice)));
	gpuErrchk(hipMemcpy(h_d_scene->lights, h_scene_to_array->lights, get_lights_array_length(h_scene_to_array->lights), hipMemcpyHostToDevice));

	// Copie de la structure finale sur le GPU, contenant les pointeurs GPU
	gpuErrchk(hipMemcpy(d_scene, h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	
	// Pixel map
	h_pixel_map = (t_color *)malloc(sizeof(t_color) * scene->res.y * scene->res.x);
	
  	hipMemGetInfo(&available, &total);
 	printf("available memory: [%'lu]\ntotal memory:   [%'lu]\n", available, total);
	gpuErrchk((hipMalloc(&d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x)));
	hipMemGetInfo(&available, &total);
 	printf("available memory: [%'lu]\ntotal memory:   [%'lu]\n", available, total);

	 //prep kernel
	dim3 blockSize 	= dim3(BLOCK_DIM, BLOCK_DIM, 1);
	dim3 gridSize	= dim3(scene->res.x / BLOCK_DIM + 1, scene->res.y / BLOCK_DIM + 1);
	// printf("gridsize: [%d][%d][%d] blocksize: [%d][%d][%d]\n", gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);
	render_pixel<<<gridSize, blockSize>>>(d_scene, d_pixel_map/*, progress*/);
	
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk((hipDeviceSynchronize()));
	gpuErrchk(hipMemcpy(h_pixel_map, d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x, hipMemcpyDeviceToHost));
	memcpy(scene->cameras->pixel_map, h_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x);

	//free dat shit
	hipFree(h_d_scene->cameras);
	hipFree(h_d_scene->lights);
	hipFree(h_d_scene->objects);
	hipFree(d_scene);
	hipFree(d_pixel_map);

	//free dis shit
	free(h_scene_to_array->objects);
	free(h_scene_to_array->lights);
	free(h_scene_to_array->cameras);
	free(h_scene_to_array);
	free(h_d_scene);
	free(h_pixel_map);
}
