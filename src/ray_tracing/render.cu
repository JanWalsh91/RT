#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/02 18:17:24 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"
#include "../../inc/cuda_call.h"

// static void	display_percentage(int num);

/*
** Updates a camera's pixel_map (color of image pixels).
*/


// __host__ void	cuda_push_camera(t_camera **cameras_head, t_camera *new_camera)
// {
// 	t_camera	*cam_ptr;

// 	if (new_camera)
// 	{
// 		if (!(*cameras_head))
// 			*cameras_head = new_camera;
// 		else
// 		{
// 			cam_ptr = *cameras_head;
// 			while (cam_ptr->next)
// 				cam_ptr = cam_ptr->next;
// 			cam_ptr->next = new_camera;
// 			new_camera->prev = cam_ptr;
// 		}
// 		new_camera->next = NULL;
// 	}
// }
/*
__host__ void	allocate_cameras(t_camera **d_cams, t_camera *h_cams, t_pt2 res)
{
	t_camera *p_cam;
	t_camera *new_cam;

	new_cam = NULL;
	*d_cams = NULL;
	p_cam = h_cams;

	hipMallocManaged(&new_cam, sizeof(t_camera));
	memcpy(new_cam, p_cam, sizeof(t_camera));
	p_cam = p_cam->next;
}

__host__ void	cuda_push_light(t_light **lights_head, t_light *new_light)
{
	t_light	*light_ptr;

	if (new_light)
	{
		if (!(*lights_head))
			*lights_head = new_light;
		else
		{
			light_ptr = *lights_head;
			while (light_ptr->next)
				light_ptr = light_ptr->next;
			light_ptr->next = new_light;
		}
		new_light->next = NULL;
	}
}*/
/*
__host__ void 	allocate_lights(t_light **d_lights, t_light *h_lights)
{
	t_light *p_light;
	t_light *new_light;

	new_light = NULL;
	*d_lights = NULL;
	p_light = h_lights;
	while (p_light)
	{
		hipMallocManaged(&new_light, sizeof(t_light));
		memcpy(new_light, p_light, sizeof(t_light));
		cuda_push_light(d_lights, new_light);
		p_light = p_light->next;
	}
}
*//*
__host__ void	cuda_push_object(t_object **objects_head, t_object *new_object)
{
	t_object	*obj_ptr;

	if (new_object)
	{
		if (!(*objects_head))
			*objects_head = new_object;
		else
		{
			obj_ptr = *objects_head;
			while (obj_ptr->next)
				obj_ptr = obj_ptr->next;
			obj_ptr->next = new_object;
		}
		new_object->next = NULL;
	}
}*/
/*
__host__ void	allocate_objects(t_object **d_objs, t_object *h_objs)
{
	t_object *p_obj;
	t_object *new_obj;

	new_obj = NULL;
	*d_objs = NULL;
	p_obj = h_objs;
	while (p_obj)
	{
		hipMallocManaged(&new_obj, sizeof(t_object));
		memcpy(new_obj, p_obj, sizeof(t_object));
		cuda_push_object(d_objs, new_obj);
		p_obj = p_obj->next;
	}
}
*/
// __host__
// void	cuda_push_scene(t_scene **scenes_head, t_scene *new_scene)
// {
// 	t_scene	*scene_ptr;

// 	if (new_scene)
// 	{
// 		if (!*scenes_head)
// 			*scenes_head = new_scene;
// 		else
// 		{
// 			scene_ptr = *scenes_head;
// 			while (scene_ptr->next)
// 				scene_ptr = scene_ptr->next;
// 			scene_ptr->next = new_scene;
// 			new_scene->prev = scene_ptr;
// 		}
// 	}
// }
/*
__host__ void	allocate_scenes(t_raytracing_tools *d_r, t_raytracing_tools *h_r)
{
	t_scene	*p_scene;
	t_scene	*new_scene;

	new_scene = NULL;
	d_r->scenes = NULL;
	p_scene = h_r->scenes;
	while (p_scene)
	{
		//allocate mem
		hipMallocManaged(&new_scene, sizeof(t_scene));
		memcpy(new_scene, p_scene, sizeof(t_scene));
		if (!d_r->scenes)
			d_r->scenes = new_scene;
		//allocate cameras
		allocate_cameras(&new_scene->cameras, p_scene->cameras, p_scene->res);
		//allocate lights
		allocate_lights(&new_scene->lights, p_scene->lights);
		//allocate objects
		allocate_objects(&new_scene->objects, p_scene->objects);
		cuda_push_scene(&d_r->scenes, new_scene);
		p_scene = p_scene->next;
	}
}*/
/*
__host__ void allocate_camera(t_scene *h_scene)
{
	t_camera	*h_camera;
	t_camera	*d_camera;
	t_color		**d_pixel_map;
	int			i;

	h_camera = (t_camera *)malloc(sizeof(t_camera));
	memcpy(h_camera, h_scene->cameras, sizeof(t_camera));

	hipMalloc(&d_pixel_map, sizeof(t_color *) * h_scene->res.y);
	i = -1;
	while (++i < h_scene->res.x)
		hipMalloc(&(d_pixel_map[i]), sizeof(t_color) * h_scene->res.x);
	h_camera->pixel_map = d_pixel_map;

	hipMalloc(&d_camera, sizeof(t_camera));
	hipMemcpy(d_camera, h_camera, sizeof(t_camera), hipMemcpyHostToDevice);
}

__host__ void	allocate_lights(t_scene *h_scene)
{
	t_light *p_light;
	t_light *h_light;
	t_light *d_light;

	p_light = h_scene->lights;
	while (p_light)
	{
		//hipMallocManaged(&new_light, sizeof(t_light));
		//memcpy(new_light, p_light, sizeof(t_light));
		//cuda_push_light(d_lights, new_light);
		
		h_light = (t_light *)malloc(sizeof(t_light));
		memcpy(h_light, p_light, sizeof(t_light));

		hipMalloc(&d_light, sizeof(t_light));
		h_light->next = NULL;
		hipMemcpy(d_light, h_light, sizeof(t_light), hipMemcpyHostToDevice);
		
		p_light = p_light->next;
	}
}

__host__ t_object	*allocate_objects(t_object *h_scene)
{
	t_object *p_object;
	t_object *h_object;
	t_object *d_object;

	p_object = h_scene->objects;
	if (p_object)
	{
		h_object = (t_object *)malloc(sizeof(t_object));
		memcpy(h_object, p_object, sizeof(t_object));
		allocate_objects(h_scene);
		hipMalloc(&d_object, sizeof(t_object));
		h_object->next = NULL;
		hipMemcpy(d_object, h_object, sizeof(t_object), hipMemcpyHostToDevice);
		
		p_object = p_object->next;
	}
}

__host__ void allocate_scene(t_raytracing_tools *h_r_tmp)
{
	t_scene	*h_scene;
	t_scene	*d_scene;

	h_scene = (t_scene *)malloc(sizeof(t_scene));
	memcpy(h_scene, h_r_tmp->scenes, sizeof(t_scene));

	allocate_camera(h_scene);
	allocate_lights(h_scene);
	h_r_tmp->objects = allocate_objects(h_r_tmp->objects);

	hipMalloc(&d_scene, sizeof(t_scene));
	h_r_tmp->scenes = d_scene;
	hipMemcpy(d_scene, h_scene, sizeof(t_scene), hipMemcpyHostToDevice);
}

__host__ __device__ t_raytracing_tools  *allocate_memory(t_raytracing_tools *h_r)
{
	C(11)
	t_raytracing_tools	*d_r;
	t_raytracing_tools	*h_r_tmp;

	h_r_tmp = (t_raytracing_tools *)malloc(sizeof(t_raytracing_tools));
	memcpy(h_r_tmp, h_r, sizeof(t_raytracing_tools));

	allocate_scenes(d_r, h_r);

	hipMalloc(&d_r, sizeof(t_raytracing_tools));
	hipMemcpy(d_r, h_r_tmp, sizeof(t_raytracing_tools), hipMemcpyHostToDevice);
	// printf("%d\n", d_r->pix.x);
	C(12)
	// memcpy(d_r, h_r, sizeof(t_raytracing_tools));
	C(13)
	return (d_r);
}*/

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map)
{
	t_ray	cam_ray;
	t_pt2 	pixel;
	int x;
	int y;


	x = (blockDim.x * blockIdx.x) + threadIdx.x;
	y = (blockDim.y * blockIdx.y) + threadIdx.y;
	pixel.x = x;
	pixel.y = y;

	// printf("Coucou : %d\n", r->scenes->res.x * y + x);
    int idx = 1000 * y + x;
	cam_ray = init_camera_ray(pixel, scene);
	if (idx < 1000 * 1000)
		d_pixel_map[idx] = cast_primary_ray(scene, &cam_ray);
}

t_object	*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	array = (t_object *)malloc(sizeof(t_object) * size + 1); // malloc error
	array[size].type = T_INVALID_TOKEN;
	object = head;
	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object)); // stack memcpy ?
		object = object->next;
	}
	return (array);
}

size_t			get_object_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}

t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light	*head;
	t_light	*array;

	size = 0;
	head = light;
	while (light)
	{
		++size;
		light = light->next;
	}
	array = (t_light *)malloc(sizeof(t_light) * size + 1); // malloc error
	array[size].col.x = -1;
	light = head;
	size = -1;
	while (light)
	{
		memcpy(&array[++size], light, sizeof(t_light));
		light = light->next;
	}
	return (array);
}

size_t			get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	while (lights[size].col.x != -1)
		++size;
	return ((size + 1) * sizeof(t_light));
}

__host__
void		render(t_scene *scene)
{
	t_color	*d_pixel_map;
	t_color *h_pixel_map;

	t_scene		*h_scene;
	t_scene		*d_scene;

	t_object	*d_objects;
	t_light		*d_lights;
	t_camera	*d_camera;

	h_scene = (t_scene *)malloc(sizeof(t_scene)); // malloc error
	h_scene->objects = list_to_array_objects(scene->objects);
	h_scene->lights = list_to_array_lights(scene->lights);
	h_scene->cameras = (t_camera *)malloc(sizeof(t_camera));
	memcpy(h_scene->cameras, scene->cameras, sizeof(t_camera));

	hipMalloc(&d_scene, sizeof(t_scene *));
	hipMalloc(&d_objects, get_object_array_length(h_scene->objects));
	hipMalloc(&d_lights, get_lights_array_length(h_scene->lights));
	hipMalloc(&d_camera, sizeof(t_camera));

	hipMemcpy(d_objects, h_scene->objects, get_object_array_length(h_scene->objects), hipMemcpyHostToDevice);
	hipMemcpy(d_lights, h_scene->lights, get_lights_array_length(h_scene->lights), hipMemcpyHostToDevice);
	hipMemcpy(d_camera, h_scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice);

	h_pixel_map = (t_color *)malloc(sizeof(t_color) * scene->res.y * scene->res.x);
	hipMalloc((void**) &d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x);

	dim3 block_size;
	dim3 grid_size;

	dim3 blockSize = dim3(32, 32, 1);
	dim3 gridSize = dim3(scene->res.x / 32 + 1, scene->res.y / 32 + 1);
	render_pixel<<<gridSize, blockSize>>>(d_scene, d_pixel_map);
	hipDeviceSynchronize();

	hipMemcpy(h_pixel_map, d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x, hipMemcpyDeviceToHost);

	// Copie la pixel map cree a partir du device dans la scene courante. A modifier, buerk, caca!
	t_pt2 coord;
	coord.y = -1;
	while (++coord.y < scene->res.y)
	{
		coord.x = -1;
		while (++coord.x < scene->res.x)
		{
			scene->cameras->pixel_map[coord.y][coord.x] = h_pixel_map[scene->res.y * coord.x + coord.y];
		}
	}

	free(h_scene->objects);
	free(h_scene->lights);
	free(h_scene->cameras);
	free(h_scene);
	free(h_pixel_map);
	hipFree(d_objects);
	hipFree(d_lights);
	hipFree(d_camera);
	hipFree(d_scene);
	hipFree(d_pixel_map);

	// Malloc une scene sur le CPU
	// Malloc tableau d'objets sur le CPU
	// Malloc tableau de lights sur le CPU
	// Malloc camera CPU
	// Malloc tableau d'objets sur le GPU
	// Malloc tableau de lights sur le GPU
	// Malloc camera GPU
	// Copier tout du CPU au GPU
	// Malloc une scene sur le GPU
	// Copie la scene CPU sur le GPU
	// Lance le kernel
	// Free les listes GPU a partir de scene CPU
	// Free la scene GPU
	// Free la scene CPU et ses objets




















}
