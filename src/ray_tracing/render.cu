#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/03 18:05:56 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"
#include "../../inc/cuda_call.h"

// static void	display_percentage(int num);

/*
** Updates a camera's pixel_map (color of image pixels).
*/
#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void	printte_matrix(t_matrix m)
{
	int i;
	int	y;

	y = -1;
	while (++y < 4)
	{
		i = -1;
		while (++i < 4)
			printf("[%f]", m[y][i]);
		printf("\n");
	}
}

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map)
{
	t_ray	cam_ray;
	t_pt2 	pixel;
	int 	x;
	int 	y;
	int		idx;


	x = (blockDim.x * blockIdx.x) + threadIdx.x;
	y = (blockDim.y * blockIdx.y) + threadIdx.y;

	// printf("%d %d\n", blockIdx.x, blockIdx.y);
	// printf("%d %d\n", threadIdx.x, threadIdx.y);	

    idx = scene->res.x * y + x;

	// d_pixel_map[idx].x = 0;
	// d_pixel_map[idx].y = 0;
	// d_pixel_map[idx].z = 0;
	if (idx < scene->res.x * scene->res.y)
	{
		printf("%d\n", idx);
		pixel.x = x;
		pixel.y = y;
		scene->pix.x = x;
		scene->pix.y = y;
		// printf("%d, %d\n", blockIdx.x, blockIdx.y);
		cam_ray = init_camera_ray(pixel, scene);
		__syncthreads();
		
		// if (idx < 1)
		// {
		// 	printf("Pointer to camera fov, device: %p\n", &(scene->cameras[0].fov));
		// 	printf("Camera fov in the device: %f\n", scene->cameras[0].fov);
		// 	printf("Scene res : [%d, %d]\n", scene->res.x, scene->res.y);
		// 	printf("Scene ray depth: %d\n", scene->ray_depth);
		// 	printf("Scene background color: %f, %f, %f\n", scene->background_color.x, scene->background_color.y, scene->background_color.z);
		// 	printf("Scene ka: %f\n", scene->ka);
		// 	printf("Scene ka: %f\n", scene->image_aspect_ratio);
		// 	printf("Scene camera pos: %f, %f, %f\n", scene->cameras[0].pos.x, scene->cameras[0].pos.y, scene->cameras[0].pos.z);
		// 	printf("Scene first obj pos: %f, %f, %f\n", scene->objects[0].pos.x, scene->objects[0].pos.y, scene->objects[0].pos.z);
		// 	printf("Scene first light pos: %f, %f, %f\n", scene->lights[0].pos.x, scene->lights[0].pos.y, scene->lights[0].pos.z);

		// 	// pointer access to variable
		// 	// t_camera	*cam;

		// 	// init_camera_ray(pixel, scene);
		// 	// cam = &scene->cameras[0];

		// 	// printte_matrix(cam->ctw);

		// 	// printf("Cameras by pointer value : \n");
			
		// 	// printf("Scene camera pos: %f, %f, %f\n", cam->pos.x, cam->pos.y, cam->pos.z);

		// }
			// printf("%d\n", scene->lights[0].intensity);


		// if (x == 600 && y == 600)
		t_color col = cast_primary_ray(scene, &cam_ray);
		
		// t_color col = v_new(0, 0, 0);
		// if (200 < x && x < 300)
		// {	
		// 	col.x = idx % 255;
		// 	col.y = 255;
		// 	col.z = 255;
		// }
		__syncthreads();
		d_pixel_map[idx] = col;
		__syncthreads();
			// d_pixel_map[idx].y = 0;//cast_primary_ray(scene, &cam_ray);
		// d_pixel_map[idx].x = 255;
		// d_pixel_map[idx].y = 255;
		// d_pixel_map[idx].z = 255;
			// d_pixel_map[idx].z = 0;//cast_primary_ray(scene, &cam_ray);
			// if (d_pixel_map[idx].x > 0.0001)
				// printf("%f\n", d_pixel_map[idx].x);
	}
	// printf("%d %d\n", x, y);
}



t_object	*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	array = (t_object *)malloc(sizeof(t_object) * size + 1); // malloc error
	array[size].type = T_INVALID_TOKEN;
	object = head;
	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object)); // stack memcpy ?
		object = object->next;
	}
	return (array);
}

size_t			get_object_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}

t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light	*head;
	t_light	*array;

	size = 0;
	head = light;
	while (light)
	{
		++size;
		light = light->next;
	}
	array = (t_light *)malloc(sizeof(t_light) * size + 1); // malloc error
	array[size].col.x = -1;
	light = head;
	size = -1;
	// printf("COLOR: %f\n", array[size].col.x);
	while (light)
	{
		memcpy(&array[++size], light, sizeof(t_light));
		light = light->next;
		// printf("%p\n", light);
	}
	// printf("COLOR: %f\n", array[size].col.x);
	return (array);
}

size_t			get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	// printf("%f\n", lights[size].col.x);
	// while (lights[size].col.x != -1)
		// ++size;

	return ((size + 1) * sizeof(t_light));
}

__host__
void		render(t_scene *scene)
{
	t_color	*d_pixel_map;
	t_color *h_pixel_map;

	t_scene		*h_scene_to_array;
	t_scene		*h_d_scene;
	t_scene		*d_scene;

	t_object	*d_objects;
	t_light		*d_lights;
	t_camera	*d_camera;

	// Preparation des deux structures temporarires
	h_scene_to_array = (t_scene *)malloc(sizeof(t_scene)); // malloc error
	h_d_scene = (t_scene *)malloc(sizeof(t_scene)); // malloc error
	memcpy(h_scene_to_array, scene, sizeof(t_scene));
	memcpy(h_d_scene, scene, sizeof(t_scene));

	// Creation des tableaux 1D pour les objets et lumieres
	h_scene_to_array->objects = list_to_array_objects(scene->objects);
	h_scene_to_array->lights = list_to_array_lights(scene->lights);
	h_scene_to_array->cameras = (t_camera *)malloc(sizeof(t_camera));
	memcpy(h_scene_to_array->cameras, scene->cameras, sizeof(t_camera));

	// Allocation de la memoire GPU
	hipMalloc(&(h_d_scene->objects), get_object_array_length(h_scene_to_array->objects));
	hipMalloc(&(h_d_scene->lights), get_lights_array_length(h_scene_to_array->lights));
	hipMalloc(&(h_d_scene->cameras), sizeof(t_camera));
	hipMalloc(&d_scene, sizeof(t_scene));

	printf("Pointer to camera fov, host: %p\n", &(h_d_scene->cameras[0].fov));

	// Copie des tableaux du CPU vers le GPU, en passant par la structure contenant des pointeurs sur GPU
	hipMemcpy(h_d_scene->cameras, h_scene_to_array->cameras, sizeof(t_camera), hipMemcpyHostToDevice);
	hipMemcpy(h_d_scene->objects, h_scene_to_array->objects, get_object_array_length(h_scene_to_array->objects), hipMemcpyHostToDevice);
	hipMemcpy(h_d_scene->lights, h_scene_to_array->lights, get_lights_array_length(h_scene_to_array->lights), hipMemcpyHostToDevice);

	// Copie de la structure finale sur le GPU, contenant les pointeurs GPU
	hipMemcpy(d_scene, h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice);

	// Pixel map
	h_pixel_map = (t_color *)malloc(sizeof(t_color) * scene->res.y * scene->res.x);
	hipMalloc(&d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x);

	dim3 block_size;
	dim3 grid_size;

	int div = 16;

	dim3 gridSize= dim3(scene->res.x / div, scene->res.y);
	dim3 blockSize= dim3(div, 1, 1);
	render_pixel<<<gridSize, blockSize>>>(d_scene, d_pixel_map);
	//gpuErrchk( hipPeekAtLastError() ); // Debug
	// gpuErrchk( hipDeviceSynchronize() ); // Debug
	hipDeviceSynchronize();
	C(666)
	hipMemcpy(h_pixel_map, d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x, hipMemcpyDeviceToHost);
	
	printf("h_pix_map: %f\n", h_pixel_map[0].z);
	memcpy(scene->cameras->pixel_map, h_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x);
	printf("scene->cameras->pixel_map: %f\n", scene->cameras->pixel_map[0].z);

	t_pt2 loop;
	loop.y = -1;
	// while (++loop.y < scene->res.y)
	// {
		// loop.x = -1;
		while (++loop.y < scene->res.y)
		{
			fflush(stdout);
			printf("%f - ", scene->cameras->pixel_map[loop.y * scene->res.x + 200].z);
		}
		printf("\n");
	// }

	// t_pt2 coord;
	// coord.y = -1;
	// while (++coord.y < scene->res.y)
	// {
	// 	coord.x = -1;
	// 	while (++coord.x < scene->res.x)
	// 	{
	// 		scene->cameras->pixel_map[coord.y][coord.x] = h_pixel_map[scene->res.x * coord.y + coord.x];
	// 	}
	// }

/*	free(h_scene->objects);
	free(h_scene->lights);
	free(h_scene->cameras);
	free(h_scene);
	free(h_pixel_map);
	hipFree(d_objects);
	hipFree(d_lights);
	hipFree(d_camera);
	hipFree(d_scene);
	hipFree(d_pixel_map);
*/
	// Malloc une scene sur le CPU
	// Malloc tableau d'objets sur le CPU
	// Malloc tableau de lights sur le CPU
	// Malloc camera CPU
	// Malloc tableau d'objets sur le GPU
	// Malloc tableau de lights sur le GPU
	// Malloc camera GPU
	// Copier tout du CPU au GPU
	// Malloc une scene sur le GPU
	// Copie la scene CPU sur le GPU
	// Lance le kernel
	// Free les listes GPU a partir de scene CPU
	// Free la scene GPU
	// Free la scene CPU et ses objets
}
