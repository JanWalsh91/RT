#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/04 13:38:12 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"
#include "../../inc/cuda_call.h"


/*
** Updates a camera's pixel_map (color of image pixels).
*/
#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void	printte_matrix(t_matrix m)
{
	int i;
	int	y;

	y = -1;
	while (++y < 4)
	{
		i = -1;
		while (++i < 4)
			printf("[%f]", m[y][i]);
		printf("\n");
	}
}

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	int					idx;

	r.pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scenes = scene;
	
	// printf("%d %d\n", blockIdx.x, blockIdx.y);
	// printf("%d %d\n", threadIdx.x, threadIdx.y);	

    idx = scene->res.x * r.pix.y + r.pix.x;

	if (idx < 1)
	{
		// printf("Coucou\n");
		printf("%f\n", scene->lights[1].col.x);
	}

	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		cam_ray = init_camera_ray(&r);	
		d_pixel_map[idx] = cast_primary_ray(&r, &cam_ray);
	}
}



t_object	*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	array = (t_object *)malloc(sizeof(t_object) * (size + 1)); // malloc error
	array[size].type = T_INVALID_TOKEN;
	object = head;
	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object)); // stack memcpy ?
		object = object->next;
	}
	return (array);
}

size_t			get_object_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}

t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light	*head;
	t_light	*array;

	size = 0;
	head = light;
	while (light)
	{
		++size;
		light = light->next;
	}
	array = (t_light *)malloc(sizeof(t_light) * (size + 1)); // malloc error
	bzero(array, sizeof(t_light) * (size + 1));
	array[size].col = v_new(NAN, NAN, NAN);
	light = head;
	size = -1;
	printf("COLOR: %f\n", array[size].col.x);
	while (light)
	{
		array[++size].col = v_new(NAN, NAN, NAN);
		memcpy(&array[size], light, sizeof(t_light));
		light = light->next;
		// printf("%p\n", light);
	}
	// printf("COLOR: %f\n", array[size].col.x);
	return (array);
}

size_t			get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	// printf("%f\n", lights[size].col.x);
	while (!v_isnan(lights[size].col))
		++size;
	C(3)
	return ((size + 1) * sizeof(t_light));
}

		#include <time.h>

__host__
void		render(t_scene *scene)
{
	t_color		*d_pixel_map;
	t_color 	*h_pixel_map;
	t_scene		*h_scene_to_array;
	t_scene		*h_d_scene;
	t_scene		*d_scene;
	dim3		block_size;
	dim3		grid_size;

	clock_t	start = clock();
	// Preparation des deux structures temporarires
	h_scene_to_array = (t_scene *)malloc(sizeof(t_scene)); // malloc error
	h_d_scene = (t_scene *)malloc(sizeof(t_scene)); // malloc error
	memcpy(h_scene_to_array, scene, sizeof(t_scene));
	memcpy(h_d_scene, scene, sizeof(t_scene));

	// Creation des tableaux 1D pour les objets et lumieres
	h_scene_to_array->objects = list_to_array_objects(scene->objects);
	h_scene_to_array->lights = list_to_array_lights(scene->lights);
	h_scene_to_array->cameras = (t_camera *)malloc(sizeof(t_camera));
	memcpy(h_scene_to_array->cameras, scene->cameras, sizeof(t_camera));

	// Allocation de la memoire GPU
	hipMalloc(&(h_d_scene->objects), get_object_array_length(h_scene_to_array->objects));
	hipMalloc(&(h_d_scene->lights), get_lights_array_length(h_scene_to_array->lights));
	hipMalloc(&(h_d_scene->cameras), sizeof(t_camera));
	hipMalloc(&d_scene, sizeof(t_scene));

	//printf("Pointer to camera fov, host: %p\n", &(h_d_scene->cameras[0].fov));

	// Copie des tableaux du CPU vers le GPU, en passant par la structure contenant des pointeurs sur GPU
	hipMemcpy(h_d_scene->cameras, h_scene_to_array->cameras, sizeof(t_camera), hipMemcpyHostToDevice);
	hipMemcpy(h_d_scene->objects, h_scene_to_array->objects, get_object_array_length(h_scene_to_array->objects), hipMemcpyHostToDevice);
	hipMemcpy(h_d_scene->lights, h_scene_to_array->lights, get_lights_array_length(h_scene_to_array->lights), hipMemcpyHostToDevice);

	// Copie de la structure finale sur le GPU, contenant les pointeurs GPU
	hipMemcpy(d_scene, h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice);

	// Pixel map
	h_pixel_map = (t_color *)malloc(sizeof(t_color) * scene->res.y * scene->res.x);
	hipMalloc(&d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x);

	dim3 blockSize 	= dim3(BLOCK_DIM, BLOCK_DIM, 1);
	dim3 gridSize	= dim3(scene->res.x / BLOCK_DIM + 1, scene->res.y / BLOCK_DIM + 1);


	render_pixel<<<gridSize, blockSize>>>(d_scene, d_pixel_map);
	
	gpuErrchk( hipPeekAtLastError() ); // Debug
	// gpuErrchk( hipDeviceSynchronize() ); // Debug
	hipDeviceSynchronize();
	C(666)
	hipMemcpy(h_pixel_map, d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x, hipMemcpyDeviceToHost);
	
	//printf("h_pix_map: %f\n", h_pixel_map[0].z);
	memcpy(scene->cameras->pixel_map, h_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x);
	//printf("scene->cameras->pixel_map: %f\n", scene->cameras->pixel_map[0].z);

	hipFree(h_d_scene->cameras);
	hipFree(h_d_scene->lights);
	hipFree(h_d_scene->objects);
	hipFree(d_scene);
	hipFree(d_pixel_map);

	free(h_scene_to_array->objects);
	free(h_scene_to_array->lights);
	free(h_scene_to_array->cameras);
	free(h_scene_to_array);
	free(h_d_scene);
	free(h_pixel_map);

	clock_t stop = clock();
	printf("Time taken %f milliseconds\n",
  	(float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
}
