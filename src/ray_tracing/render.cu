#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/26 20:43:59 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"

/*
** Updates a camera's pixel_map (color of image pixels).
*/

#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void	printte_matrix(t_matrix m)
{
	int i;
	int	y;

	y = -1;
	while (++y < 4)
	{
		i = -1;
		while (++i < 4)
			printf("[%f]", m[y][i]);
		printf("\n");
	}
}

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map, t_pt2 tileId, int tile_size)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	t_dpt2				aa_i;
	float				sample_size;
	int					i;
	t_vec3				moyenne;

	r.pix.x = (tileId.x * tile_size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (tileId.y * tile_size) + (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene;
    r.idx = scene->res.x * r.pix.y + r.pix.x;

	// if (r.idx == 0)
	// {
	// 	t_vec3 q;
	// 	float d;
	// 	t_vec3im sol;
		
	// 	q.x = 1;
	// 	q.y = 2;
	// 	q.z = 3;
	// 	d = 4;
		//test polynomial functions
		// if (solve_cubic(q, d, &sol));
		// 	printf("cubic solutions: %f, %f, %f\n", sol.x.r, sol.y.r, sol.z.r);
		// exit(0);
	// }
	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		// initialize ior list
		// r.ior_list = (float *)malloc(sizeof(float) * (scene->ray_depth + 1));
		// printf("AA de la scene : %d\n", scene->is_aa);
		if (scene->is_aa == 1)
		{
			aa_i.x = 0.5;
			aa_i.y = 0.5;
			memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
			cam_ray = init_camera_ray(&r, aa_i);
			d_pixel_map[r.idx] = filter(cast_primary_ray(&r, &cam_ray), scene->cameras->filter);
		}
		else
		{
			sample_size =  1 / (float)scene->is_aa;
			aa_i.x = 0.0;
			aa_i.y = 0.0;
			i = -1;
			moyenne.x = 0;
			moyenne.y = 0;
			moyenne.z = 0;
			while (++i < scene->is_aa * scene->is_aa)
			{
				aa_i.x += sample_size;
				if (i % (scene->is_aa - 1) == 0)
				{
					aa_i.x = 0.0;
					aa_i.y += sample_size;
				}
				memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
				cam_ray = init_camera_ray(&r, aa_i);
				moyenne = v_add(moyenne, col_to_vec(cast_primary_ray(&r, &cam_ray)));
			}
			moyenne.x /= (scene->is_aa * scene->is_aa);
			moyenne.y /= (scene->is_aa * scene->is_aa);
			moyenne.z /= (scene->is_aa * scene->is_aa);
			d_pixel_map[r.idx] = filter(vec_to_col(moyenne), scene->cameras->filter);
		}
	}
}

//'dis is wonderful
__global__ void create_anaglyph(t_color *left, t_color *right, t_scene *scene, int tile_size, t_pt2 tileId)
{
	int		idx;
	t_pt2	pixel;

	pixel.x = (tileId.x * tile_size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	pixel.y = (tileId.y * tile_size) + (blockDim.y * blockIdx.y) + threadIdx.y;
  	idx = scene->res.x * pixel.y + pixel.x;

	if (pixel.x < scene->res.x && pixel.y < scene->res.y)
	{
		left[idx].g = right[idx].g;
		left[idx].b = right[idx].b;
	}
	// __syncthreads();
}

// Trouver un moyen pour appeler cette fonction ><
void	update_camera(t_camera *camera)
{
	t_vec3	forward;
	t_vec3	right;
	t_vec3	up;

	forward = v_norm(camera->dir);
	if (v_dot(forward, v_new(0, 1, 0)) > 0.9999 ||
		v_dot(forward, v_new(0, 1, 0)) < -0.9999)
		right = v_new(1, 0, 0);
	else
		right = v_norm(v_cross(v_new(0, 1, 0), forward));
	up = v_norm(v_cross(forward, right));
	m_new_identity(&camera->ctw);
	camera->ctw[0][0] = right.x;
	camera->ctw[0][1] = right.y;
	camera->ctw[0][2] = right.z;
	camera->ctw[1][0] = up.x;
	camera->ctw[1][1] = up.y;
	camera->ctw[1][2] = up.z;
	camera->ctw[2][0] = forward.x;
	camera->ctw[2][1] = forward.y;
	camera->ctw[2][2] = forward.z;
	camera->ctw[3][0] = camera->pos.x;
	camera->ctw[3][1] = camera->pos.y;
	camera->ctw[3][2] = camera->pos.z;
}

void		render(t_raytracing_tools *r, t_pt2 tileId)
{
	dim3 		blockSize;
	dim3 		gridSize;
	int			size;

	size = (r->settings.tile_size / BLOCK_DIM) + ((r->settings.tile_size % BLOCK_DIM) ? 1 : 0);
	blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	gridSize = dim3(size, size);

	hipEvent_t start, stop;
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	hipEventRecord(start);
	printf("launch kernel:\n");
	render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map, tileId, r->settings.tile_size);
	// printf("Iteration i = %d	\n", i++);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);


	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	if (errSync != hipSuccess || errAsync != hipSuccess)
		exit(-1);
	//beautiful....
	// printf("=============== EXECUTION ================== \n");
	// printf("Kernel duration: %f milliseconds\n", milliseconds);
	// printf("============================================ \n");

	// gpuErrchk((hipDeviceSynchronize()));

	if (r->scene->is_3d)
	{
		printf("3d\n");
		r->scene->cameras->pos.x += 0.05;
		r->scene->cameras->dir.x -= 0.01;
		r->scene->cameras->dir = v_norm(r->scene->cameras->dir);
		update_camera(r->scene->cameras);
		r->scene->cameras->filter = F_RIGHT_CYAN;
		gpuErrchk(hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice));
		gpuErrchk((hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice)));
		render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map_3d, tileId, r->settings.tile_size);
		gpuErrchk((hipDeviceSynchronize()));
		r->scene->cameras->pos.x -= 0.05;
		r->scene->cameras->dir.x += 0.01;
		r->scene->cameras->dir = v_norm(r->scene->cameras->dir);
		update_camera(r->scene->cameras);
		r->scene->cameras->filter = F_LEFT_RED;
		gpuErrchk(hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice));
		gpuErrchk((hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice)));
		create_anaglyph<<<gridSize, blockSize>>>(r->d_pixel_map, r->d_pixel_map_3d, r->d_scene, r->settings.tile_size, tileId);
		gpuErrchk((hipDeviceSynchronize()));
	}

}
