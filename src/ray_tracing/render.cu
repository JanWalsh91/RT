#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/02 13:58:07 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rtv1.cuh"
#include "../../inc/cuda_call.h"

// static void	display_percentage(int num);

/*
** Updates a camera's pixel_map (color of image pixels).
*/


__host__ void	cuda_push_camera(t_camera **cameras_head, t_camera *new_camera)
{
	t_camera	*cam_ptr;

	if (new_camera)
	{
		if (!(*cameras_head))
			*cameras_head = new_camera;
		else
		{
			cam_ptr = *cameras_head;
			while (cam_ptr->next)
				cam_ptr = cam_ptr->next;
			cam_ptr->next = new_camera;
			new_camera->prev = cam_ptr;
		}
		new_camera->next = NULL;
	}
}

__host__ void	allocate_cameras(t_camera **d_cams, t_camera *h_cams, t_pt2 res)
{
	t_camera *p_cam;
	t_camera *new_cam;
	int		i;

	new_cam = NULL;
	*d_cams = NULL;
	p_cam = h_cams;
	while (p_cam)
	{
		hipMallocManaged(&new_cam, sizeof(t_camera));
		memcpy(new_cam, p_cam, sizeof(t_camera));
		i = -1;
		hipMallocManaged(&new_cam->pixel_map, sizeof(t_color *) * res.y);
		while (++i < res.y)
			hipMallocManaged(&new_cam->pixel_map[i], sizeof(t_color) * res.x);
		cuda_push_camera(d_cams, new_cam);
		p_cam = p_cam->next;
	}
}

__host__ void	cuda_push_light(t_light **lights_head, t_light *new_light)
{
	t_light	*light_ptr;

	if (new_light)
	{
		if (!(*lights_head))
			*lights_head = new_light;
		else
		{
			light_ptr = *lights_head;
			while (light_ptr->next)
				light_ptr = light_ptr->next;
			light_ptr->next = new_light;
		}
		new_light->next = NULL;
	}
}
/*
__host__ void 	allocate_lights(t_light **d_lights, t_light *h_lights)
{
	t_light *p_light;
	t_light *new_light;

	new_light = NULL;
	*d_lights = NULL;
	p_light = h_lights;
	while (p_light)
	{
		hipMallocManaged(&new_light, sizeof(t_light));
		memcpy(new_light, p_light, sizeof(t_light));
		cuda_push_light(d_lights, new_light);
		p_light = p_light->next;
	}
}
*/
__host__ void	cuda_push_object(t_object **objects_head, t_object *new_object)
{
	t_object	*obj_ptr;

	if (new_object)
	{
		if (!(*objects_head))
			*objects_head = new_object;
		else
		{
			obj_ptr = *objects_head;
			while (obj_ptr->next)
				obj_ptr = obj_ptr->next;
			obj_ptr->next = new_object;
		}
		new_object->next = NULL;
	}
}
/*
__host__ void	allocate_objects(t_object **d_objs, t_object *h_objs)
{
	t_object *p_obj;
	t_object *new_obj;

	new_obj = NULL;
	*d_objs = NULL;
	p_obj = h_objs;
	while (p_obj)
	{
		hipMallocManaged(&new_obj, sizeof(t_object));
		memcpy(new_obj, p_obj, sizeof(t_object));
		cuda_push_object(d_objs, new_obj);
		p_obj = p_obj->next;
	}
}
*/
__host__
void	cuda_push_scene(t_scene **scenes_head, t_scene *new_scene)
{
	t_scene	*scene_ptr;

	if (new_scene)
	{
		if (!*scenes_head)
			*scenes_head = new_scene;
		else
		{
			scene_ptr = *scenes_head;
			while (scene_ptr->next)
				scene_ptr = scene_ptr->next;
			scene_ptr->next = new_scene;
			new_scene->prev = scene_ptr;
		}
	}
}
/*
__host__ void	allocate_scenes(t_raytracing_tools *d_r, t_raytracing_tools *h_r)
{
	t_scene	*p_scene;
	t_scene	*new_scene;

	new_scene = NULL;
	d_r->scenes = NULL;
	p_scene = h_r->scenes;
	while (p_scene)
	{
		//allocate mem
		hipMallocManaged(&new_scene, sizeof(t_scene));
		memcpy(new_scene, p_scene, sizeof(t_scene));
		if (!d_r->scenes)
			d_r->scenes = new_scene;
		//allocate cameras
		allocate_cameras(&new_scene->cameras, p_scene->cameras, p_scene->res);
		//allocate lights
		allocate_lights(&new_scene->lights, p_scene->lights);
		//allocate objects
		allocate_objects(&new_scene->objects, p_scene->objects);
		cuda_push_scene(&d_r->scenes, new_scene);
		p_scene = p_scene->next;
	}
}*/

__host__ void allocate_camera(t_scene *h_scene)
{
	t_camera	*h_camera;
	t_camera	*d_camera;
	t_color		**d_pixel_map;
	int			i;

	h_camera = (t_camera *)malloc(sizeof(t_camera));
	memcpy(h_camera, h_scene->cameras, sizeof(t_camera));

	hipMalloc(&d_pixel_map, sizeof(t_color *) * h_scene->res.y);
	i = -1;
	while (++i < h_scene->res.x)
		hipMalloc(&(d_pixel_map[i]), sizeof(t_color) * h_scene->res.x);
	h_camera->pixel_map = d_pixel_map;

	hipMalloc(&d_camera, sizeof(t_camera));
	hipMemcpy(d_camera, h_camera, sizeof(t_camera), hipMemcpyHostToDevice);
}

__host__ void	allocate_lights(t_scene *h_scene)
{
	t_light *p_light;
	t_light *h_light;
	t_light *d_light;

	p_light = h_scene->lights;
	while (p_light)
	{
		//hipMallocManaged(&new_light, sizeof(t_light));
		//memcpy(new_light, p_light, sizeof(t_light));
		//cuda_push_light(d_lights, new_light);
		
		h_light = (t_light *)malloc(sizeof(t_light));
		memcpy(h_light, p_light, sizeof(t_light));

		hipMalloc(&d_light, sizeof(t_light));
		h_light->next = NULL;
		hipMemcpy(d_light, h_light, sizeof(t_light), hipMemcpyHostToDevice);
		
		p_light = p_light->next;
	}
}

__host__ void	allocate_objects(t_scene *h_scene)
{
	t_light *p_object;
	t_light *h_object;
	t_light *d_object;

	p_object = h_scene->lights;
	while (p_object)
	{
		//hipMallocManaged(&new_light, sizeof(t_light));
		//memcpy(new_light, p_object, sizeof(t_light));
		//cuda_push_object(d_objects, new_light);
		
		h_object = (t_light *)malloc(sizeof(t_light));
		memcpy(h_object, p_object, sizeof(t_light));

		hipMalloc(&d_object, sizeof(t_light));
		h_object->next = NULL;
		hipMemcpy(d_object, h_object, sizeof(t_light), hipMemcpyHostToDevice);
		
		p_object = p_object->next;
	}
}

__host__ void allocate_scene(t_raytracing_tools *h_r_tmp)
{
	t_scene	*h_scene;
	t_scene	*d_scene;

	h_scene = (t_scene *)malloc(sizeof(t_scene));
	memcpy(h_scene, h_r_tmp->scenes, sizeof(t_scene));

	allocate_camera(h_scene);
	allocate_lights(h_scene);
	allocate_objects(h_scene);

	hipMalloc(&d_scene, sizeof(t_scene));
	h_r_tmp->scenes = d_scene;
	hipMemcpy(d_scene, h_scene, sizeof(t_scene), hipMemcpyHostToDevice);
}

__host__ t_raytracing_tools  *allocate_memory(t_raytracing_tools *h_r)
{
	C(11)
	t_raytracing_tools	*d_r;
	t_raytracing_tools	*h_r_tmp;

	h_r_tmp = (t_raytracing_tools *)malloc(sizeof(t_raytracing_tools));
	memcpy(h_r_tmp, h_r, sizeof(t_raytracing_tools));

	// allocate_scenes(d_r, h_r);


	// hipMallocManaged(&d_r, sizeof(t_raytracing_tools));
	hipMalloc(&d_r, sizeof(t_raytracing_tools));
	hipMemcpy(d_r, h_r_tmp, sizeof(t_raytracing_tools), hipMemcpyHostToDevice);
	// printf("%d\n", d_r->pix.x);
	C(12)
	// memcpy(d_r, h_r, sizeof(t_raytracing_tools));
	C(13)
	return (d_r);
}

__global__ void render_pixel(t_raytracing_tools *r, t_color *d_pixel_map)
{
	// t_ray	cam_ray;

	(void)r;


	// int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	// int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int x = (blockDim.x * blockIdx.x) + threadIdx.x;
    int y = (blockDim.y * blockIdx.y) + threadIdx.y;
    int idx = r->scenes->res.x * y + x;
	// r->pix.x = pixel.x;
	// r->pix.y = pixel.y;
	// cam_ray = init_camera_ray(pixel, r->scenes);*/
	if (idx < r->scenes->res.x * r->scenes->res.y)
	{
		d_pixel_map[idx].x = 255;//cast_primary_ray(r, &cam_ray);
		d_pixel_map[idx].y = 255;//cast_primary_ray(r, &cam_ray);
		d_pixel_map[idx].z = 255;//cast_primary_ray(r, &cam_ray);
	}
}

__host__
void		render(t_raytracing_tools *r)
{
	t_raytracing_tools *d_r;
	t_color	*d_pixel_map;
	t_color *h_pixel_map;

	d_r = allocate_memory(r);
	hipMalloc((void**) &d_pixel_map, sizeof(t_color) * r->scenes->res.y * r->scenes->res.x);
	h_pixel_map = (t_color *)malloc(sizeof(t_color) * r->scenes->res.y * r->scenes->res.x);

	dim3 block_size;
	dim3 grid_size;

	h_pixel_map[0].x = -1;

	dim3 blockSize = dim3(32, 32, 1);
	dim3 gridSize = dim3((blockSize.x + r->scenes->res.x - 1) / blockSize.x, (blockSize.y + r->scenes->res.y - 1) / blockSize.y, 1);

	printf("grid size: %i, %i\n", gridSize.x, gridSize.y); 
	// dim3	blockSize(100);
	// dim3	gridSize(10, 10, 10);


	render_pixel<<<gridSize, blockSize>>>(d_r, d_pixel_map);

	// render_pixel<<<grid_size, block_size>>>(d_r, d_pixel_map);
	hipDeviceSynchronize();

	C(4)
	// hipMemcpy(r->scenes->cameras->pixel_map, d_r->scenes->cameras->pixel_map, sizeof(t_color *) * r->scenes->res.y, hipMemcpyDeviceToHost);
	// r->scenes->cameras->pixel_map = d_r->scenes->cameras->pixel_map;
	hipMemcpy(h_pixel_map, d_pixel_map, sizeof(t_color) * r->scenes->res.y * r->scenes->res.x, hipMemcpyDeviceToHost);
	// h_pixel_map[0].x = 255;
	C(5)
	// while (++i < r->scenes->res.y)
		// hipMemcpy(h_pixel_map[i], d_pixel_map[i], sizeof(t_color) * r->scenes->res.x, hipMemcpyDeviceToHost);
		//printf("%f\n", d_r->scenes->cameras->pixel_map[i][0].x);
		//memcpy(r->scenes->cameras->pixel_map[i], d_r->scenes->cameras->pixel_map[i], sizeof(t_color) * r->scenes->res.x);
		// r->scenes->cameras->pixel_map[i] = d_r->scenes->cameras->pixel_map[i];
	C(6)
	int i = -1;
	while (++i < 30) //r->scenes->res.y * r->scenes->res.x)
		printf("%f\n", h_pixel_map[i].x);

	t_pt2 coord;

	coord.y = -1;
	while (++coord.y < r->scenes->res.y)
	{
		coord.x = -1;
		while (++coord.x < r->scenes->res.x)
		{
			r->scenes->cameras->pixel_map[coord.y][coord.x] = h_pixel_map[r->scenes->res.y * coord.x + coord.y];
		}
	}
}

// __device__
// static void	display_percentage(int num)
// {
// 	ft_printf("Loading...%i%%\r", num);
// }
