#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/19 15:53:41 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"

/*
** Updates a camera's pixel_map (color of image pixels).
*/

#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void	printte_matrix(t_matrix m)
{
	int i;
	int	y;

	y = -1;
	while (++y < 4)
	{
		i = -1;
		while (++i < 4)
			printf("[%f]", m[y][i]);
		printf("\n");
	}
}

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map, t_pt2 tileId, int tile_size)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	t_dpt2				aa_i;
	float				sample_size;
	int					i;
	t_vec3				moyenne;

	r.pix.x = (tileId.x * tile_size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (tileId.y * tile_size) + (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene;
    r.idx = scene->res.x * r.pix.y + r.pix.x;

	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		// initialize ior list
		// r.ior_list = (float *)malloc(sizeof(float) * (scene->ray_depth + 1));
		// printf("AA de la scene : %d\n", scene->is_aa);
		if (scene->is_aa == 1)
		{
			aa_i.x = 0.5;
			aa_i.y = 0.5;
			memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
			cam_ray = init_camera_ray(&r, aa_i);
			d_pixel_map[r.idx] = filter(cast_primary_ray(&r, &cam_ray), scene->cameras->filter);
		}
		else
		{
			sample_size =  1 / (float)scene->is_aa;
			aa_i.x = 0.0;
			aa_i.y = 0.0;
			i = -1;
			moyenne.x = 0;
			moyenne.y = 0;
			moyenne.z = 0;
			while (++i < scene->is_aa * scene->is_aa)
			{
				aa_i.x += sample_size;
				if (i % (scene->is_aa - 1) == 0)
				{
					aa_i.x = 0.0;
					aa_i.y += sample_size;
				}
				memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
				cam_ray = init_camera_ray(&r, aa_i);
				moyenne = v_add(moyenne, col_to_vec(cast_primary_ray(&r, &cam_ray)));
			}
			moyenne.x /= (scene->is_aa * scene->is_aa);
			moyenne.y /= (scene->is_aa * scene->is_aa);
			moyenne.z /= (scene->is_aa * scene->is_aa);
			d_pixel_map[r.idx] = filter(vec_to_col(moyenne), scene->cameras->filter);
		}
	}
}

//'dis is wonderful
__global__ void create_anaglyph(t_color *left, t_color *right, t_scene *scene, int tile_size, t_pt2 tileId)
{
	// int	idx;

	// // printf("Debut du deuxieme kernel\n");

	// idx = scene->res.x * ((blockDim.y * blockIdx.y) + threadIdx.y) + ((blockDim.x * blockIdx.x) + threadIdx.x);

	// if (idx == 10)
	// {
	// 	printf("Other kernel\n");
	// 	printf("%d, %d, %d\n", right[10].r, right[10].g, right[10].b);
	// }


	int		idx;
	t_pt2	pixel;

	pixel.x = (tileId.x * tile_size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	pixel.y = (tileId.y * tile_size) + (blockDim.y * blockIdx.y) + threadIdx.y;
    idx = scene->res.x * pixel.y + pixel.x;

	if (pixel.x < scene->res.x && pixel.y < scene->res.y)
	{
		left[idx].g = right[idx].g;
		left[idx].b = right[idx].b;
	}
	// __syncthreads();
}

void		render(t_raytracing_tools *r, t_pt2 tileId)
{
	dim3 		blockSize;
	dim3 		gridSize;
	int			size;

	size = (r->settings.tile_size / BLOCK_DIM) + ((r->settings.tile_size % BLOCK_DIM) ? 1 : 0);
	blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	gridSize = dim3(size, size);

	hipEvent_t start, stop;
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	hipEventRecord(start);
	render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map, tileId, r->settings.tile_size);
	// printf("Iteration i = %d	\n", i++);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);


	hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();
if (errSync != hipSuccess) 
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	//beautiful....
	// printf("=============== EXECUTION ================== \n");
	// printf("Kernel duration: %f milliseconds\n", milliseconds);
	// printf("============================================ \n");

	// gpuErrchk((hipDeviceSynchronize()));

	// if (r->scene->is_3d)
	// {
	// 	printf("3d\n");
	// 	r->scene->cameras->pos.x += 0.2;
	// 	r->scene->cameras->filter = F_RIGHT_CYAN;
	// 	gpuErrchk((hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice)));
	// 	gpuErrchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	// 	// render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map_3d, tileId, r->settings.tile_size);
	// 	gpuErrchk((hipDeviceSynchronize()));
	// 	r->scene->cameras->pos.x -= 0.2;

	// 	// create_anaglyph<<<gridSize, blockSize>>>(r->d_pixel_map, r->d_pixel_map_3d, r->d_scene, r->settings.tile_size, tileId);

	// 	gpuErrchk((hipDeviceSynchronize()));
	// }
}