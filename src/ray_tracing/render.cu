#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/23 18:49:54 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"

/*
** Updates a camera's pixel_map (color of image pixels).
*/
#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void	printte_matrix(t_matrix m)
{
	int i;
	int	y;

	y = -1;
	while (++y < 4)
	{
		i = -1;
		while (++i < 4)
			printf("[%f]", m[y][i]);
		printf("\n");
	}
}

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map/*, t_pt2 *loading*/)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	int					idx;

	r.pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene;
    idx = scene->res.x * r.pix.y + r.pix.x;


	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		cam_ray = init_camera_ray(&r);	
		// d_pixel_map[idx] = filter(cast_primary_ray(&r, &cam_ray), scene->cameras->filter);
		d_pixel_map[idx] = filter(cast_primary_ray(&r, &cam_ray), scene->cameras->filter);
		// d_pixel_map[idx] = cast_primary_ray(&r, &cam_ray);
	}
	//loading->x = atomicAdd(&(loading->x), 1);
	//__syncthreads();
}

void		render(t_raytracing_tools *r)
{
	// t_clock start = clock();
	// t_color		*d_pixel_map;
	// t_color 	*h_pixel_map;
	// t_scene		*h_scene_to_array;
	// t_scene		*h_d_scene;
	// t_scene		*d_scene;
	dim3		block_size;
	dim3		grid_size;


	// size_t       available;
  	// size_t       total;

	

	// Preparation des deux structures temporarires
	// if (!(h_scene_to_array = (t_scene *)malloc(sizeof(t_scene))))
		// exit(0); // malloc error
	// if (!(h_d_scene = (t_scene *)malloc(sizeof(t_scene))))
	// 	exit(0); // malloc error
	// if (!(memcpy(h_scene_to_array, scene, sizeof(t_scene))))
	// 	exit(0);
	// memcpy(h_d_scene, scene, sizeof(t_scene));

	// Creation des tableaux 1D pour les objets et lumieres
	// h_scene_to_array->objects = list_to_array_objects(scene->objects);
	// h_scene_to_array->lights = list_to_array_lights(scene->lights);
	// h_scene_to_array->cameras = (t_camera *)malloc(sizeof(t_camera));
	

	// memcpy(h_scene_to_array->cameras, scene->cameras, sizeof(t_camera));
	
	// Allocation de la memoire GPU
	// gpuErrchk(hipMalloc(&(h_d_scene->lights), get_lights_array_length(h_scene_to_array->lights)));
	// gpuErrchk(hipMalloc(&(h_d_scene->objects), get_object_array_length(h_scene_to_array->objects)));
	// gpuErrchk(hipMalloc(&(h_d_scene->cameras), sizeof(t_camera)));
	// gpuErrchk(hipMalloc(&d_scene, sizeof(t_scene)));

	// Copie des tableaux du CPU vers le GPU, en passant par la structure contenant des pointeurs sur GPU
	// gpuErrchk(hipMemcpy(h_d_scene->cameras, h_scene_to_array->cameras, sizeof(t_camera), hipMemcpyHostToDevice));
	// gpuErrchk((hipMemcpy(h_d_scene->objects, h_scene_to_array->objects, get_object_array_length(h_scene_to_array->objects), hipMemcpyHostToDevice)));
	// gpuErrchk(hipMemcpy(h_d_scene->lights, h_scene_to_array->lights, get_lights_array_length(h_scene_to_array->lights), hipMemcpyHostToDevice));

	// Copie de la structure finale sur le GPU, contenant les pointeurs GPU
	// gpuErrchk(hipMemcpy(d_scene, h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	
	// Pixel map
	// h_pixel_map = (t_color *)malloc(sizeof(t_color) * r->scene->res.y * r->scene->res.x);
	
  	// hipMemGetInfo(&available, &total);
 	// printf("available memory: [%'lu]\ntotal memory:   [%'lu]\n", available, total);
	// gpuErrchk((hipMalloc(&d_pixel_map, sizeof(t_color) * scene->res.y * scene->res.x)));
	// hipMemGetInfo(&available, &total);
 	// printf("available memory: [%'lu]\ntotal memory:   [%'lu]\n", available, total);

	 //prep kernel
	dim3 blockSize 	= dim3(BLOCK_DIM, BLOCK_DIM, 1);
	dim3 gridSize	= dim3(r->scene->res.x / BLOCK_DIM + 1, r->scene->res.y / BLOCK_DIM + 1);


	// printf("gridsize: [%d][%d][%d] blocksize: [%d][%d][%d]\n", gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);
	
	// printf("Nouvelle position: %f\n", r->scene->objects->pos.x);
	// printf("d_scene : [%f]\n", r->d_scene->ka);;
	// clock_t start = clock();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map/*, progress*/);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("=============== EXECUTION ================== \n");
	printf("Kernel duration: %f milliseconds\n", milliseconds);
	printf("============================================ \n");

	// clock_t stop = clock();
	// printf("\nAppel du kernel : %f\n",
  	// (float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	
	// printf("%d %d %d\n", r->d_pixel_map[25].r, r->d_pixel_map[25].g, r->d_pixel_map[25].b );

	// gpuErrchk( hipPeekAtLastError() );
	gpuErrchk((hipDeviceSynchronize()));
	// start = clock();
	// memcpy(r->scene->cameras->pixel_map, r->d_pixel_map, sizeof(t_color) * r->scene->res.y * r->scene->res.x);
	// r->scene->cameras->pixel_map = r->d_pixel_map;
	// gpuErrchk(hipMemcpy(r->scene->cameras->pixel_map, r->d_pixel_map, sizeof(t_color) * r->scene->res.y * r->scene->res.x, hipMemcpyDeviceToHost));
	//  stop = clock();
	// printf("\nmemcpy pixelmap : %f\n",
  	// (float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f);
	// gpuErrchk((hipMemcpy(r->scene->cameras->pixel_map, h_pixel_map, sizeof(t_color) * r->scene->res.y * r->scene->res.x, hipMemcpyDeviceToHost)));
	// memcpy(r->scene->cameras->pixel_map, h_pixel_map, sizeof(t_color) * r->scene->res.y * r->scene->res.x);
	//free dat shit
	// hipFree(h_d_scene->cameras);
	// hipFree(h_d_scene->lights);
	// hipFree(h_d_scene->objects);
	// hipFree(d_scene);
	// hipFree(d_pixel_map);

	//free dis shit
	// free(h_scene_to_array->objects);
	// free(h_scene_to_array->lights);
	// free(h_scene_to_array->cameras);
	// free(h_scene_to_array);
	// free(h_d_scene);
	// free(h_pixel_map);
}
