#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/04 14:25:24 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.h"

/*
** Updates a camera's pixel_map (color of image pixels).
*/
#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void	printte_matrix(t_matrix m)
{
	int i;
	int	y;

	y = -1;
	while (++y < 4)
	{
		i = -1;
		while (++i < 4)
			printf("[%f]", m[y][i]);
		printf("\n");
	}
}

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map, t_pt2 tileId, int tile_size)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	int					idx;
	
	r.pix.x = (tileId.x * tile_size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (tileId.y * tile_size) + (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene;
	
    idx = scene->res.x * r.pix.y + r.pix.x;
	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		//initialize ior list
		r.ior_list = (float *)malloc(sizeof(float) * (scene->ray_depth + 1));
		memset(r.ior_list, 0, sizeof(float) * (scene->ray_depth + 1));
		cam_ray = init_camera_ray(&r);
		d_pixel_map[idx] = filter(cast_primary_ray(&r, &cam_ray), scene->cameras->filter);
		free(r.ior_list);
	}
}

//'dis is wonderful
__global__ void create_anaglyph(t_color *left, t_color *right, t_scene *scene, int tile_size, t_pt2 tileId)
{
	// int	idx;

	// // printf("Debut du deuxieme kernel\n");

	// idx = scene->res.x * ((blockDim.y * blockIdx.y) + threadIdx.y) + ((blockDim.x * blockIdx.x) + threadIdx.x);

	// if (idx == 10)
	// {
	// 	printf("Other kernel\n");
	// 	printf("%d, %d, %d\n", right[10].r, right[10].g, right[10].b);
	// }


	int		idx;
	t_pt2	pixel;

	pixel.x = (tileId.x * tile_size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	pixel.y = (tileId.y * tile_size) + (blockDim.y * blockIdx.y) + threadIdx.y;
    idx = scene->res.x * pixel.y + pixel.x;

	if (pixel.x < scene->res.x && pixel.y < scene->res.y)
	{
		left[idx].g = right[idx].g;
		left[idx].b = right[idx].b;
	}
	// __syncthreads();
}

void		render(t_raytracing_tools *r, t_pt2 tileId)
{
	dim3 		blockSize;
	dim3 		gridSize;
	int			size;

	size = (r->settings.tile_size / BLOCK_DIM) + ((r->settings.tile_size % BLOCK_DIM) ? 1 : 0);
	blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	gridSize = dim3(size, size);

	hipEvent_t start, stop;
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	hipEventRecord(start);
	render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map, tileId, r->settings.tile_size);
	// printf("Iteration i = %d	\n", i++);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	//beautiful....
	// printf("=============== EXECUTION ================== \n");
	// printf("Kernel duration: %f milliseconds\n", milliseconds);
	// printf("============================================ \n");

	gpuErrchk((hipDeviceSynchronize()));
	// if (r->scene->is_3d)
	// {
	// 	printf("3d\n");
	// 	r->scene->cameras->pos.x += 0.2;
	// 	r->scene->cameras->filter = F_RIGHT_CYAN;
	// 	gpuErrchk((hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice)));
	// 	gpuErrchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice));
	// 	// render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map_3d, tileId, r->settings.tile_size);
	// 	gpuErrchk((hipDeviceSynchronize()));
	// 	r->scene->cameras->pos.x -= 0.2;

	// 	// create_anaglyph<<<gridSize, blockSize>>>(r->d_pixel_map, r->d_pixel_map_3d, r->d_scene, r->settings.tile_size, tileId);

	// 	gpuErrchk((hipDeviceSynchronize()));
	// }
}