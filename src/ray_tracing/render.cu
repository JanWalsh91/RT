#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by jwalsh            #+#    #+#             */
/*   Updated: 2017/05/31 16:02:29 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "photon_mapping.h"
#include "../../inc/cuda_call.h"

/*
** Updates a camera's pixel_map (color of image pixels).
*/

#define N 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// __device__
// void	printte_matrix(t_matrix m)
// {
// 	int i;
// 	int	y;

// 	y = -1;
// 	while (++y < 4)
// 	{
// 		i = -1;
// 		while (++i < 4)
// 			printf("[%f]", m[y][i]);
// 		printf("\n");
// 	}
// }

__global__ void render_pixel(t_scene *scene, t_color *d_pixel_map, t_region *region_map, t_tile tile)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	t_dpt2				aa_i;
	float				sample_size;
	int					i;
	t_vec3				moyenne;
	int					i2;

	r.pix.x = (tile.id.x * tile.size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (tile.id.y * tile.size) + (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene; 
    r.idx = scene->res.x * r.pix.y + r.pix.x;
	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		if (region_map)
		{
			i2 = (r.pix.x % tile.size) + (r.pix.y % tile.size) * tile.size;
			// printf("idx: [%d]i2: %d\n", r.idx, i2);
			r.d_region_map = &region_map[i2];
		}
		// initialize ior list
		// r.ior_list = (float *)malloc(sizeof(float) * (scene->ray_depth + 1));
		// printf("AA de la scene : %d\n", scene->is_aa);
		if (scene->is_aa == 1)
		{
			aa_i.x = 0.5;
			aa_i.y = 0.5;
			memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
			cam_ray = init_camera_ray(&r, aa_i);
			d_pixel_map[r.idx] = filter(cast_primary_ray(&r, &cam_ray), scene->cameras->filter);
			if (region_map)
			{
				r.d_region_map->hit_pt = cam_ray.hit;
				r.d_region_map->ray_dir = cam_ray.dir;
				r.d_region_map->normal = v_scale(cam_ray.nhit, cam_ray.n_dir);
				r.d_region_map->kd = scene->objects[cam_ray.hit_obj].kd;
			
			
				if (r.idx == 1)
					printf("hit_pt: [%f, %f, %f], ray_dir: [%f, %f, %f], normal: [%f, %f, %f], kd: %f\n", r.d_region_map->hit_pt.x, r.d_region_map->hit_pt.y, r.d_region_map->hit_pt.z, 
					r.d_region_map->ray_dir.x, r.d_region_map->ray_dir.y, r.d_region_map->ray_dir.z, r.d_region_map->normal.x, r.d_region_map->normal.y, r.d_region_map->normal.z,
					r.d_region_map->kd);
			}
			
		}
		else
		{
			sample_size =  1 / (float)scene->is_aa;
			aa_i.x = 0.0;
			aa_i.y = 0.0;
			i = -1;
			moyenne.x = 0;
			moyenne.y = 0;
			moyenne.z = 0;
			while (++i < scene->is_aa * scene->is_aa)
			{
				aa_i.x += sample_size;
				if (i % (scene->is_aa - 1) == 0)
				{
					aa_i.x = 0.0;
					aa_i.y += sample_size;
				}
				memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
				cam_ray = init_camera_ray(&r, aa_i);
				moyenne = v_add(moyenne, col_to_vec(cast_primary_ray(&r, &cam_ray)));
			}
			moyenne.x /= (scene->is_aa * scene->is_aa);
			moyenne.y /= (scene->is_aa * scene->is_aa);
			moyenne.z /= (scene->is_aa * scene->is_aa);
			d_pixel_map[r.idx] = filter(vec_to_col(moyenne), scene->cameras->filter);
		}
	}
}

//'dis is wonderful
__global__ void create_anaglyph(t_color *left, t_color *right, t_scene *scene, t_tile tile)
{
	int		idx;
	t_pt2	pixel;

	pixel.x = (tile.id.x * tile.size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	pixel.y = (tile.id.y * tile.size) + (blockDim.y * blockIdx.y) + threadIdx.y;
  	idx = scene->res.x * pixel.y + pixel.x;

	if (pixel.x < scene->res.x && pixel.y < scene->res.y)
	{
		left[idx].g = right[idx].g;
		left[idx].b = right[idx].b;
	}
	// __syncthreads();
}

// Trouver un moyen pour appeler cette fonction ><
void	update_camera(t_camera *camera)
{
	t_vec3	forward;
	t_vec3	right;
	t_vec3	up;

	forward = v_norm(camera->dir);
	if (v_dot(forward, v_new(0, 1, 0)) > 0.9999 ||
		v_dot(forward, v_new(0, 1, 0)) < -0.9999)
		right = v_new(1, 0, 0);
	else
		right = v_norm(v_cross(v_new(0, 1, 0), forward));
	up = v_norm(v_cross(forward, right));
	m_new_identity(&camera->ctw);
	camera->ctw[0][0] = right.x;
	camera->ctw[0][1] = right.y;
	camera->ctw[0][2] = right.z;
	camera->ctw[1][0] = up.x;
	camera->ctw[1][1] = up.y;
	camera->ctw[1][2] = up.z;
	camera->ctw[2][0] = forward.x;
	camera->ctw[2][1] = forward.y;
	camera->ctw[2][2] = forward.z;
	camera->ctw[3][0] = camera->pos.x;
	camera->ctw[3][1] = camera->pos.y;
	camera->ctw[3][2] = camera->pos.z;
}

void		render(t_raytracing_tools *r, t_tile tile)
{
	dim3 		blockSize;
	dim3 		gridSize;
	int			size;

	size = (tile.size / BLOCK_DIM) + ((tile.size % BLOCK_DIM) ? 1 : 0);
	blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	gridSize = dim3(size, size);

	// hipEvent_t start, stop;
	// hipEventCreate(&start); 
	// hipEventCreate(&stop);
	// hipEventRecord(start);
	printf("launch kernel:\n");
	render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map, r->d_region_map, tile);
	// printf("Iteration i = %d	\n", i++);
	// hipEventRecord(stop);
	// hipEventSynchronize(stop);
	// float milliseconds = 0;
	// hipEventElapsedTime(&milliseconds, start, stop);


	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	if (errSync != hipSuccess || errAsync != hipSuccess)
		exit(-1);
	//beautiful....
	// printf("=============== EXECUTION ================== \n");
	// printf("Kernel duration: %f milliseconds\n", milliseconds);
	// printf("============================================ \n");

	// gpuErrchk((hipDeviceSynchronize()));

	if (r->scene->is_3d)
	{
		printf("3d\n");
		//IS HARDCODING THESE VALUES CORRECT?
		r->scene->cameras->pos.x += 0.05;
		r->scene->cameras->dir.x -= 0.01;
		r->scene->cameras->dir = v_norm(r->scene->cameras->dir);
		update_camera(r->scene->cameras);
		r->scene->cameras->filter = F_RIGHT_CYAN;
		gpuErrchk(hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice));
		gpuErrchk((hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice)));
		render_pixel<<<gridSize, blockSize>>>(r->d_scene, r->d_pixel_map_3d, r->d_region_map, tile);
		gpuErrchk((hipDeviceSynchronize()));
		r->scene->cameras->pos.x -= 0.05;
		r->scene->cameras->dir.x += 0.01;
		r->scene->cameras->dir = v_norm(r->scene->cameras->dir);
		update_camera(r->scene->cameras);
		r->scene->cameras->filter = F_LEFT_RED;
		gpuErrchk(hipMemcpy(r->h_d_scene->cameras, r->scene->cameras, sizeof(t_camera), hipMemcpyHostToDevice));
		gpuErrchk((hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene), hipMemcpyHostToDevice)));
		create_anaglyph<<<gridSize, blockSize>>>(r->d_pixel_map, r->d_pixel_map_3d, r->d_scene, tile);
		gpuErrchk((hipDeviceSynchronize()));
	}

}
