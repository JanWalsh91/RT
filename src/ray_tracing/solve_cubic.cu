#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   solve_cubic.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/09 14:16:04 by tgros             #+#    #+#             */
/*   Updated: 2017/05/10 14:04:48 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"

/*
** Solves a cubic equation.
** Returns true if the cubic has at least one solution.
*/

__device__
bool	solve_cubic(t_vec3 q, float d, t_vec3im *sol)
{
	t_vec4	tmp;
	t_vec3	res;
	t_vec3	tmp2;

	res.x = (((3 * q.z) / q.x) - ((q.y * q.y) / (q.x * q.x))) / 3;
	res.y = ((((2 * powf(q.y, 3)) / powf(q.x, 3))) - (9 * q.y * q.z / (q.x * q.x)) + (27 * d / q.x)) / 27;
	res.z = (res.y * res.y / 4) + (powf(res.x, 3) / 27);

	if (res.x < 0.0001 && res.x > -0.0001 && res.z  < 0.0001 && res.z > -0.0001 && res.z  < 0.0001 && res.z > -0.0001)
	{
		sol->x.r = cbrtf(d / q.x) * -1;
		sol->y.r = sol->x.r;
		sol->z.r = sol->x.r;
		sol->x.i = NAN;
		sol->y.i = NAN;
		sol->z.i = NAN;
	}
	else if (res.z > 0)
	{
		tmp.w = -(res.y / 2.0) + sqrtf(res.z);
		tmp.x = cbrtf(tmp.w);
		tmp.y = -(res.y / 2.0) - sqrtf(res.z);
		tmp.z = cbrtf(tmp.y);
		sol->x.r = (tmp.x + tmp.z) - (q.y / (q.x * 3));
		sol->y.r = -(tmp.x + tmp.z) / 2.0 - (q.y / (q.x * 3));
		sol->z.r = -(tmp.x + tmp.z) / 2.0 - (q.y / (q.x * 3));
		sol->x.i = NAN;
		sol->y.i = (tmp.x - tmp.z) * (sqrtf(3.0) / 2.0);
		sol->z.i = -(tmp.x - tmp.z) * (sqrtf(3.0) / 2.0);
	}
	else if (res.z <= 0)
	{
		tmp2.x = sqrtf(((res.y * res.y) / 4) - res.z);
		tmp2.y = cbrtf(tmp2.x);
		tmp2.z = acosf(-(res.y / (2 * tmp2.x)));
		tmp.w = tmp2.y * -1;
		tmp.x = cosf(tmp2.z / 3.0);
		tmp.y = sqrtf(3.0) * sinf(tmp2.z / 3.0);
		tmp.z = (q.y / (3.0 * q.x)) * -1;
		sol->x.r = ((2 * tmp2.y) * cosf(tmp2.z / 3.0)) - (q.y / (3.0 * q.x));
		sol->y.r = tmp.w * (tmp.x + tmp.y) + tmp.z;
		sol->z.r = tmp.w * (tmp.x - tmp.y) + tmp.z;
		sol->x.i = NAN;
		sol->y.i = NAN;
		sol->z.i = NAN;
	}
	return (true);
}