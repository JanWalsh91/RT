#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   solve_cubic.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/09 14:16:04 by tgros             #+#    #+#             */
/*   Updated: 2017/06/05 10:22:37 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

// NOT NORMED

#include "rt.cuh"

/*
** Solves a cubic equation.
** Returns true if the cubic has at least one solution.
*/

__device__
void	solve_third_case(t_vec3 q, float d, t_vec3 res, t_vec3im *sol)
{
	t_vec4 tmp;
	t_vec3 tmp2;

	tmp2.x = sqrtf(((res.y * res.y) / 4) - res.z);
	tmp2.y = cbrtf(tmp2.x);
	tmp2.z = acosf(-(res.y / (2 * tmp2.x)));
	tmp.w = tmp2.y * -1;
	tmp.x = cosf(tmp2.z / 3.0);
	tmp.y = sqrtf(3.0) * sinf(tmp2.z / 3.0);
	tmp.z = (q.y / (3.0 * q.x)) * -1;
	sol->x.r = ((2 * tmp2.y) * cosf(tmp2.z / 3.0)) - (q.y / (3.0 * q.x));
	sol->y.r = tmp.w * (tmp.x + tmp.y) + tmp.z;
	sol->z.r = tmp.w * (tmp.x - tmp.y) + tmp.z;
	sol->x.i = NAN;
	sol->y.i = NAN;
	sol->z.i = NAN;
}

__device__
void	solve_second_case(t_vec3 q, float d, t_vec3 res, t_vec3im *sol)
{
	t_vec4 tmp;

	tmp.w = -(res.y / 2.0) + sqrtf(res.z);
	tmp.x = cbrtf(tmp.w);
	tmp.y = -(res.y / 2.0) - sqrtf(res.z);
	tmp.z = cbrtf(tmp.y);
	sol->x.r = (tmp.x + tmp.z) - (q.y / (q.x * 3));
	sol->y.r = -(tmp.x + tmp.z) / 2.0 - (q.y / (q.x * 3));
	sol->z.r = -(tmp.x + tmp.z) / 2.0 - (q.y / (q.x * 3));
	sol->x.i = NAN;
	sol->y.i = (tmp.x - tmp.z) * (sqrtf(3.0) / 2.0);
	sol->z.i = -(tmp.x - tmp.z) * (sqrtf(3.0) / 2.0);
}

__device__
void	solve_first_case(t_vec3 q, float d, t_vec3im *sol)
{
	sol->x.r = cbrtf(d / q.x) * -1;
	sol->y.r = sol->x.r;
	sol->z.r = sol->x.r;
	sol->x.i = NAN;
	sol->y.i = NAN;
	sol->z.i = NAN;
}

__device__
bool	solve_cubic(t_vec3 q, float d, t_vec3im *sol)
{
	t_vec3	res;

	// q.x = 2;
	// q.y = -4;
	// q.z = -22;
	// d = 24;
	//printf("a b c d cubic: %f, %f, %f, %f\n", q.x, q.y, q.z, d);
	res.x = (((3 * q.z) / q.x) - ((q.y * q.y) / (q.x * q.x))) / 3; //f 
	res.y = ((((2 * powf(q.y, 3)) / powf(q.x, 3))) - (9 * q.y * q.z / (q.x * q.x)) + (27 * d / q.x)) / 27; // g
	res.z = (res.y * res.y / 4) + (powf(res.x, 3) / 27); // h
	//printf("f g h cubic: %f, %f, %f, \n", res.x, res.y, res.z);
	if (res.x < 0.0001 && res.x > -0.0001 && res.z  < 0.0001 && res.z > -0.0001 && res.z  < 0.0001 && res.z > -0.0001)
		solve_first_case(q, d, sol);
	else if (res.z > 0)
		solve_second_case(q, d, res, sol);
	else if (res.z <= 0)
		solve_third_case(q, d, res, sol);
	return (true);
}