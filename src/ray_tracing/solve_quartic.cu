#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   solve_quartic.cu                                   :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/09 13:46:42 by tgros             #+#    #+#             */
/*   Updated: 2017/05/18 16:31:25 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"

/*
** Solves a quartic equation.
** Updates r1 and r2 if at least a solution exists. ?????????????????????????????
** Returns true if the quartic has at least one solution.
*/


__device__
void	solve_four_root(t_quartic *qua, t_vec3 fgh, t_vec3im sol_cubic, t_vec4im *sol)
{
	t_squart q;

	q.p = sqrt(sol_cubic.x.r);
	q.q = sqrt(sol_cubic.z.r);
	q.r = -fgh.y / (8 * q.p * q.q);
	q.s = qua->b / (4 * qua->a);
	sol->w.r = q.p + q.q + q.r -q.s;
	sol->x.r = q.p - q.q - q.r -q.s;
	sol->y.r = -q.p + q.q - q.r -q.s;
	sol->z.r = -q.p - q.q + q.r -q.s;
	sol->w.i = NAN;
	sol->x.i = NAN;
	sol->y.i = NAN;
	sol->z.i = NAN;
}

__device__
void	solve_im_root(t_quartic *qua, t_vec3 fgh, t_vec3im sol_cubic, t_vec4im *sol)
{
	t_squart q;

	q.pim = ft_sqrtfi(sol_cubic.y);
	q.qim.r = q.pim.r;
	q.qim.i = -q.pim.i;
	q.r = -fgh.y / (8 * ((q.pim.r * q.qim.r) - (q.pim.i * q.qim.i)));
	q.s = qua->b / (4 * qua->a);
	sol->w.r = q.pim.r + q.qim.r + q.r - q.s; 
	sol->x.r = q.pim.r - q.qim.r - q.r - q.s;
	sol->y.r = -q.pim.r + q.qim.r - q.r - q.s;
	sol->z.r = -q.pim.r - q.qim.r + q.r - q.s;
	sol->w.i = NAN;
	sol->x.i = q.pim.i - q.qim.i;
	sol->y.i = -q.pim.i + q.qim.i;
	sol->z.i = NAN;
}

__device__
bool	solve_quartic(t_quartic *qua, t_vec4im *sol)
{
	t_vec3		fgh;
	t_vec3im	sol_cubic;
	
	// divide all members by a
	qua->a = 1; 
	qua->b = -0.25;
	qua->c = -0.85;
	qua->d = 1.45;
	qua->e = -4.35;
	qua->b /= qua->a;
	qua->c /= qua->a;
	qua->d /= qua->a;
	qua->e /= qua->a;
	qua->a /= qua->a;
	//printf("After complication : %f, %f, %f, %f, %f\n", qua->a, qua->b, qua->c, qua->d, qua->e);
	fgh.x = qua->c - (3 * qua->b * qua->b / 8); // f
	fgh.y = qua->d + ((qua->b * qua->b * qua->b) / 8) - (qua->b * qua->c / 2); // g
	fgh.z = qua->e - (3 * pow(qua->b, 4) / 256) + (qua->b * qua->b * qua->c / 16) - (qua->b * qua->d / 4); // h
	//printf("f g h : %f, %f, %f\n", fgh.x, fgh.y, fgh.z);

	// Y3 + (f/2)*Y2 + ((f2 -4*h)/16)*Y -g2/64 = 0
	// solve cubic equation 
	// get 3 roots
	solve_cubic(v_new(1, fgh.x / 2.0, ((fgh.x * fgh.x - 4 * fgh.z) / 16)), - (fgh.y * fgh.y) / 64, &sol_cubic);

	// printf("Sols cubics x : %f, %f, \n", sol_cubic.x.r, sol_cubic.x.i);
	// printf("Sols cubics y : %f, %f, \n", sol_cubic.y.r, sol_cubic.y.i);
	// printf("Sols cubics z : %f, %f, \n", sol_cubic.z.r, sol_cubic.z.i);
	if (!isnan(sol_cubic.y.i) || !isnan(sol_cubic.z.i))
		solve_im_root(qua, fgh, sol_cubic, sol);
	else
		solve_four_root(qua, fgh, sol_cubic, sol);
	//printf("pim %f %f\n qim %f %f\n", pim.r, pim.i, qim.r, qim.i);
	printf("sol quartic\nX1 : %f, %f\nX2 : %f, %f\nX3 : %f. %f\nX4 : %f, %f\n", sol->w.r, sol->w.i,
		sol->x.r, sol->x.i, sol->y.r, sol->y.i, sol->z.r, sol->z.i);
	//printf("rs : %f, %f,\n",r, s);

	//printf("Before simplification : %f, %f, %f, %f, %f sols : %f, %f, %f, %f\n", qua->a, qua->b, qua->c, qua->d, qua->e, sol->w, sol->x, sol->y, sol->z);

	// if (sol->w < 0 && sol->x < 0 && sol->y < 0 && sol->z < 0)
	// 	return (false);
	return (true);
}