#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   solve_quartic.cu                                   :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/09 13:46:42 by tgros             #+#    #+#             */
/*   Updated: 2017/05/10 14:20:53 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"

/*
** Solves a quartic equation.
** Updates r1 and r2 if at least a solution exists.
** Returns true if the quartic has at least one solution.
*/

__device__
bool	solve_quartic(t_quartic *qua, t_vec4 *sol)
{
	t_vec3		fgh;
	t_vec3im	sol_cubic;
	float p, q, r, s;
	// divide all members by a
	qua->b /= qua->a;
	qua->c /= qua->a;
	qua->d /= qua->a;
	qua->e /= qua->a;
	qua->a = 1.0;

	// printf("After simplification : %f, %f, %f, %f, %f\n", qua->a, qua->b, qua->c, qua->d, qua->e);

	fgh.x = qua->c - (3 * qua->b * qua->b / 8);
	fgh.y = qua->d + ((qua->b * qua->b * qua->b) / 8) - (qua->b * qua->c / 2);
	fgh.z = qua->e - (3 * pow(qua->b, 4) / 256) + (qua->b * qua->b * qua->c / 16) - (qua->b * qua->d / 4);

	printf("f g h : %f, %f, %f\n", fgh.x, fgh.y, fgh.z);

	// Y3 + (f/2)*Y2 + ((f2 -4*h)/16)*Y -g2/64 = 0
	// solve cubic equation 
	// get 3 roots

	solve_cubic(v_new(1, fgh.x / 2.0, ((fgh.x * fgh.x - 4 * fgh.z) / 16)), - (fgh.y * fgh.y) / 64, &sol_cubic);

	printf("Sols cubics : %f, %f, %f\n", sol_cubic.x, sol_cubic.y, sol_cubic.z);

	if (!isnan(sol_cubic.y.i) || !isnan(sol_cubic.z.i))
	{
		p = ft_sqrtfi(sol_cubic.y).r;
		q = ft_sqrtfi(sol_cubic.z).r;
		r = -fgh.y / (8 * p * q);
		s = qua->b / (4 * qua->a);
		sol->w = p + q + r - s; 
		sol->x = NAN;
		sol->y = NAN;
		sol->z = -p -q + r - s;
	}
	else
	{
		p = sqrt(sol_cubic.x.r);
		q = sqrt(sol_cubic.z.r);
		r = -fgh.y / (8 * p * q);
		s = qua->b / (4 * qua->a);
		sol->w = p + q + r -s;
		sol->x = p - q - r -s;
		sol->y = -p + q - r -s;
		sol->z = -p - q + r -s;
	}

	printf("pqrs : %f, %f, %f, %f\n", p, q, r, s);

	printf("Before simplification : %f, %f, %f, %f, %f sols : %f, %f, %f, %f\n", qua->a, qua->b, qua->c, qua->d, qua->e, sol->w, sol->x, sol->y, sol->z);

	if (sol->w < 0 && sol->x < 0 && sol->y < 0 && sol->z < 0)
		return (false);
	return (true);
}