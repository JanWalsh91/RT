#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   solve_quartic.cu                                   :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/09 13:46:42 by tgros             #+#    #+#             */
/*   Updated: 2017/05/09 16:12:20 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"

/*
** Solves a quartic equation.
** Updates r1 and r2 if at least a solution exists.
** Returns true if the quartic has at least one solution.
*/

__device__
bool	solve_quartic(t_quartic *qua, t_vec4 *sol)
{
	t_vec3	fgh;
	t_vec3	sol_cubic;
	// divide all members by a
	printf("Before simplification : %f, %f, %f, %f\n", qua->a, qua->b, qua->c, qua->d, qua->e);
	qua->b /= qua->a;
	qua->c /= qua->a;
	qua->d /= qua->a;
	qua->e /= qua->a;
	qua->a = 1.0;

	printf("After simplification : %f, %f, %f, %f\n", qua->a, qua->b, qua->c, qua->d, qua->e);

	fgh.x = qua->c - (3 * qua->b * qua->b / 8);
	fgh.y = qua->d + ((qua->b * qua->b * qua->b) / 8) - (qua->b * qua->c / 2);
	fgh.z = qua->e - (3 * pow(qua->b, 4) / 256) + (qua->b * qua->b * qua->c / 16) - (qua->b * qua->d / 4);

	printf("f g h : %f, %f, %f\n", fgh.x, fgh.y, fgh.z);

	// Y3 + (f/2)*Y2 + ((f2 -4*h)/16)*Y -g2/64 = 0
	// solve cubic equation 
	// get 3 roots

	solve_cubic(v_new(1, fgh.x / 2.0, ((fgh.x * fgh.x - 4 * fgh.z) / 16)), - (fgh.y * fgh.y) / 64, &sol_cubic);

	printf("Sols cubics : %f, %f, %f\n", sol_cubic.x, sol_cubic.y, sol_cubic.z);

	float p = sol_cubic.x > 0 ? sqrt(sol_cubic.x) : 0;
	float q = sqrt(sol_cubic.z);
	float r = -fgh.y / (8 * p * q);
	float s = qua->b / (4 * qua->a);

	printf("pqrs : %f, %f, %f, %f\n", p, q, r, s);

	sol->w = p + q + r -s;
	sol->x = p - q - r -s;
	sol->y = -p + q - r -s;
	sol->z = -p - q + r -s;

	if (sol->w < 0 && sol->x < 0 && sol->y < 0 && sol->z < 0)
		return (false);
	return (true);
}